#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>
#include <fmt/core.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdlib>
#include <initializer_list>
#include <iomanip>
#include <iostream>
#include <limits>
#include <map>
#include <memory>
#include <set>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "../optimization/optimization.hpp"
#include "../profiling/annotation.hpp"
#include "../profiling/memoryManager.hpp"
#include "../utilities/cudaUtilities.hpp"

constexpr size_t N = 512;
constexpr size_t B = 128;

constexpr size_t T = N / B;

// Credit to: https://math.stackexchange.com/questions/357980/how-to-generate-random-symmetric-positive-definite-matrices-using-matlab
void generateRandomSymmetricPositiveDefiniteMatrix(double *h_A, const size_t n) {
  srand(time(NULL));

  double *h_A_temp = (double *)malloc(n * n * sizeof(double));

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      h_A_temp[i * n + j] = (float)rand() / (float)RAND_MAX;

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      h_A[i * n + j] = 0.5 * (h_A_temp[i * n + j] + h_A_temp[j * n + i]);

  for (int i = 0; i < n; i++) h_A[i * n + i] = h_A[i * n + i] + n;
}

void printSquareMatrix(double *h_A, const size_t n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (j != 0) std::cout << " ";
      std::cout << std::setw(6) << std::setprecision(3) << h_A[i * n + j];
    }
    std::cout << std::endl;
  }
}

// Restore the element order before blocks were moved to contiguous spaces.
// Set upper triangle entries (excluding diagonal entries) in column-major order to zero.
// Transpose to row-major order.
void cleanTiledCholeskyDecompositionResult(double *L, const int n, const int b) {
  auto L_copy = std::make_unique<double[]>(N * N);
  memcpy(L_copy.get(), L, N * N * sizeof(double));

  const int t = n / b;
  for (int i = 0; i < t; i++)
    for (int j = 0; j < t; j++)
      for (int k = 0; k < b; k++)
        for (int l = 0; l < b; l++)
          L[(i * b + k) + (j * b * n + l * n)] = L_copy[(b * b) * (i + j * t) + k + l * b];

  for (int i = 0; i < n; i++) {
    for (int j = i + 1; j < n; j++) {
      L[i + j * n] = 0;
      std::swap(L[i + j * n], L[i * n + j]);
    }
  }
}

bool verifyCholeskyDecomposition(double *A, double *L, const int n, bool verbose = false) {
  auto newA = std::make_unique<double[]>(n * n);
  memset(newA.get(), 0, n * n * sizeof(double));
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
        newA[i * n + j] += L[i * n + k] * L[k + j * n];
      }
    }
  }

  double error = 0;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      error += fabs(A[i * n + j] - newA[i * n + j]);
    }
  }

  if (verbose) {
    fmt::print("A:\n");
    printSquareMatrix(A, n);

    fmt::print("\nnewA:\n");
    printSquareMatrix(newA.get(), n);

    fmt::print("\nL:\n");
    printSquareMatrix(L, n);
    fmt::print("\n");

    fmt::print("error = {:.6f}\n", error);
  }

  return error <= 1e-6;
}

typedef std::pair<int, int> MatrixTile;

class TiledCholeskyGraphCreator {
 public:
  TiledCholeskyGraphCreator(hipStream_t stream, hipGraph_t graph) : stream(stream), graph(graph) {
    this->lastModifiedTile = std::make_pair(-1, -1);
  }
  void beginCaptureOperation(MatrixTile tileToWrite, std::initializer_list<MatrixTile> tilesToRead) {
    auto tiles = std::vector<MatrixTile>(tilesToRead);
    tiles.push_back(tileToWrite);
    auto dependencies = this->getDependencies(tiles);

    this->lastModifiedTile = tileToWrite;
    this->lastDependencies = dependencies;

    checkCudaErrors(hipStreamBeginCaptureToGraph(this->stream, this->graph, dependencies.data(), nullptr, dependencies.size(), hipStreamCaptureModeGlobal));
  }

  void endCaptureOperation() {
    assert(this->lastModifiedTile.first != -1 && this->lastModifiedTile.second != -1);
    checkCudaErrors(hipStreamEndCapture(this->stream, &this->graph));
    this->tileLastModifiedByMap[this->lastModifiedTile] = this->getTailOfLastCapturedNodeChain();
    this->lastModifiedTile = std::make_pair(-1, -1);
  };

 private:
  std::map<MatrixTile, hipGraphNode_t> tileLastModifiedByMap;
  std::map<hipGraphNode_t, bool> visited;
  hipStream_t stream;
  hipGraph_t graph;
  MatrixTile lastModifiedTile;
  std::vector<hipGraphNode_t> lastDependencies;

  std::vector<hipGraphNode_t> getDependencies(std::vector<MatrixTile> tiles) {
    std::vector<hipGraphNode_t> dependencies;
    for (auto tile : tiles) {
      auto it = this->tileLastModifiedByMap.find(tile);
      if (it != this->tileLastModifiedByMap.end()) {
        dependencies.push_back(it->second);
      }
    }

    auto dedupedEnd = std::unique(dependencies.begin(), dependencies.end());
    dependencies.resize(std::distance(dependencies.begin(), dedupedEnd));
    return dependencies;
  }

  hipGraphNode_t getTailOfLastCapturedNodeChain() {
    if (lastDependencies.size() == 0) {
      size_t numEdges;
      checkCudaErrors(hipGraphGetEdges(this->graph, nullptr, nullptr, &numEdges));
      auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
      auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
      checkCudaErrors(hipGraphGetEdges(this->graph, from.get(), to.get(), &numEdges));

      std::map<hipGraphNode_t, bool> hasOutGoingEdge;
      std::set<hipGraphNode_t> noOutGoingEdgeNodes;
      for (int i = 0; i < numEdges; i++) {
        hasOutGoingEdge[from[i]] = true;
        noOutGoingEdgeNodes.erase(from[i]);
        if (!hasOutGoingEdge[to[i]])
          noOutGoingEdgeNodes.insert(to[i]);
      }

      assert(noOutGoingEdgeNodes.size() == 1);

      return *noOutGoingEdgeNodes.begin();
    } else {
      auto nodeBeforeChain = lastDependencies[0];
      size_t numDependentNodes;
      checkCudaErrors(hipGraphNodeGetDependentNodes(nodeBeforeChain, nullptr, &numDependentNodes));

      assert(numDependentNodes > 0);

      auto dependentNodes = std::make_unique<hipGraphNode_t[]>(numDependentNodes);
      checkCudaErrors(hipGraphNodeGetDependentNodes(nodeBeforeChain, dependentNodes.get(), &numDependentNodes));

      hipGraphNode_t chainBeginningNode;
      for (int i = 0; i < numDependentNodes; i++) {
        if (!visited[dependentNodes[i]]) {
          chainBeginningNode = dependentNodes[i];
          break;
        }
      }

      auto u = chainBeginningNode;
      while (true) {
        visited[u] = true;
        checkCudaErrors(hipGraphNodeGetDependentNodes(u, nullptr, &numDependentNodes));
        if (numDependentNodes == 0) break;

        assert(numDependentNodes == 1);

        hipGraphNode_t v;
        checkCudaErrors(hipGraphNodeGetDependentNodes(u, &v, &numDependentNodes));
        u = v;
      }

      return u;
    }
  }
};

void initializeHostData(double *h_originalMatrix) {
  generateRandomSymmetricPositiveDefiniteMatrix(h_originalMatrix, N);
}

__global__ void storeBlockMatrixInContiguousSpace(double *d_matrix, double *d_originalMatrix) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  size_t i = (idx % N) / B;
  size_t k = (idx % N) - (i * B);
  size_t j = (idx / N) / B;
  size_t l = (idx / N) - (j * B);

  if (i >= T || j >= T || k >= B || l >= B) return;

  d_matrix[(B * B) * (i + j * T) + k + l * B] = d_originalMatrix[(i * B + k) + (j * B * N + l * N)];
}

void initializeDeviceData(double *h_originalMatrix, double *d_matrix) {
  double *d_originalMatrix;
  checkCudaErrors(hipMalloc(&d_originalMatrix, N * N * sizeof(double)));
  checkCudaErrors(hipMemcpy(d_originalMatrix, h_originalMatrix, N * N * sizeof(double), hipMemcpyHostToDevice));

  // Reorder elements in d_matrix, such that each block matrix is stored in a contiguous space
  constexpr size_t NUM_THREADS = 1024;
  constexpr size_t NUM_BLOCKS = (N * N + NUM_THREADS) / NUM_THREADS;
  storeBlockMatrixInContiguousSpace<<<NUM_BLOCKS, NUM_THREADS>>>(d_matrix, d_originalMatrix);

  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipFree(d_originalMatrix));
}

void tiledCholesky(bool optimized) {
  // Initialize data
  auto h_originalMatrix = std::make_unique<double[]>(N * N);  // Column-major
  initializeHostData(h_originalMatrix.get());

  // Initialize device data
  double *d_matrix;
  checkCudaErrors(hipMallocManaged(&d_matrix, N * N * sizeof(double)));
  initializeDeviceData(h_originalMatrix.get(), d_matrix);

  // Register matrix block addresses
  for (int i = 0; i < T; i++)
    for (int j = 0; j < T; j++)
      registerManagedMemoryAddress(d_matrix + (B * B) * (i + j * T), B * B * sizeof(double));

  auto getMatrixBlock = [&](int i, int j) {
    return d_matrix + (B * B) * (i + j * T);
  };

  // Initialize libraries
  hipsolverHandle_t cusolverDnHandle;
  hipsolverDnParams_t cusolverDnParams;
  hipblasHandle_t cublasHandle;
  checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));
  checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));
  checkCudaErrors(hipblasCreate(&cublasHandle));

  // Prepare constants
  double *one, *minusOne;
  checkCudaErrors(hipMallocManaged(&one, sizeof(double)));
  checkCudaErrors(hipMallocManaged(&minusOne, sizeof(double)));
  *one = 1.0;
  *minusOne = -1.0;

  // Prepare buffer for potrf
  size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
  checkCudaErrors(hipsolverDnXpotrf_bufferSize(
    cusolverDnHandle,
    cusolverDnParams,
    HIPBLAS_FILL_MODE_LOWER,
    B,
    HIP_R_64F,
    d_matrix,
    B,
    HIP_R_64F,
    &workspaceInBytesOnDevice,
    &workspaceInBytesOnHost
  ));
  void *h_workspace, *d_workspace;
  int *d_info;
  checkCudaErrors(hipMallocManaged(&h_workspace, workspaceInBytesOnHost));
  checkCudaErrors(hipMallocManaged(&d_workspace, workspaceInBytesOnDevice));
  checkCudaErrors(hipMallocManaged(&d_info, sizeof(int)));

  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t s;
  checkCudaErrors(hipStreamCreate(&s));

  checkCudaErrors(hipsolverSetStream(cusolverDnHandle, s));
  checkCudaErrors(hipblasSetStream(cublasHandle, s));

  auto tiledCholeskyGraphCreator = std::make_unique<TiledCholeskyGraphCreator>(s, graph);

  for (int k = 0; k < T; k++) {
    // A[k][k] = POTRF(A[k][k])
    // L[k][k] = POTRF(A[k][k])
    tiledCholeskyGraphCreator->beginCaptureOperation(
      std::make_pair(k, k),
      {std::make_pair(k, k)}
    );
    annotateNextKernel({getMatrixBlock(k, k)}, {getMatrixBlock(k, k)}, s);
    checkCudaErrors(hipsolverDnXpotrf(
      cusolverDnHandle,
      cusolverDnParams,
      HIPBLAS_FILL_MODE_LOWER,
      B,
      HIP_R_64F,
      getMatrixBlock(k, k),
      B,
      HIP_R_64F,
      d_workspace,
      workspaceInBytesOnDevice,
      h_workspace,
      workspaceInBytesOnHost,
      d_info
    ));
    tiledCholeskyGraphCreator->endCaptureOperation();

    for (int i = k + 1; i < T; i++) {
      // A[i][k] = TRSM(A[k][k], A[i][k])
      // L[i][k] * L[k][k]^T = A[i][k]
      tiledCholeskyGraphCreator->beginCaptureOperation(
        std::make_pair(i, k),
        {std::make_pair(k, k), std::make_pair(i, k)}
      );
      annotateNextKernel({getMatrixBlock(i, k), getMatrixBlock(k, k)}, {getMatrixBlock(i, k)}, s);
      checkCudaErrors(hipblasDtrsm(
        cublasHandle,
        HIPBLAS_SIDE_RIGHT,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_T,
        HIPBLAS_DIAG_NON_UNIT,
        B, B,
        one,
        getMatrixBlock(k, k), B,
        getMatrixBlock(i, k), B
      ));
      tiledCholeskyGraphCreator->endCaptureOperation();
    }

    for (int i = k + 1; i < T; i++) {
      // A[i][i] = SYRK(A[i][k], A[i][i])
      // A[i][i] = A[i][i] - L[i][k] * L[i][k]^T
      tiledCholeskyGraphCreator->beginCaptureOperation(
        std::make_pair(i, i),
        {std::make_pair(i, i), std::make_pair(i, k)}
      );
      annotateNextKernel({getMatrixBlock(i, i), getMatrixBlock(i, k)}, {getMatrixBlock(i, i)}, s);
      checkCudaErrors(hipblasDsyrk(
        cublasHandle,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_N,
        B, B,
        minusOne, getMatrixBlock(i, k), B,
        one, getMatrixBlock(i, i), B
      ));
      tiledCholeskyGraphCreator->endCaptureOperation();

      for (int j = i + 1; j < T; j++) {
        // A[j][i] = GEMM(A[j][k], A[i][k])
        // A[j][i] = A[j][i] - L[j][k] * L[i][k]^T
        tiledCholeskyGraphCreator->beginCaptureOperation(
          std::make_pair(j, i),
          {std::make_pair(j, i), std::make_pair(j, k), std::make_pair(i, k)}
        );
        annotateNextKernel({getMatrixBlock(j, i), getMatrixBlock(j, k), getMatrixBlock(i, k)}, {getMatrixBlock(j, i)}, s);
        checkCudaErrors(hipblasGemmEx(
          cublasHandle,
          HIPBLAS_OP_N,
          HIPBLAS_OP_T,
          B, B, B,
          minusOne,
          getMatrixBlock(j, k), HIP_R_64F, B,
          getMatrixBlock(i, k), HIP_R_64F, B,
          one,
          getMatrixBlock(j, i), HIP_R_64F, B,
          HIPBLAS_COMPUTE_64F,
          HIPBLAS_GEMM_DEFAULT
        ));
        tiledCholeskyGraphCreator->endCaptureOperation();
      }
    }
  }

  CudaEventClock clock;

  if (optimized) {
    auto optimizedGraph = profileAndOptimize(graph);

    initializeDeviceData(h_originalMatrix.get(), d_matrix);

    clock.start();
    executeOptimizedGraph(optimizedGraph);
    clock.end();

    cleanTiledCholeskyDecompositionResult(d_matrix, N, B);
    fmt::print("Result passes verification: {}\n", verifyCholeskyDecomposition(h_originalMatrix.get(), d_matrix, N));
    fmt::print("Total time used (s): {}\n", clock.getTimeInSeconds());
  } else {
    checkCudaErrors(hipGraphDebugDotPrint(graph, "./graph.dot", 0));

    hipGraphExec_t graphExec;
    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

    clock.start();
    checkCudaErrors(hipGraphLaunch(graphExec, s));
    clock.end();

    checkCudaErrors(hipDeviceSynchronize());

    cleanTiledCholeskyDecompositionResult(d_matrix, N, B);
    fmt::print("Result passes verification: {}\n", verifyCholeskyDecomposition(h_originalMatrix.get(), d_matrix, N));
    fmt::print("Total time used (s): {}\n", clock.getTimeInSeconds());
  }

  free(h_workspace);
  hipFree(d_matrix);
  hipFree(d_workspace);
}

int main(int argc, char **argv) {
  auto cmdl = argh::parser(argc, argv);

  tiledCholesky(cmdl["optimized"]);

  return 0;
}
