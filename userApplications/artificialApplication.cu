#include "hip/hip_runtime.h"
#include <cstdio>

#include "../optimization/optimization.hpp"
#include "../profiling/annotation.hpp"
#include "../profiling/memoryManager.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "../utilities/utilities.hpp"

namespace case_chainOfStreams {
template <typename T>
__global__ void initializeArraysKernel(T *a, T *b, T *c, T initA, T initB, T initC) {
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] = initA;
  b[i] = initB;
  c[i] = initC;
}

template <typename T>
__global__ void addKernel(const T *a, const T *b, T *c) {
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i] + b[i];
}

template <typename T>
__global__ void checkResultKernel(const T *c, const T expectedValue) {
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (c[i] != expectedValue) {
    printf("[checkResultKernel] found c[%d] = %f, while expectedValue = %f\n", i, c[i], expectedValue);
  }
}

void runChainOfStreams(bool optimized = true) {
  constexpr size_t CHAIN_LEN = 16;
  constexpr size_t ARRAY_SIZE = 1 << 30;  // 1GiB
  constexpr size_t ARRAY_LEN = ARRAY_SIZE / sizeof(float);
  constexpr size_t BLOCK_SIZE = 1024;
  constexpr size_t GRID_SIZE = ARRAY_LEN / BLOCK_SIZE;

  constexpr float initA = 1;
  constexpr float initB = 2;
  constexpr float initC = 0;
  constexpr float expectedC = initA + initB;

  // Allocate memory
  float *a[CHAIN_LEN], *b[CHAIN_LEN], *c[CHAIN_LEN];
  for (int i = 0; i < CHAIN_LEN; i++) {
    checkCudaErrors(wrappedCudaMallocManaged(&a[i], ARRAY_SIZE));
    checkCudaErrors(wrappedCudaMallocManaged(&b[i], ARRAY_SIZE));
    checkCudaErrors(wrappedCudaMallocManaged(&c[i], ARRAY_SIZE));
  }

  // Initialize data
  for (int i = 0; i < CHAIN_LEN; i++) {
    initializeArraysKernel<<<GRID_SIZE, BLOCK_SIZE>>>(a[i], b[i], c[i], initA, initB, initC);
  }
  checkCudaErrors(hipDeviceSynchronize());

  CudaEventClock clock;

  if (optimized) {
    hipStream_t stream;
    checkCudaErrors(hipStreamCreate(&stream));
    checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    for (int i = 0; i < CHAIN_LEN; i++) {
      annotateNextKernel({a[i], b[i]}, {c[i]}, stream);
      addKernel<<<GRID_SIZE, BLOCK_SIZE, 0, stream>>>(a[i], b[i], c[i]);
    }

    checkCudaErrors(hipGetLastError());

    hipGraph_t graph;
    checkCudaErrors(hipStreamEndCapture(stream, &graph));

    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipStreamDestroy(stream));

    auto optimizedGraph = profileAndOptimize(graph);

    // Initialize data again, because the kernels are executed during profiling
    for (int i = 0; i < CHAIN_LEN; i++) {
      initializeArraysKernel<<<GRID_SIZE, BLOCK_SIZE>>>(a[i], b[i], c[i], initA, initB, initC);
      hipMemPrefetchAsync(a[i], ARRAY_SIZE, hipCpuDeviceId);
      hipMemPrefetchAsync(b[i], ARRAY_SIZE, hipCpuDeviceId);
      hipMemPrefetchAsync(c[i], ARRAY_SIZE, hipCpuDeviceId);
    }
    checkCudaErrors(hipDeviceSynchronize());

    clock.start();
    executeOptimizedGraph(optimizedGraph);
    clock.end();

    LOG_TRACE_WITH_INFO("Verify the result");
    for (int i = 0; i < CHAIN_LEN; i++) {
      checkResultKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
        c[i],
        expectedC
      );
    }
    checkCudaErrors(hipDeviceSynchronize());
  } else {
    // Force all the data to be on CPU initially
    for (int i = 0; i < CHAIN_LEN; i++) {
      hipMemPrefetchAsync(a[i], ARRAY_SIZE, hipCpuDeviceId);
      hipMemPrefetchAsync(b[i], ARRAY_SIZE, hipCpuDeviceId);
      hipMemPrefetchAsync(c[i], ARRAY_SIZE, hipCpuDeviceId);
    }
    checkCudaErrors(hipDeviceSynchronize());

    clock.start();
    for (int i = 0; i < CHAIN_LEN; i++) {
      addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(a[i], b[i], c[i]);
    }
    clock.end();

    LOG_TRACE_WITH_INFO("Verify the result");
    for (int i = 0; i < CHAIN_LEN; i++) {
      checkResultKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
        c[i],
        expectedC
      );
    }
    checkCudaErrors(hipDeviceSynchronize());
  }

  LOG_TRACE_WITH_INFO("Total time used (s): %.2f", clock.getTimeInSeconds());

  // Clean up

  for (int i = 0; i < CHAIN_LEN; i++) {
    checkCudaErrors(hipFree(a[i]));
    checkCudaErrors(hipFree(b[i]));
    checkCudaErrors(hipFree(c[i]));
  }
}
}  // namespace case_chainOfStreams

int main() {
  initializeCudaDevice();

  case_chainOfStreams::runChainOfStreams();

  return 0;
}
