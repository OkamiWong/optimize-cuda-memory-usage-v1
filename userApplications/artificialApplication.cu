#include "hip/hip_runtime.h"
#include <hipblas.h>

#include <cstdio>
#include <memory>

#include "../profiling/annotation.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "../utilities/utilities.hpp"

void tf32GemmUsingTensorCore(hipblasHandle_t cublasHandle, int m, int n, int k, float *d_A, float *d_B, float *d_C) {
  const float alpha = 1.0f;
  const float beta = 0.0f;

  checkCudaErrors(
    hipblasGemmEx(
      cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha,
      d_B, HIP_R_32F, n,
      d_A, HIP_R_32F, k,
      &beta,
      d_C, HIP_R_32F, n,
      HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP
    )
  );
}

void case_chainOfGemms(bool useGraph = true) {
  constexpr size_t CHAIN_LEN = 2;
  constexpr size_t DIMENSION = 14 * (1 << 10);

  // Calculate matrix dimensions
  const int m = DIMENSION;
  const int k = DIMENSION;
  const int n = DIMENSION;
  const size_t A_SIZE = m * k * sizeof(float);
  const size_t B_SIZE = k * n * sizeof(float);
  const size_t C_SIZE = m * n * sizeof(float);

  // Initialzie
  hipblasHandle_t cublasHandle;
  checkCudaErrors(hipblasCreate(&cublasHandle));
  checkCudaErrors(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

  // Allocate memory
  float *a[CHAIN_LEN], *b[CHAIN_LEN], *c[CHAIN_LEN];
  for (int i = 0; i < CHAIN_LEN; i++) {
    checkCudaErrors(hipMallocManaged(&a[i], A_SIZE));
    checkCudaErrors(hipMallocManaged(&b[i], B_SIZE));
    checkCudaErrors(hipMallocManaged(&c[i], C_SIZE));
  }

  // Initialize memory
  for (int i = 0; i < CHAIN_LEN; i++) {
    fillRandomEntries(a[i], m, k, k);
    fillRandomEntries(b[i], k, n, n);
  }

  CudaEventClock clock;

  if (useGraph) {
    hipStream_t stream;
    checkCudaErrors(hipStreamCreate(&stream));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream));
    checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    for (int i = 0; i < CHAIN_LEN; i++) {
      annotateNextKernel({a[i], b[i]}, {c[i]}, stream);
      tf32GemmUsingTensorCore(cublasHandle, m, n, k, a[i], b[i], c[i]);
    }

    checkCudaErrors(hipGetLastError());

    hipGraph_t graph;
    checkCudaErrors(hipStreamEndCapture(stream, &graph));

    // Debug
    checkCudaErrors(hipGraphDebugDotPrint(graph, "/home/twang/sources/projects/optimize-cuda-memory-usage-v1/graph.dot", hipGraphDebugDotFlagsVerbose));
    size_t numRootNodes;
    checkCudaErrors(hipGraphGetRootNodes(graph, nullptr, &numRootNodes));
    LOG_TRACE_WITH_INFO("%llu", numRootNodes);
    auto rootNodes = std::make_unique<hipGraphNode_t[]>(numRootNodes);
    checkCudaErrors(hipGraphGetRootNodes(graph, rootNodes.get(), &numRootNodes));
    hipKernelNodeParams rootNodeParams;
    checkCudaErrors(hipGraphKernelNodeGetParams(rootNodes[0], &rootNodeParams));
    auto io = reinterpret_cast<KernelIO *>(rootNodeParams.kernelParams[0]);
    LOG_TRACE_WITH_INFO("%p, %p", io->outputs[0], c[0]);
    // Debug END

    hipGraphExec_t graphExec;
    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

    clock.start(stream);
    checkCudaErrors(hipGraphLaunch(graphExec, stream));
    clock.end(stream);
  } else {
    clock.start();
    for (int i = 0; i < CHAIN_LEN; i++) {
      tf32GemmUsingTensorCore(cublasHandle, m, n, k, a[i], b[i], c[i]);
    }
    clock.end();
  }

  checkCudaErrors(hipDeviceSynchronize());

  LOG_TRACE_WITH_INFO("Total time used (s): %.2f", clock.getTimeInSeconds());

  // Clean up
  for (int i = 0; i < CHAIN_LEN; i++) {
    checkCudaErrors(hipFree(a[i]));
    checkCudaErrors(hipFree(b[i]));
    checkCudaErrors(hipFree(c[i]));
  }

  checkCudaErrors(hipblasDestroy(cublasHandle));
}

int main() {
  initializeCudaDevice();

  case_chainOfGemms();

  return 0;
}
