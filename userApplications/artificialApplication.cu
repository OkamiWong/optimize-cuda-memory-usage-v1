#include "hip/hip_runtime.h"
#include <hipblas.h>

#include <cstdio>

#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "../utilities/utilities.hpp"

void tf32GemmUsingTensorCore(hipblasHandle_t handle, int m, int n, int k, float *d_A, float *d_B, float *d_C) {
  const float alpha = 1.0f;
  const float beta = 0.0f;

  checkCudaErrors(
    hipblasGemmEx(
      handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha,
      d_B, HIP_R_32F, n,
      d_A, HIP_R_32F, k,
      &beta,
      d_C, HIP_R_32F, n,
      HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP
    )
  );
}

void case_chainOfGemms() {
  constexpr size_t CHAIN_LEN = 16;
  constexpr size_t DIMENSION = 14 * (1 << 10);

  // Calculate matrix dimensions
  const int m = DIMENSION;
  const int k = DIMENSION;
  const int n = DIMENSION;
  const size_t A_SIZE = m * k * sizeof(float);
  const size_t B_SIZE = k * n * sizeof(float);
  const size_t C_SIZE = m * n * sizeof(float);

  // Initialzie
  hipblasHandle_t handle;
  checkCudaErrors(hipblasCreate(&handle));
  checkCudaErrors(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

  // Allocate memory
  float *a[CHAIN_LEN], *b[CHAIN_LEN], *c[CHAIN_LEN];
  for (int i = 0; i < CHAIN_LEN; i++) {
    checkCudaErrors(hipMallocManaged(&a[i], A_SIZE));
    checkCudaErrors(hipMallocManaged(&b[i], B_SIZE));
    checkCudaErrors(hipMallocManaged(&c[i], C_SIZE));
  }

  // Initialize memory
  for (int i = 0; i < CHAIN_LEN; i++) {
    fillRandomEntries(a[i], m, k, k);
    fillRandomEntries(b[i], k, n, n);
  }

  CudaEventClock clock;

  clock.start();

  // Compute
  for (int i = 0; i < CHAIN_LEN; i++) {
    tf32GemmUsingTensorCore(handle, m, n, k, a[i], b[i], c[i]);
  }

  clock.end();

  checkCudaErrors(hipDeviceSynchronize());

  LOG_TRACE_WITH_INFO("Total time used (s): %.2f", clock.getTimeInSeconds());

  // Clean up
  for (int i = 0; i < CHAIN_LEN; i++) {
    checkCudaErrors(hipFree(a[i]));
    checkCudaErrors(hipFree(b[i]));
    checkCudaErrors(hipFree(c[i]));
  }

  checkCudaErrors(hipblasDestroy(handle));
}

int main() {
  initializeCudaDevice();

  case_chainOfGemms();

  return 0;
}
