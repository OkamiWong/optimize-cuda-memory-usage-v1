#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <fmt/core.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdlib>
#include <initializer_list>
#include <iomanip>
#include <iostream>
#include <limits>
#include <map>
#include <memory>
#include <set>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "memopt.hpp"

using namespace memopt;

constexpr size_t N = 71680;
constexpr size_t B = N / 4;

constexpr size_t T = N / B;

__global__ void makeMatrixSymmetric(double *d_matrix, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t x = idx / n;
  size_t y = idx % n;

  if (x >= y || x >= n || y >= n) {
    return;
  }

  double average = 0.5 * (d_matrix[x * n + y] + d_matrix[y * n + x]);
  d_matrix[x * n + y] = average;
  d_matrix[y * n + x] = average;
}

__global__ void addIdenticalMatrix(double *d_matrix, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n) {
    return;
  }
  d_matrix[idx * n + idx] += n;
}

// Credit to: https://math.stackexchange.com/questions/357980/how-to-generate-random-symmetric-positive-definite-matrices-using-matlab
void generateRandomSymmetricPositiveDefiniteMatrix(double *h_A, const size_t n) {
  double *d_A;
  checkCudaErrors(hipMalloc(&d_A, n * n * sizeof(double)));

  // Generate random matrix d_A
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());
  hiprandGenerateUniformDouble(prng, d_A, n * n);

  // d_A = (d_A + d_A^T) / 2
  size_t numThreads = 1024;
  size_t numBlocks = (N * N + numThreads) / numThreads;
  makeMatrixSymmetric<<<numBlocks, numThreads>>>(d_A, N);

  // d_A = d_A + n * I
  numThreads = 1024;
  numBlocks = (N + numThreads) / numThreads;
  addIdenticalMatrix<<<numBlocks, numThreads>>>(d_A, N);

  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMemcpy(h_A, d_A, n * n * sizeof(double), hipMemcpyDefault));

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipFree(d_A));
}

void printSquareMatrix(double *h_A, const size_t n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (j != 0) std::cout << " ";
      std::cout << std::setw(6) << std::setprecision(3) << h_A[i * n + j];
    }
    std::cout << std::endl;
  }
}

// Set upper triangle entries (excluding diagonal entries) in column-major order to zero.
// Then, transpose to row-major order.
void cleanCusolverCholeskyDecompositionResult(double *L, const size_t n) {
  for (int i = 0; i < n; i++) {
    for (int j = i + 1; j < n; j++) {
      L[i + j * n] = 0;
      std::swap(L[i + j * n], L[i * n + j]);
    }
  }
}

bool verifyCholeskyDecomposition(double *A, double *L, const size_t n) {
  auto newA = std::make_unique<double[]>(n * n);
  memset(newA.get(), 0, n * n * sizeof(double));
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
        newA[i * n + j] += L[i * n + k] * L[k + j * n];
      }
    }
  }

  double error = 0;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      error += fabs(A[i * n + j] - newA[i * n + j]);
    }
  }

  fmt::print("A:\n");
  printSquareMatrix(A, n);

  fmt::print("\nnewA:\n");
  printSquareMatrix(newA.get(), n);

  fmt::print("\nL:\n");
  printSquareMatrix(L, n);
  fmt::print("\n");

  fmt::print("error = {:.6f}\n", error);

  return error <= 1e-6;
}

// Only verify the last row of L * L^T = A
bool verifyCholeskyDecompositionPartially(double *A, double *L, const size_t n) {
  auto getAEntry = [&](size_t row, size_t col) {
    return A[row * n + col];
  };

  auto getLEntry = [&](size_t row, size_t col) {
    if (row < col) {
      return static_cast<double>(0);
    }
    return L[col * n + row];
  };

  // Only check the last row;
  const size_t rowIndex = n - 1;

  const size_t rowLength = std::min((size_t)1024, n);

  auto firstRow = std::make_unique<double[]>(rowLength);
  memset(firstRow.get(), 0, rowLength * sizeof(double));
  for (int j = 0; j < rowLength; j++) {
    for (int k = 0; k < n; k++) {
      firstRow[j] += getLEntry(rowIndex, k) * getLEntry(j, k);
    }
  }

  double error = 0;
  for (int j = 0; j < rowLength; j++) {
    error += fabs(getAEntry(rowIndex, j) - firstRow[j]);
  }

  fmt::print("error = {:.6f}\n", error);

  return error <= 1e-6;
}

void trivialCholesky(bool verify) {
  // Initialize libaries
  hipsolverHandle_t cusolverDnHandle;
  checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));

  hipsolverDnParams_t cusolverDnParams;
  checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));

  // Initialize data
  double *h_A = (double *)malloc(N * N * sizeof(double));
  generateRandomSymmetricPositiveDefiniteMatrix(h_A, N);

  double *d_A;
  checkCudaErrors(hipMalloc(&d_A, N * N * sizeof(double)));
  checkCudaErrors(hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice));

  size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;

  checkCudaErrors(hipsolverDnXpotrf_bufferSize(
    cusolverDnHandle,
    cusolverDnParams,
    HIPBLAS_FILL_MODE_LOWER,
    N,
    HIP_R_64F,
    d_A,
    N,
    HIP_R_64F,
    &workspaceInBytesOnDevice,
    &workspaceInBytesOnHost
  ));

  void *h_workspace = malloc(workspaceInBytesOnHost);

  void *d_workspace;
  checkCudaErrors(hipMalloc(&d_workspace, workspaceInBytesOnDevice));

  int *d_info;
  checkCudaErrors(hipMalloc(&d_info, sizeof(int)));

  CudaEventClock clock;

  clock.start();

  // Calculate
  checkCudaErrors(hipsolverDnXpotrf(
    cusolverDnHandle,
    cusolverDnParams,
    HIPBLAS_FILL_MODE_LOWER,
    N,
    HIP_R_64F,
    d_A,
    N,
    HIP_R_64F,
    d_workspace,
    workspaceInBytesOnDevice,
    h_workspace,
    workspaceInBytesOnHost,
    d_info
  ));

  clock.end();

  // Check
  int h_info = 0;
  checkCudaErrors(hipMemcpy(&h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));
  if (h_info != 0) {
    std::cout << "Unsuccessful potrf execution\n\n"
              << "d_info = " << h_info << "\n\n";
  }

  // Verify
  if (verify) {
    double *h_L = (double *)malloc(N * N * sizeof(double));
    checkCudaErrors(hipMemcpy(h_L, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost));
    cleanCusolverCholeskyDecompositionResult(h_L, N);
    fmt::print("Result passes verification: {}\n", verifyCholeskyDecompositionPartially(h_A, h_L, N));
    free(h_L);
  }

  fmt::print("Total time used (s): {}\n", clock.getTimeInSeconds());

  // Clean
  free(h_A);
  free(h_workspace);
  checkCudaErrors(hipsolverDnDestroy(cusolverDnHandle));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_workspace));
  checkCudaErrors(hipFree(d_info));
}

typedef std::pair<int, int> MatrixTile;

class TiledCholeskyGraphCreator {
 public:
  TiledCholeskyGraphCreator(hipStream_t stream, hipGraph_t graph) : stream(stream), graph(graph) {
    this->lastModifiedTile = std::make_pair(-1, -1);
  }
  void beginCaptureOperation(MatrixTile tileToWrite, std::initializer_list<MatrixTile> tilesToRead) {
    auto tiles = std::vector<MatrixTile>(tilesToRead);
    tiles.push_back(tileToWrite);
    auto dependencies = this->getDependencies(tiles);

    this->lastModifiedTile = tileToWrite;
    this->lastDependencies = dependencies;

    checkCudaErrors(hipStreamBeginCaptureToGraph(this->stream, this->graph, dependencies.data(), nullptr, dependencies.size(), hipStreamCaptureModeGlobal));
  }

  void endCaptureOperation() {
    assert(this->lastModifiedTile.first != -1 && this->lastModifiedTile.second != -1);
    checkCudaErrors(hipStreamEndCapture(this->stream, &this->graph));
    this->tileLastModifiedByMap[this->lastModifiedTile] = this->getTailOfLastCapturedNodeChain();
    this->lastModifiedTile = std::make_pair(-1, -1);
  };

 private:
  std::map<MatrixTile, hipGraphNode_t> tileLastModifiedByMap;
  std::map<hipGraphNode_t, bool> visited;
  hipStream_t stream;
  hipGraph_t graph;
  MatrixTile lastModifiedTile;
  std::vector<hipGraphNode_t> lastDependencies;

  std::vector<hipGraphNode_t> getDependencies(std::vector<MatrixTile> tiles) {
    std::vector<hipGraphNode_t> dependencies;
    for (auto tile : tiles) {
      auto it = this->tileLastModifiedByMap.find(tile);
      if (it != this->tileLastModifiedByMap.end()) {
        dependencies.push_back(it->second);
      }
    }

    auto dedupedEnd = std::unique(dependencies.begin(), dependencies.end());
    dependencies.resize(std::distance(dependencies.begin(), dedupedEnd));
    return dependencies;
  }

  hipGraphNode_t getTailOfLastCapturedNodeChain() {
    if (lastDependencies.size() == 0) {
      size_t numEdges;
      checkCudaErrors(hipGraphGetEdges(this->graph, nullptr, nullptr, &numEdges));
      auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
      auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
      checkCudaErrors(hipGraphGetEdges(this->graph, from.get(), to.get(), &numEdges));

      std::map<hipGraphNode_t, bool> hasOutGoingEdge;
      std::set<hipGraphNode_t> noOutGoingEdgeNodes;
      for (int i = 0; i < numEdges; i++) {
        hasOutGoingEdge[from[i]] = true;
        noOutGoingEdgeNodes.erase(from[i]);
        if (!hasOutGoingEdge[to[i]])
          noOutGoingEdgeNodes.insert(to[i]);
      }

      assert(noOutGoingEdgeNodes.size() == 1);

      return *noOutGoingEdgeNodes.begin();
    } else {
      auto nodeBeforeChain = lastDependencies[0];
      size_t numDependentNodes;
      checkCudaErrors(hipGraphNodeGetDependentNodes(nodeBeforeChain, nullptr, &numDependentNodes));

      assert(numDependentNodes > 0);

      auto dependentNodes = std::make_unique<hipGraphNode_t[]>(numDependentNodes);
      checkCudaErrors(hipGraphNodeGetDependentNodes(nodeBeforeChain, dependentNodes.get(), &numDependentNodes));

      hipGraphNode_t chainBeginningNode;
      for (int i = 0; i < numDependentNodes; i++) {
        if (!visited[dependentNodes[i]]) {
          chainBeginningNode = dependentNodes[i];
          break;
        }
      }

      auto u = chainBeginningNode;
      while (true) {
        visited[u] = true;
        checkCudaErrors(hipGraphNodeGetDependentNodes(u, nullptr, &numDependentNodes));
        if (numDependentNodes == 0) break;

        assert(numDependentNodes == 1);

        hipGraphNode_t v;
        checkCudaErrors(hipGraphNodeGetDependentNodes(u, &v, &numDependentNodes));
        u = v;
      }

      return u;
    }
  }
};

void tiledCholesky(bool verify) {
  SystemWallClock clock;
  clock.start();

  // Initialize data
  clock.logWithCurrentTime("Initialize host data");
  auto originalMatrix = std::make_unique<double[]>(N * N);  // Column-major
  generateRandomSymmetricPositiveDefiniteMatrix(originalMatrix.get(), N);
  clock.logWithCurrentTime("Host data initialized");

  // Copy to device
  clock.logWithCurrentTime("Initialize device data");
  double *d_matrix;
  checkCudaErrors(hipMallocManaged(&d_matrix, N * N * sizeof(double)));
  checkCudaErrors(hipMemcpy(d_matrix, originalMatrix.get(), N * N * sizeof(double), hipMemcpyHostToDevice));
  clock.logWithCurrentTime("Device data initialized");

  auto getMatrixBlock = [&](int i, int j) {
    return d_matrix + i * B + j * B * N;
  };

  // Initialize libraries
  hipsolverHandle_t cusolverDnHandle;
  hipsolverDnParams_t cusolverDnParams;
  hipblasHandle_t cublasHandle;
  checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));
  checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));
  checkCudaErrors(hipblasCreate(&cublasHandle));

  // Prepare constants
  double *one, *minusOne;
  checkCudaErrors(hipMallocManaged(&one, sizeof(double)));
  checkCudaErrors(hipMallocManaged(&minusOne, sizeof(double)));
  *one = 1.0;
  *minusOne = -1.0;

  // Prepare buffer for potrf
  size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
  checkCudaErrors(hipsolverDnXpotrf_bufferSize(
    cusolverDnHandle,
    cusolverDnParams,
    HIPBLAS_FILL_MODE_LOWER,
    B,
    HIP_R_64F,
    d_matrix,
    N,
    HIP_R_64F,
    &workspaceInBytesOnDevice,
    &workspaceInBytesOnHost
  ));
  void *h_workspace, *d_workspace;
  int *d_info;
  checkCudaErrors(hipMallocManaged(&h_workspace, workspaceInBytesOnHost));
  checkCudaErrors(hipMallocManaged(&d_workspace, workspaceInBytesOnDevice));
  checkCudaErrors(hipMallocManaged(&d_info, sizeof(int)));

  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t s;
  checkCudaErrors(hipStreamCreate(&s));

  checkCudaErrors(hipsolverSetStream(cusolverDnHandle, s));
  checkCudaErrors(hipblasSetStream(cublasHandle, s));

  clock.logWithCurrentTime("Start to record graph");

  auto tiledCholeskyGraphCreator = std::make_unique<TiledCholeskyGraphCreator>(s, graph);

  for (int k = 0; k < T; k++) {
    // A[k][k] = POTRF(A[k][k])
    // L[k][k] = POTRF(A[k][k])
    tiledCholeskyGraphCreator->beginCaptureOperation(
      std::make_pair(k, k),
      {std::make_pair(k, k)}
    );
    checkCudaErrors(hipsolverDnXpotrf(
      cusolverDnHandle,
      cusolverDnParams,
      HIPBLAS_FILL_MODE_LOWER,
      B,
      HIP_R_64F,
      getMatrixBlock(k, k),
      N,
      HIP_R_64F,
      d_workspace,
      workspaceInBytesOnDevice,
      h_workspace,
      workspaceInBytesOnHost,
      d_info
    ));
    tiledCholeskyGraphCreator->endCaptureOperation();

    for (int i = k + 1; i < T; i++) {
      // A[i][k] = TRSM(A[k][k], A[i][k])
      // L[i][k] * L[k][k]^T = A[i][k]
      tiledCholeskyGraphCreator->beginCaptureOperation(
        std::make_pair(i, k),
        {std::make_pair(k, k), std::make_pair(i, k)}
      );
      checkCudaErrors(hipblasDtrsm(
        cublasHandle,
        HIPBLAS_SIDE_RIGHT,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_T,
        HIPBLAS_DIAG_NON_UNIT,
        B, B,
        one,
        getMatrixBlock(k, k), N,
        getMatrixBlock(i, k), N
      ));
      tiledCholeskyGraphCreator->endCaptureOperation();
    }

    for (int i = k + 1; i < T; i++) {
      // A[i][i] = SYRK(A[i][k], A[i][i])
      // A[i][i] = A[i][i] - L[i][k] * L[i][k]^T
      tiledCholeskyGraphCreator->beginCaptureOperation(
        std::make_pair(i, i),
        {std::make_pair(i, i), std::make_pair(i, k)}
      );
      checkCudaErrors(hipblasDsyrk(
        cublasHandle,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_N,
        B, B,
        minusOne, getMatrixBlock(i, k), N,
        one, getMatrixBlock(i, i), N
      ));
      tiledCholeskyGraphCreator->endCaptureOperation();

      for (int j = i + 1; j < T; j++) {
        // A[j][i] = GEMM(A[j][k], A[i][k])
        // A[j][i] = A[j][i] - L[j][k] * L[i][k]^T
        tiledCholeskyGraphCreator->beginCaptureOperation(
          std::make_pair(j, i),
          {std::make_pair(j, i), std::make_pair(j, k), std::make_pair(i, k)}
        );
        checkCudaErrors(hipblasGemmEx(
          cublasHandle,
          HIPBLAS_OP_N,
          HIPBLAS_OP_T,
          B, B, B,
          minusOne,
          getMatrixBlock(j, k), HIP_R_64F, N,
          getMatrixBlock(i, k), HIP_R_64F, N,
          one,
          getMatrixBlock(j, i), HIP_R_64F, N,
          HIPBLAS_COMPUTE_64F,
          HIPBLAS_GEMM_DEFAULT
        ));
        tiledCholeskyGraphCreator->endCaptureOperation();
      }
    }
  }

  clock.logWithCurrentTime("Graph recorded");

  checkCudaErrors(hipGraphDebugDotPrint(graph, "./graph.dot", 0));

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  checkCudaErrors(hipProfilerStart());

  CudaEventClock cudaEventClock;

  clock.logWithCurrentTime("Launch graph");
  cudaEventClock.start();
  checkCudaErrors(hipGraphLaunch(graphExec, s));
  cudaEventClock.end();
  clock.logWithCurrentTime("Graph launched");

  checkCudaErrors(hipDeviceSynchronize());
  clock.logWithCurrentTime("Synchronization done");

  checkCudaErrors(hipProfilerStop());

  if (verify) {
    clock.logWithCurrentTime("Start to verify");
    fmt::print("Result passes partial verification: {}\n", verifyCholeskyDecompositionPartially(originalMatrix.get(), d_matrix, N));
    clock.logWithCurrentTime("Verification done");

    // cleanCusolverCholeskyDecompositionResult(d_matrix, N);
    // fmt::print("Result passes verification: {}\n", verifyCholeskyDecomposition(originalMatrix.get(), d_matrix, N));
  }

  fmt::print("Total time used (s): {}\n", cudaEventClock.getTimeInSeconds());

  free(h_workspace);
  hipFree(d_matrix);
  hipFree(d_workspace);
}

void cholesky(bool tiled, bool verify) {
  if (tiled) {
    tiledCholesky(verify);
  } else {
    trivialCholesky(verify);
  }
}

int main(int argc, char **argv) {
  auto cmdl = argh::parser(argc, argv);

  cholesky(cmdl["tiled"], cmdl["verify"]);

  return 0;
}
