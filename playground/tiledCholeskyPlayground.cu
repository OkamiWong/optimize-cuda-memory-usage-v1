#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>
#include <fmt/core.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdlib>
#include <initializer_list>
#include <iomanip>
#include <iostream>
#include <limits>
#include <map>
#include <memory>
#include <set>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/utilities.hpp"

using namespace memopt;

constexpr size_t N = 1024 * 20;
constexpr size_t B = N / 4;

constexpr size_t T = N / B;

// Credit to: https://math.stackexchange.com/questions/357980/how-to-generate-random-symmetric-positive-definite-matrices-using-matlab
void generateRandomSymmetricPositiveDefiniteMatrix(double *h_A, const size_t n) {
  srand(time(NULL));

  double *h_A_temp = (double *)malloc(n * n * sizeof(double));

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      h_A_temp[i * n + j] = (float)rand() / (float)RAND_MAX;

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      h_A[i * n + j] = 0.5 * (h_A_temp[i * n + j] + h_A_temp[j * n + i]);

  for (int i = 0; i < n; i++) h_A[i * n + i] = h_A[i * n + i] + n;
}

void printSquareMatrix(double *h_A, const size_t n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (j != 0) std::cout << " ";
      std::cout << std::setw(6) << std::setprecision(3) << h_A[i * n + j];
    }
    std::cout << std::endl;
  }
}

// Set upper triangle entries (excluding diagonal entries) in column-major order to zero.
// Then, transpose to row-major order.
void cleanCusolverCholeskyDecompositionResult(double *L, const int n) {
  for (int i = 0; i < n; i++) {
    for (int j = i + 1; j < n; j++) {
      L[i + j * n] = 0;
      std::swap(L[i + j * n], L[i * n + j]);
    }
  }
}

bool verifyCholeskyDecomposition(double *A, double *L, const int n) {
  auto newA = std::make_unique<double[]>(n * n);
  memset(newA.get(), 0, n * n * sizeof(double));
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
        newA[i * n + j] += L[i * n + k] * L[k + j * n];
      }
    }
  }

  double error = 0;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      error += fabs(A[i * n + j] - newA[i * n + j]);
    }
  }

  fmt::print("A:\n");
  printSquareMatrix(A, n);

  fmt::print("\nnewA:\n");
  printSquareMatrix(newA.get(), n);

  fmt::print("\nL:\n");
  printSquareMatrix(L, n);
  fmt::print("\n");

  fmt::print("error = {:.6f}\n", error);

  return error <= 1e-6;
}

// Only verify the last row of L * L^T = A
bool verifyCholeskyDecompositionPartially(double *A, double *L, const int n) {
  auto getAEntry = [&](int row, int col) {
    return A[row * n + col];
  };

  auto getLEntry = [&](int row, int col) {
    if (row < col) {
      return static_cast<double>(0);
    }
    return L[col * n + row];
  };

  // Only check the last row;
  const int rowIndex = n - 1;

  const int rowLength = n;

  auto firstRow = std::make_unique<double[]>(rowLength);
  memset(firstRow.get(), 0, rowLength * sizeof(double));
  for (int j = 0; j < rowLength; j++) {
    for (int k = 0; k < n; k++) {
      firstRow[j] += getLEntry(rowIndex, k) * getLEntry(j, k);
    }
  }

  double error = 0;
  for (int j = 0; j < rowLength; j++) {
    error += fabs(getAEntry(rowIndex, j) - firstRow[j]);
  }

  fmt::print("error = {:.6f}\n", error);

  return error <= 1e-6;
}

void trivialCholesky(bool verify) {
  // Initialize libaries
  hipsolverHandle_t cusolverDnHandle;
  checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));

  hipsolverDnParams_t cusolverDnParams;
  checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));

  // Initialize data
  double *h_A = (double *)malloc(N * N * sizeof(double));
  generateRandomSymmetricPositiveDefiniteMatrix(h_A, N);

  double *d_A;
  checkCudaErrors(hipMalloc(&d_A, N * N * sizeof(double)));
  checkCudaErrors(hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice));

  size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;

  checkCudaErrors(hipsolverDnXpotrf_bufferSize(
    cusolverDnHandle,
    cusolverDnParams,
    HIPBLAS_FILL_MODE_LOWER,
    N,
    HIP_R_64F,
    d_A,
    N,
    HIP_R_64F,
    &workspaceInBytesOnDevice,
    &workspaceInBytesOnHost
  ));

  void *h_workspace = malloc(workspaceInBytesOnHost);

  void *d_workspace;
  checkCudaErrors(hipMalloc(&d_workspace, workspaceInBytesOnDevice));

  int *d_info;
  checkCudaErrors(hipMalloc(&d_info, sizeof(int)));

  CudaEventClock clock;

  clock.start();

  // Calculate
  checkCudaErrors(hipsolverDnXpotrf(
    cusolverDnHandle,
    cusolverDnParams,
    HIPBLAS_FILL_MODE_LOWER,
    N,
    HIP_R_64F,
    d_A,
    N,
    HIP_R_64F,
    d_workspace,
    workspaceInBytesOnDevice,
    h_workspace,
    workspaceInBytesOnHost,
    d_info
  ));

  clock.end();

  // Check
  int h_info = 0;
  checkCudaErrors(hipMemcpy(&h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));
  if (h_info != 0) {
    std::cout << "Unsuccessful potrf execution\n\n"
              << "d_info = " << h_info << "\n\n";
  }

  // Verify
  if (verify) {
    double *h_L = (double *)malloc(N * N * sizeof(double));
    checkCudaErrors(hipMemcpy(h_L, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost));
    cleanCusolverCholeskyDecompositionResult(h_L, N);
    fmt::print("Result passes verification: {}\n", verifyCholeskyDecomposition(h_A, h_L, N));
    free(h_L);
  }

  fmt::print("Total time used (s): {}\n", clock.getTimeInSeconds());

  // Clean
  free(h_A);
  free(h_workspace);
  checkCudaErrors(hipsolverDnDestroy(cusolverDnHandle));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_workspace));
  checkCudaErrors(hipFree(d_info));
}

typedef std::pair<int, int> MatrixTile;

class TiledCholeskyGraphCreator {
 public:
  TiledCholeskyGraphCreator(hipStream_t stream, hipGraph_t graph) : stream(stream), graph(graph) {
    this->lastModifiedTile = std::make_pair(-1, -1);
  }
  void beginCaptureOperation(MatrixTile tileToWrite, std::initializer_list<MatrixTile> tilesToRead) {
    auto tiles = std::vector<MatrixTile>(tilesToRead);
    tiles.push_back(tileToWrite);
    auto dependencies = this->getDependencies(tiles);

    this->lastModifiedTile = tileToWrite;
    this->lastDependencies = dependencies;

    checkCudaErrors(hipStreamBeginCaptureToGraph(this->stream, this->graph, dependencies.data(), nullptr, dependencies.size(), hipStreamCaptureModeGlobal));
  }

  void endCaptureOperation() {
    assert(this->lastModifiedTile.first != -1 && this->lastModifiedTile.second != -1);
    checkCudaErrors(hipStreamEndCapture(this->stream, &this->graph));
    this->tileLastModifiedByMap[this->lastModifiedTile] = this->getTailOfLastCapturedNodeChain();
    this->lastModifiedTile = std::make_pair(-1, -1);
  };

 private:
  std::map<MatrixTile, hipGraphNode_t> tileLastModifiedByMap;
  std::map<hipGraphNode_t, bool> visited;
  hipStream_t stream;
  hipGraph_t graph;
  MatrixTile lastModifiedTile;
  std::vector<hipGraphNode_t> lastDependencies;

  std::vector<hipGraphNode_t> getDependencies(std::vector<MatrixTile> tiles) {
    std::vector<hipGraphNode_t> dependencies;
    for (auto tile : tiles) {
      auto it = this->tileLastModifiedByMap.find(tile);
      if (it != this->tileLastModifiedByMap.end()) {
        dependencies.push_back(it->second);
      }
    }

    auto dedupedEnd = std::unique(dependencies.begin(), dependencies.end());
    dependencies.resize(std::distance(dependencies.begin(), dedupedEnd));
    return dependencies;
  }

  hipGraphNode_t getTailOfLastCapturedNodeChain() {
    if (lastDependencies.size() == 0) {
      size_t numEdges;
      checkCudaErrors(hipGraphGetEdges(this->graph, nullptr, nullptr, &numEdges));
      auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
      auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
      checkCudaErrors(hipGraphGetEdges(this->graph, from.get(), to.get(), &numEdges));

      std::map<hipGraphNode_t, bool> hasOutGoingEdge;
      std::set<hipGraphNode_t> noOutGoingEdgeNodes;
      for (int i = 0; i < numEdges; i++) {
        hasOutGoingEdge[from[i]] = true;
        noOutGoingEdgeNodes.erase(from[i]);
        if (!hasOutGoingEdge[to[i]])
          noOutGoingEdgeNodes.insert(to[i]);
      }

      assert(noOutGoingEdgeNodes.size() == 1);

      return *noOutGoingEdgeNodes.begin();
    } else {
      auto nodeBeforeChain = lastDependencies[0];
      size_t numDependentNodes;
      checkCudaErrors(hipGraphNodeGetDependentNodes(nodeBeforeChain, nullptr, &numDependentNodes));

      assert(numDependentNodes > 0);

      auto dependentNodes = std::make_unique<hipGraphNode_t[]>(numDependentNodes);
      checkCudaErrors(hipGraphNodeGetDependentNodes(nodeBeforeChain, dependentNodes.get(), &numDependentNodes));

      hipGraphNode_t chainBeginningNode;
      for (int i = 0; i < numDependentNodes; i++) {
        if (!visited[dependentNodes[i]]) {
          chainBeginningNode = dependentNodes[i];
          break;
        }
      }

      auto u = chainBeginningNode;
      while (true) {
        visited[u] = true;
        checkCudaErrors(hipGraphNodeGetDependentNodes(u, nullptr, &numDependentNodes));
        if (numDependentNodes == 0) break;

        assert(numDependentNodes == 1);

        hipGraphNode_t v;
        checkCudaErrors(hipGraphNodeGetDependentNodes(u, &v, &numDependentNodes));
        u = v;
      }

      return u;
    }
  }
};

void tiledCholesky(bool verify) {
  SystemWallClock clock;
  clock.start();

  // Initialize data
  clock.logWithCurrentTime("Initialize host data");
  auto originalMatrix = std::make_unique<double[]>(N * N);  // Column-major
  generateRandomSymmetricPositiveDefiniteMatrix(originalMatrix.get(), N);
  clock.logWithCurrentTime("Host data initialized");

  // Copy to device
  clock.logWithCurrentTime("Initialize device data");
  double *d_matrix;
  checkCudaErrors(hipMallocManaged(&d_matrix, N * N * sizeof(double)));
  checkCudaErrors(hipMemcpy(d_matrix, originalMatrix.get(), N * N * sizeof(double), hipMemcpyHostToDevice));
  clock.logWithCurrentTime("Device data initialized");

  auto getMatrixBlock = [&](int i, int j) {
    return d_matrix + i * B + j * B * N;
  };

  // Initialize libraries
  hipsolverHandle_t cusolverDnHandle;
  hipsolverDnParams_t cusolverDnParams;
  hipblasHandle_t cublasHandle;
  checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));
  checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));
  checkCudaErrors(hipblasCreate(&cublasHandle));

  // Prepare constants
  double *one, *minusOne;
  checkCudaErrors(hipMallocManaged(&one, sizeof(double)));
  checkCudaErrors(hipMallocManaged(&minusOne, sizeof(double)));
  *one = 1.0;
  *minusOne = -1.0;

  // Prepare buffer for potrf
  size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
  checkCudaErrors(hipsolverDnXpotrf_bufferSize(
    cusolverDnHandle,
    cusolverDnParams,
    HIPBLAS_FILL_MODE_LOWER,
    B,
    HIP_R_64F,
    d_matrix,
    N,
    HIP_R_64F,
    &workspaceInBytesOnDevice,
    &workspaceInBytesOnHost
  ));
  void *h_workspace, *d_workspace;
  int *d_info;
  checkCudaErrors(hipMallocManaged(&h_workspace, workspaceInBytesOnHost));
  checkCudaErrors(hipMallocManaged(&d_workspace, workspaceInBytesOnDevice));
  checkCudaErrors(hipMallocManaged(&d_info, sizeof(int)));

  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t s;
  checkCudaErrors(hipStreamCreate(&s));

  checkCudaErrors(hipsolverSetStream(cusolverDnHandle, s));
  checkCudaErrors(hipblasSetStream(cublasHandle, s));

  clock.logWithCurrentTime("Start to record graph");

  auto tiledCholeskyGraphCreator = std::make_unique<TiledCholeskyGraphCreator>(s, graph);

  for (int k = 0; k < T; k++) {
    // A[k][k] = POTRF(A[k][k])
    // L[k][k] = POTRF(A[k][k])
    tiledCholeskyGraphCreator->beginCaptureOperation(
      std::make_pair(k, k),
      {std::make_pair(k, k)}
    );
    checkCudaErrors(hipsolverDnXpotrf(
      cusolverDnHandle,
      cusolverDnParams,
      HIPBLAS_FILL_MODE_LOWER,
      B,
      HIP_R_64F,
      getMatrixBlock(k, k),
      N,
      HIP_R_64F,
      d_workspace,
      workspaceInBytesOnDevice,
      h_workspace,
      workspaceInBytesOnHost,
      d_info
    ));
    tiledCholeskyGraphCreator->endCaptureOperation();

    for (int i = k + 1; i < T; i++) {
      // A[i][k] = TRSM(A[k][k], A[i][k])
      // L[i][k] * L[k][k]^T = A[i][k]
      tiledCholeskyGraphCreator->beginCaptureOperation(
        std::make_pair(i, k),
        {std::make_pair(k, k), std::make_pair(i, k)}
      );
      checkCudaErrors(hipblasDtrsm(
        cublasHandle,
        HIPBLAS_SIDE_RIGHT,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_T,
        HIPBLAS_DIAG_NON_UNIT,
        B, B,
        one,
        getMatrixBlock(k, k), N,
        getMatrixBlock(i, k), N
      ));
      tiledCholeskyGraphCreator->endCaptureOperation();
    }

    for (int i = k + 1; i < T; i++) {
      // A[i][i] = SYRK(A[i][k], A[i][i])
      // A[i][i] = A[i][i] - L[i][k] * L[i][k]^T
      tiledCholeskyGraphCreator->beginCaptureOperation(
        std::make_pair(i, i),
        {std::make_pair(i, i), std::make_pair(i, k)}
      );
      checkCudaErrors(hipblasDsyrk(
        cublasHandle,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_N,
        B, B,
        minusOne, getMatrixBlock(i, k), N,
        one, getMatrixBlock(i, i), N
      ));
      tiledCholeskyGraphCreator->endCaptureOperation();

      for (int j = i + 1; j < T; j++) {
        // A[j][i] = GEMM(A[j][k], A[i][k])
        // A[j][i] = A[j][i] - L[j][k] * L[i][k]^T
        tiledCholeskyGraphCreator->beginCaptureOperation(
          std::make_pair(j, i),
          {std::make_pair(j, i), std::make_pair(j, k), std::make_pair(i, k)}
        );
        checkCudaErrors(hipblasGemmEx(
          cublasHandle,
          HIPBLAS_OP_N,
          HIPBLAS_OP_T,
          B, B, B,
          minusOne,
          getMatrixBlock(j, k), HIP_R_64F, N,
          getMatrixBlock(i, k), HIP_R_64F, N,
          one,
          getMatrixBlock(j, i), HIP_R_64F, N,
          HIPBLAS_COMPUTE_64F,
          HIPBLAS_GEMM_DEFAULT
        ));
        tiledCholeskyGraphCreator->endCaptureOperation();
      }
    }
  }

  clock.logWithCurrentTime("Graph recorded");

  checkCudaErrors(hipGraphDebugDotPrint(graph, "./graph.dot", 0));

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  checkCudaErrors(hipProfilerStart());

  CudaEventClock cudaEventClock;

  clock.logWithCurrentTime("Launch graph");
  cudaEventClock.start();
  checkCudaErrors(hipGraphLaunch(graphExec, s));
  cudaEventClock.end();
  clock.logWithCurrentTime("Graph launched");

  checkCudaErrors(hipDeviceSynchronize());
  clock.logWithCurrentTime("Synchronization done");

  checkCudaErrors(hipProfilerStop());

  if (verify) {
    clock.logWithCurrentTime("Start to verify");
    fmt::print("Result passes partial verification: {}\n", verifyCholeskyDecompositionPartially(originalMatrix.get(), d_matrix, N));
    clock.logWithCurrentTime("Verification done");

    // cleanCusolverCholeskyDecompositionResult(d_matrix, N);
    // fmt::print("Result passes verification: {}\n", verifyCholeskyDecomposition(originalMatrix.get(), d_matrix, N));
  }

  fmt::print("Total time used (s): {}\n", cudaEventClock.getTimeInSeconds());

  free(h_workspace);
  hipFree(d_matrix);
  hipFree(d_workspace);
}

void cholesky(bool tiled, bool verify) {
  if (tiled) {
    tiledCholesky(verify);
  } else {
    trivialCholesky(verify);
  }
}

int main(int argc, char **argv) {
  auto cmdl = argh::parser(argc, argv);

  cholesky(cmdl["tiled"], cmdl["verify"]);

  return 0;
}
