#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>
#include <fmt/core.h>

#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <limits>
#include <memory>

#include "../utilities/cudaUtilities.hpp"

const int N = 8;

// Credit to: https://math.stackexchange.com/questions/357980/how-to-generate-random-symmetric-positive-definite-matrices-using-matlab
void generateRandomSymmetricPositiveDefiniteMatrix(double *h_A, const int n) {
  // --- Initialize random seed
  srand(time(NULL));

  double *h_A_temp = (double *)malloc(n * n * sizeof(double));

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      h_A_temp[i * n + j] = (float)rand() / (float)RAND_MAX;

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      h_A[i * n + j] = 0.5 * (h_A_temp[i * n + j] + h_A_temp[j * n + i]);

  for (int i = 0; i < n; i++) h_A[i * n + i] = h_A[i * n + i] + n;
}

void printSquareMatrix(double *h_A, const int N) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      if (j != 0) std::cout << " ";
      std::cout << std::setw(6) << std::setprecision(3) << h_A[i * N + j];
    }
    std::cout << std::endl;
  }
}

void cleanCusolverCholeskyDecompositionResult(double *L, const int n) {
  for (int i = 0; i < n; i++) {
    for (int j = i + 1; j < n; j++) {
      L[i + j * n] = 0;
      std::swap(L[i + j * n], L[i * n + j]);
    }
  }
}

bool verifyCholeskyDecomposition(double *A, double *L, const int n) {
  auto newA = std::make_unique<double[]>(n * n);
  memset(newA.get(), 0, n * n * sizeof(double));
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
        newA[i * n + j] += L[i * n + k] * L[k + j * n];
      }
    }
  }

  double error = 0;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      error += fabs(A[i * n + j] - newA[i * n + j]);
    }
  }

  // fmt::print("A:\n");
  // printSquareMatrix(A, n);

  // fmt::print("\nnewA:\n");
  // printSquareMatrix(newA.get(), n);

  // fmt::print("\nL:\n");
  // printSquareMatrix(L, n);
  // fmt::print("\n");

  fmt::print("error = {:.6f}\n", error);

  return error <= 1e-6;
}

int main() {
  hipsolverHandle_t solver_handle;
  checkCudaErrors(hipsolverDnCreate(&solver_handle));

  hipblasHandle_t cublas_handle;
  checkCudaErrors(hipblasCreate(&cublas_handle));

  // Init
  double *h_A = (double *)malloc(N * N * sizeof(double));
  generateRandomSymmetricPositiveDefiniteMatrix(h_A, N);

  double *d_A;
  checkCudaErrors(hipMalloc(&d_A, N * N * sizeof(double)));
  checkCudaErrors(hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice));

  int work_size = 0;
  checkCudaErrors(hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_LOWER, N, d_A, N, &work_size));

  double *work;
  checkCudaErrors(hipMalloc(&work, work_size * sizeof(double)));

  int *devInfo;
  checkCudaErrors(hipMalloc(&devInfo, sizeof(int)));

  // Calculate
  checkCudaErrors(hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_LOWER, N, d_A, N, work, work_size, devInfo));

  // Check
  int devInfo_h = 0;
  checkCudaErrors(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
  if (devInfo_h != 0) {
    std::cout << "Unsuccessful potrf execution\n\n"
              << "devInfo = " << devInfo_h << "\n\n";
  }

  // Verify
  double *h_L = (double *)malloc(N * N * sizeof(double));
  checkCudaErrors(hipMemcpy(h_L, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost));
  cleanCusolverCholeskyDecompositionResult(h_L, N);
  fmt::print("Result passes verification: {}\n", verifyCholeskyDecomposition(h_A, h_L, N));

  // Clean
  checkCudaErrors(hipsolverDnDestroy(solver_handle));

  return 0;
}
