#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>

#include <cstdlib>
#include <iomanip>
#include <iostream>

#include "../utilities/cudaUtilities.hpp"

const int N = 8;

// Credit to: https://math.stackexchange.com/questions/357980/how-to-generate-random-symmetric-positive-definite-matrices-using-matlab
void generateRandomSymmetricPositiveDefiniteMatrix(double *h_A, const int N) {
  // --- Initialize random seed
  srand(time(NULL));

  double *h_A_temp = (double *)malloc(N * N * sizeof(double));

  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++)
      h_A_temp[i * N + j] = (float)rand() / (float)RAND_MAX;

  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++)
      h_A[i * N + j] = 0.5 * (h_A_temp[i * N + j] + h_A_temp[j * N + i]);

  for (int i = 0; i < N; i++) h_A[i * N + i] = h_A[i * N + i] + N;
}

void printSquareMatrix(double *h_A, const int N) {
  auto originalWidth = std::cout.width();
  std::cout << std::setw(6);

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      if (j != 0) std::cout << " ";
      std::cout << h_A[i * N + j];
    }
    std::cout << std::endl;
  }

  std::cout << std::setw(originalWidth);
}

int main() {
  hipsolverHandle_t solver_handle;
  checkCudaErrors(hipsolverDnCreate(&solver_handle));

  hipblasHandle_t cublas_handle;
  checkCudaErrors(hipblasCreate(&cublas_handle));

  double *h_A = (double *)malloc(N * N * sizeof(double));
  generateRandomSymmetricPositiveDefiniteMatrix(h_A, N);

  double *d_A;
  checkCudaErrors(hipMalloc(&d_A, N * N * sizeof(double)));
  checkCudaErrors(hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice));

  // cuSOLVE input/output parameters/arrays
  int work_size = 0;
  int *devInfo;
  checkCudaErrors(hipMalloc(&devInfo, sizeof(int)));

  // CUDA CHOLESKY initialization
  checkCudaErrors(hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_LOWER, N, d_A, N, &work_size));

  // CUDA POTRF execution
  double *work;
  checkCudaErrors(hipMalloc(&work, work_size * sizeof(double)));
  checkCudaErrors(hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_LOWER, N, d_A, N, work, work_size, devInfo));
  int devInfo_h = 0;
  checkCudaErrors(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
  if (devInfo_h != 0) {
    std::cout << "Unsuccessful potrf execution\n\n"
              << "devInfo = " << devInfo_h << "\n\n";
  }

  // At this point, the lower triangular part of A contains the elements of L.
  checkCudaErrors(hipMemcpy(h_A, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost));
  printSquareMatrix(h_A, N);

  checkCudaErrors(hipsolverDnDestroy(solver_handle));

  return 0;
}