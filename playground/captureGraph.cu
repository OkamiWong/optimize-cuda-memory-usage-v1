#include "hip/hip_runtime.h"
#include <cstdio>

#include "../utilities/cudaUtilities.hpp"

struct KernelParam {
  int a[1];
};

__global__ void foo(__grid_constant__ const KernelParam p) {
  printf("Executing foo: p = %d\n", p.a[0]);
}

__global__ void bar(int p) {
  printf("Executing bar: p = %d\n", p);
}

int main() {
  KernelParam p;
  p.a[0] = 233;

  hipEvent_t e1, e2;
  checkCudaErrors(hipEventCreate(&e1));
  checkCudaErrors(hipEventCreate(&e2));

  hipStream_t s1, s2;
  checkCudaErrors(hipStreamCreate(&s1));
  checkCudaErrors(hipStreamCreate(&s2));

  checkCudaErrors(hipStreamBeginCapture(s1, hipStreamCaptureModeGlobal));

  bar<<<1, 1, 0, s1>>>(1);
  checkCudaErrors(hipEventRecord(e1, s1));
  checkCudaErrors(hipStreamWaitEvent(s2, e1));
  bar<<<1, 1, 0, s2>>>(2);
  checkCudaErrors(hipEventRecord(e2, s2));
  checkCudaErrors(hipStreamWaitEvent(s1, e2));
  bar<<<1, 1, 0, s1>>>(3);

  hipGraph_t graph;
  checkCudaErrors(hipStreamEndCapture(s1, &graph));

  checkCudaErrors(hipGraphDebugDotPrint(graph, "./graph.dot", hipGraphDebugDotFlagsVerbose));

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExec, s1));
  checkCudaErrors(hipDeviceSynchronize());
  return 0;
}
