#include "hip/hip_runtime.h"
#include <cstdio>

#include "../utilities/cudaUtilities.hpp"

struct KernelParam{
  int a[1];
};

__global__ void foo(__grid_constant__ const KernelParam p) {
  printf("Executing foo\n");
  printf("%d\n", p.a[0]);
}

int main() {
  KernelParam p;
  p.a[0] = 233;

  hipStream_t s;
  checkCudaErrors(hipStreamCreate(&s));
  checkCudaErrors(hipStreamBeginCapture(s, hipStreamCaptureModeGlobal));

  foo<<<1, 1, 0, s>>>(p);

  checkCudaErrors(hipGetLastError());

  hipGraph_t graph;
  checkCudaErrors(hipStreamEndCapture(s, &graph));

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExec, s));
  checkCudaErrors(hipDeviceSynchronize());
  return 0;
}
