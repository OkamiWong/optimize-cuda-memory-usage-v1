#include <map>

#include "../profiling/memoryManager.hpp"
#include "../utilities/constants.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "optimization.hpp"
#include "optimizer.hpp"
#include "taskManager.hpp"

CustomGraph profileAndOptimize(hipGraph_t originalGraph) {
  LOG_TRACE();
  return Optimizer::getInstance()->profileAndOptimize(originalGraph);
}

void distributeInitialData(CustomGraph& optimizedGraph) {
  LOG_TRACE();

  std::map<void*, bool> visited;
  for (const auto& [ptr, size] : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    visited[ptr] = true;
    checkCudaErrors(hipMemPrefetchAsync(ptr, size, Constants::DEVICE_ID));
  }

  for (auto ptr : MemoryManager::managedMemoryAddresses) {
    if (!visited[ptr]) {
      visited[ptr] = true;
      checkCudaErrors(hipMemPrefetchAsync(ptr, MemoryManager::managedMemoryAddressToSizeMap[ptr], Constants::DEVICE_ID));
    }
  }

  checkCudaErrors(hipDeviceSynchronize());
}

void executeOptimizedGraph(CustomGraph& optimizedGraph) {
  LOG_TRACE();
  TaskManager::getInstance()->executeOptimizedGraph(optimizedGraph);
}
