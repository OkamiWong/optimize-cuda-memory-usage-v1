#include "optimization.hpp"
#include "optimizer.hpp"
#include "taskManager.hpp"

CustomGraph profileAndOptimize(hipGraph_t originalGraph) {
  return Optimizer::getInstance()->profileAndOptimize(originalGraph);
}

void executeOptimizedGraph(CustomGraph& optimizedGraph) {
  TaskManager::getInstance()->executeOptimizedGraph(optimizedGraph);
}
