#include "hip/hip_runtime.h"
#include <cassert>

#include "../utilities/cudaGraphExecutionTimelineProfiler.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/disjointSet.hpp"
#include "optimizer.hpp"
#include "strategies/strategies.hpp"
#include "taskManager.hpp"

Optimizer *Optimizer::instance = nullptr;

Optimizer *Optimizer::getInstance() {
  if (instance == nullptr) {
    instance = new Optimizer();
  }
  return instance;
}

CudaGraphExecutionTimeline getCudaGraphExecutionTimeline(hipGraph_t graph) {
  auto profiler = CudaGraphExecutionTimelineProfiler::getInstance();
  profiler->initialize(graph);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  checkCudaErrors(hipDeviceSynchronize());

  profiler->finalize();

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipStreamDestroy(stream));

  return profiler->getTimeline();
}

void mergeConcurrentCudaGraphNodes(
  const CudaGraphExecutionTimeline &timeline,
  DisjointSet<hipGraphNode_t> &disjointSet
) {
  std::map<CudaGraphNodeLifetime, hipGraphNode_t> lifetimeToCudaGraphNodeMap;
  for (auto &[node, lifetime] : timeline) {
    lifetimeToCudaGraphNodeMap[lifetime] = node;
  }

  uint64_t currentWindowEnd = 0;
  hipGraphNode_t currentWindowRepresentativeNode = nullptr;
  for (auto &[lifetime, node] : lifetimeToCudaGraphNodeMap) {
    assert(lifetime.first != 0 && lifetime.second != 0);

    if (currentWindowRepresentativeNode != nullptr && lifetime.first <= currentWindowEnd) {
      disjointSet.unionUnderlyingSets(currentWindowRepresentativeNode, node);
      currentWindowEnd = std::max(currentWindowEnd, lifetime.second);
    } else {
      currentWindowRepresentativeNode = node;
      currentWindowEnd = lifetime.second;
    }
  }
}

void mergeCudaGraphNodesWithSameAnnotation(hipGraph_t originalGraph, DisjointSet<hipGraphNode_t> &disjointSet) {

}

OptimizationInput constructOptimizationInput(hipGraph_t originalGraph, const CudaGraphExecutionTimeline &timeline, const DisjointSet<hipGraphNode_t> &disjointSet) {
}

CustomGraph Optimizer::profileAndOptimize(hipGraph_t originalGraph) {
  // Profile
  auto taskManager = TaskManager::getInstance();
  taskManager->registerDummyKernelHandle(originalGraph);

  auto timeline = getCudaGraphExecutionTimeline(originalGraph);

  DisjointSet<hipGraphNode_t> disjointSet;
  mergeConcurrentCudaGraphNodes(timeline, disjointSet);
  mergeCudaGraphNodesWithSameAnnotation(originalGraph, disjointSet);

  auto optimizationInput = constructOptimizationInput(originalGraph, timeline, disjointSet);

  // Optimize
  auto customGraph = this->optimize<TwoStepOptimizationStrategy>(optimizationInput);
  return customGraph;
}
