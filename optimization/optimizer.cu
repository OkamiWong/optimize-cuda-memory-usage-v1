#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cassert>
#include <limits>
#include <utility>

#include "../profiling/annotation.hpp"
#include "../profiling/cudaGraphExecutionTimelineProfiler.hpp"
#include "../profiling/memoryManager.hpp"
#include "../utilities/configurationManager.hpp"
#include "../utilities/cudaGraphUtilities.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/disjointSet.hpp"
#include "../utilities/logger.hpp"
#include "optimizer.hpp"
#include "strategies/strategies.hpp"

static hipFunction_t dummyKernelFuncHandle;

void registerDummyKernelFuncHandle(hipGraph_t graph) {
  // The graph is assumed to have only one root
  // and that root is supposed to be an annotation node
  hipKernelNodeParams rootNodeParams;
  getKernelNodeParams(getRootNode(graph), rootNodeParams);
  dummyKernelFuncHandle = rootNodeParams.func;
}

CudaGraphExecutionTimeline getCudaGraphExecutionTimeline(hipGraph_t graph) {
  LOG_TRACE();

  auto profiler = CudaGraphExecutionTimelineProfiler::getInstance();
  profiler->initialize(graph);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  checkCudaErrors(hipDeviceSynchronize());

  profiler->finalize();

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipStreamDestroy(stream));

  return profiler->getTimeline();
}

void mergeConcurrentCudaGraphNodes(
  CudaGraphExecutionTimeline &timeline,
  DisjointSet<hipGraphNode_t> &disjointSet,
  const int logicalNodeSizeLimit
) {
  LOG_TRACE();

  std::map<CudaGraphNodeLifetime, hipGraphNode_t> lifetimeToCudaGraphNodeMap;
  for (auto &[node, lifetime] : timeline) {
    lifetimeToCudaGraphNodeMap[lifetime] = node;
  }

  uint64_t currentWindowEnd = 0;
  hipGraphNode_t currentWindowRepresentativeNode = nullptr;
  for (auto &[lifetime, node] : lifetimeToCudaGraphNodeMap) {
    // Ignore mem alloc node and mem free node
    hipGraphNodeType nodeType;
    checkCudaErrors(hipGraphNodeGetType(node, &nodeType));
    if (nodeType == hipGraphNodeTypeMemAlloc || nodeType == hipGraphNodeTypeMemFree) {
      continue;
    }

    assert(lifetime.first != 0 && lifetime.second != 0);

    if (currentWindowRepresentativeNode != nullptr && lifetime.first <= currentWindowEnd && disjointSet.getSetSize(currentWindowRepresentativeNode) < logicalNodeSizeLimit) {
      disjointSet.unionUnderlyingSets(currentWindowRepresentativeNode, node);
      currentWindowEnd = std::max(currentWindowEnd, lifetime.second);
    } else {
      currentWindowRepresentativeNode = node;
      currentWindowEnd = lifetime.second;
    }
  }
}

void dfs(
  hipGraphNode_t currentNode,
  hipGraphNode_t currentAnnotationNode,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges,
  std::map<hipGraphNode_t, hipGraphNode_t> &nodeToAnnotationMap
) {
  if (nodeToAnnotationMap.find(currentNode) != nodeToAnnotationMap.end()) {
    return;
  }

  bool isAnnotationNode = false;

  if (!currentAnnotationNode) {
    isAnnotationNode = true;
  } else {
    hipGraphNodeType nodeType;
    checkCudaErrors(hipGraphNodeGetType(currentNode, &nodeType));
    if (nodeType == hipGraphNodeTypeKernel) {
      // Why switch to driver API:
      // https://forums.developer.nvidia.com/t/cuda-runtime-api-error-for-cuda-graph-and-opencv/215408/13
      hipKernelNodeParams nodeParams;
      checkCudaErrors(hipGraphKernelNodeGetParams(currentNode, &nodeParams));

      if (nodeParams.func == dummyKernelFuncHandle) {
        isAnnotationNode = true;
      }
    }
  }

  if (isAnnotationNode) {
    currentAnnotationNode = currentNode;
  }

  nodeToAnnotationMap[currentNode] = currentAnnotationNode;

  for (auto nextNode : edges[currentNode]) {
    dfs(nextNode, currentAnnotationNode, edges, nodeToAnnotationMap);
  }
}

void mapNodeToAnnotation(
  hipGraph_t originalGraph,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges,
  std::map<hipGraphNode_t, hipGraphNode_t> &nodeToAnnotationMap
) {
  LOG_TRACE();

  auto rootNode = getRootNode(originalGraph);
  dfs(rootNode, nullptr, edges, nodeToAnnotationMap);
}

void mergeNodesWithSameAnnotation(
  std::vector<hipGraphNode_t> &nodes,
  std::map<hipGraphNode_t, hipGraphNode_t> &nodeToAnnotationMap,
  DisjointSet<hipGraphNode_t> &disjointSet
) {
  for (auto u : nodes) {
    disjointSet.unionUnderlyingSets(u, nodeToAnnotationMap[u]);
  }
}

OptimizationInput::TaskGroup::DataDependency convertTaskAnnotationToTaskGroupDataDependency(
  const TaskAnnotation &taskAnnotation
) {
  OptimizationInput::TaskGroup::DataDependency dep;
  for (int i = 0; i < TaskAnnotation::MAX_NUM_PTR; i++) {
    void *ptr = taskAnnotation.inputs[i];
    if (ptr == nullptr) break;
    dep.inputs.insert(ptr);
  }
  for (int i = 0; i < TaskAnnotation::MAX_NUM_PTR; i++) {
    void *ptr = taskAnnotation.outputs[i];
    if (ptr == nullptr) break;
    dep.outputs.insert(ptr);
  }
  return dep;
}

TaskId getTaskId(hipGraphNode_t annotationNode) {
  hipKernelNodeParams nodeParams;
  getKernelNodeParams(annotationNode, nodeParams);
  assert(nodeParams.func == dummyKernelFuncHandle);

  auto taskAnnotationPtr = reinterpret_cast<TaskAnnotation *>(nodeParams.kernelParams[0]);
  return taskAnnotationPtr->taskId;
}

void mergeDataDependency(OptimizationInput::TaskGroup &taskGroup, hipGraphNode_t annotationNode) {
  hipKernelNodeParams nodeParams;
  getKernelNodeParams(annotationNode, nodeParams);
  assert(nodeParams.func == dummyKernelFuncHandle);

  auto taskAnnotationPtr = reinterpret_cast<TaskAnnotation *>(nodeParams.kernelParams[0]);
  auto taskDataDependency = convertTaskAnnotationToTaskGroupDataDependency(*taskAnnotationPtr);

  taskGroup.dataDependency.inputs.insert(taskDataDependency.inputs.begin(), taskDataDependency.inputs.end());
  taskGroup.dataDependency.outputs.insert(taskDataDependency.outputs.begin(), taskDataDependency.outputs.end());
}

OptimizationInput constructOptimizationInput(
  hipGraph_t originalGraph,
  std::vector<hipGraphNode_t> &nodes,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges,
  CudaGraphExecutionTimeline &timeline,
  DisjointSet<hipGraphNode_t> &disjointSet,
  std::map<hipGraphNode_t, hipGraphNode_t> &nodeToAnnotationMap
) {
  LOG_TRACE();

  OptimizationInput optimizationInput;

  std::map<hipGraphNode_t, TaskGroupId> disjointSetRootToTaskGroupIdMap;

  auto getTaskGroupId = [&](hipGraphNode_t u) {
    auto uTaskId = getTaskId(nodeToAnnotationMap[u]);
    auto uRoot = disjointSet.findRoot(u);

    size_t uTaskGroupId;
    if (disjointSetRootToTaskGroupIdMap.count(uRoot) == 0) {
      optimizationInput.nodes.emplace_back();
      uTaskGroupId = optimizationInput.nodes.size() - 1;
      disjointSetRootToTaskGroupIdMap[uRoot] = uTaskGroupId;
    } else {
      uTaskGroupId = disjointSetRootToTaskGroupIdMap[uRoot];
    }

    optimizationInput.nodes[uTaskGroupId].nodes.insert(uTaskId);

    return uTaskGroupId;
  };

  // Add nodes and edges for both task groups and tasks
  std::set<std::pair<TaskGroupId, TaskGroupId>> existingEdges;
  for (const auto &[u, destinations] : edges) {
    auto uTaskId = getTaskId(nodeToAnnotationMap[u]);
    auto uTaskGroupId = getTaskGroupId(u);

    for (auto v : destinations) {
      auto vTaskId = getTaskId(nodeToAnnotationMap[v]);
      auto vTaskGroupId = getTaskGroupId(v);
      if (uTaskGroupId == vTaskGroupId) {
        optimizationInput.nodes[uTaskGroupId].edges[uTaskId].push_back(vTaskId);
      } else {
        // Edges between task groups need deduping
        if (existingEdges.count(std::make_pair(uTaskGroupId, vTaskGroupId)) == 0) {
          existingEdges.insert(std::make_pair(uTaskGroupId, vTaskGroupId));
          optimizationInput.edges[uTaskGroupId].push_back(vTaskGroupId);
        }
      }
    }
  }

  // Gather information about tasks
  std::map<TaskId, hipGraphNode_t> taskIdToAnnotationNodeMap;
  std::map<TaskId, uint64_t> taskIdToMinStartTimestampMap;
  std::map<TaskId, uint64_t> taskIdToMaxEndTimestampMap;
  for (hipGraphNode_t u : nodes) {
    auto uTaskId = getTaskId(nodeToAnnotationMap[u]);

    if (taskIdToAnnotationNodeMap.count(uTaskId) == 0) {
      taskIdToAnnotationNodeMap[uTaskId] = nodeToAnnotationMap[u];
      taskIdToMinStartTimestampMap[uTaskId] = std::numeric_limits<uint64_t>::max();
      taskIdToMaxEndTimestampMap[uTaskId] = 0;
    }

    // Ignore annotation node
    const bool isAnnotationNode = nodeToAnnotationMap[u] == u;
    if (isAnnotationNode) continue;

    // Ignore mem alloc node and mem free node
    hipGraphNodeType nodeType;
    checkCudaErrors(hipGraphNodeGetType(u, &nodeType));
    if (nodeType == hipGraphNodeTypeMemAlloc || nodeType == hipGraphNodeTypeMemFree) {
      continue;
    }

    taskIdToMinStartTimestampMap[uTaskId] = std::min(
      taskIdToMinStartTimestampMap[uTaskId],
      timeline[u].first
    );

    taskIdToMaxEndTimestampMap[uTaskId] = std::max(
      taskIdToMaxEndTimestampMap[uTaskId],
      timeline[u].second
    );
  }

  // Add task group running time and data dependency
  uint64_t globalMinStart = std::numeric_limits<uint64_t>::max(), globalMaxEnd = 0;
  for (auto &taskGroup : optimizationInput.nodes) {
    uint64_t minStart = std::numeric_limits<uint64_t>::max(), maxEnd = 0;

    for (auto taskId : taskGroup.nodes) {
      minStart = std::min(minStart, taskIdToMinStartTimestampMap[taskId]);
      maxEnd = std::max(maxEnd, taskIdToMaxEndTimestampMap[taskId]);

      mergeDataDependency(taskGroup, taskIdToAnnotationNodeMap[taskId]);
    }

    globalMinStart = std::min(globalMinStart, minStart);
    globalMaxEnd = std::max(globalMaxEnd, maxEnd);

    taskGroup.runningTime = static_cast<float>(maxEnd - minStart) * 1e-9f;
  }

  optimizationInput.originalTotalRunningTime = static_cast<float>(globalMaxEnd - globalMinStart) * 1e-9f;

  return optimizationInput;
}

Optimizer *Optimizer::instance = nullptr;

Optimizer *Optimizer::getInstance() {
  if (instance == nullptr) {
    instance = new Optimizer();
  }
  return instance;
}

OptimizationOutput Optimizer::profileAndOptimize(hipGraph_t originalGraph) {
  registerDummyKernelFuncHandle(originalGraph);

  auto timeline = getCudaGraphExecutionTimeline(originalGraph);

  std::vector<hipGraphNode_t> nodes;
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> edges;
  extractGraphNodesAndEdges(originalGraph, nodes, edges);

  std::map<hipGraphNode_t, hipGraphNode_t> nodeToAnnotationMap;
  mapNodeToAnnotation(originalGraph, edges, nodeToAnnotationMap);

  DisjointSet<hipGraphNode_t> disjointSet;

  if (ConfigurationManager::getConfig().mergeConcurrentCudaGraphNodes) {
    mergeConcurrentCudaGraphNodes(timeline, disjointSet, std::numeric_limits<int>::max());
  }

  mergeNodesWithSameAnnotation(nodes, nodeToAnnotationMap, disjointSet);

  auto optimizationInput = constructOptimizationInput(originalGraph, nodes, edges, timeline, disjointSet, nodeToAnnotationMap);

  auto optimizedGraph = this->optimize<TwoStepOptimizationStrategy>(optimizationInput);

  if (optimizedGraph.optimal) {
    return optimizedGraph;
  } else {
    LOG_TRACE_WITH_INFO("Could not find any feasible solution");
    exit(-1);
  }
}
