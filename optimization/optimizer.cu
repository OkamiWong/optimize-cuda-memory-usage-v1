#include "hip/hip_runtime.h"
#include <cassert>

#include "../utilities/cudaGraphExecutionTimelineProfiler.hpp"
#include "../utilities/cudaGraphUtilities.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/disjointSet.hpp"
#include "optimizer.hpp"
#include "strategies/strategies.hpp"
#include "taskManager.hpp"

Optimizer *Optimizer::instance = nullptr;

Optimizer *Optimizer::getInstance() {
  if (instance == nullptr) {
    instance = new Optimizer();
  }
  return instance;
}

CudaGraphExecutionTimeline getCudaGraphExecutionTimeline(hipGraph_t graph) {
  auto profiler = CudaGraphExecutionTimelineProfiler::getInstance();
  profiler->initialize(graph);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  checkCudaErrors(hipDeviceSynchronize());

  profiler->finalize();

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipStreamDestroy(stream));

  return profiler->getTimeline();
}

void mergeConcurrentCudaGraphNodes(
  const CudaGraphExecutionTimeline &timeline,
  DisjointSet<hipGraphNode_t> &disjointSet
) {
  std::map<CudaGraphNodeLifetime, hipGraphNode_t> lifetimeToCudaGraphNodeMap;
  for (auto &[node, lifetime] : timeline) {
    lifetimeToCudaGraphNodeMap[lifetime] = node;
  }

  uint64_t currentWindowEnd = 0;
  hipGraphNode_t currentWindowRepresentativeNode = nullptr;
  for (auto &[lifetime, node] : lifetimeToCudaGraphNodeMap) {
    assert(lifetime.first != 0 && lifetime.second != 0);

    if (currentWindowRepresentativeNode != nullptr && lifetime.first <= currentWindowEnd) {
      disjointSet.unionUnderlyingSets(currentWindowRepresentativeNode, node);
      currentWindowEnd = std::max(currentWindowEnd, lifetime.second);
    } else {
      currentWindowRepresentativeNode = node;
      currentWindowEnd = lifetime.second;
    }
  }
}

void dfs(
  hipGraphNode_t currentNode,
  hipGraphNode_t parent,
  const std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges,
  DisjointSet<hipGraphNode_t> &disjointSet
) {
  bool isAnnotationNode = false;

  if (!parent) {
    isAnnotationNode = true;
  } else {
    hipGraphNodeType nodeType;
    checkCudaErrors(hipGraphNodeGetType(currentNode, &nodeType));
    if (nodeType == hipGraphNodeTypeKernel) {
      hipKernelNodeParams nodeParams;
      checkCudaErrors(hipGraphKernelNodeGetParams(currentNode, &nodeParams));
      if (nodeParams.func == TaskManager::getInstance()->getDummyKernelHandle()) {
        isAnnotationNode = true;
      }
    }
  }

  if (!isAnnotationNode) {
    disjointSet.unionUnderlyingSets(currentNode, parent);
  }

  for (auto nextNode : edges[currentNode]) {
    dfs(nextNode, currentNode, edges, disjointSet);
  }
}

void mergeCudaGraphNodesWithSameAnnotation(
  hipGraphNode_t rootNode,
  const std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges,
  DisjointSet<hipGraphNode_t> &disjointSet
) {
  auto rootNode = getRootNode(originalGraph);
  dfs(rootNode, nullptr, edges, disjointSet);
}

OptimizationInput constructOptimizationInput(hipGraph_t originalGraph, const CudaGraphExecutionTimeline &timeline, const DisjointSet<hipGraphNode_t> &disjointSet) {
}

CustomGraph Optimizer::profileAndOptimize(hipGraph_t originalGraph) {
  // Profile
  auto taskManager = TaskManager::getInstance();
  taskManager->registerDummyKernelHandle(originalGraph);

  auto timeline = getCudaGraphExecutionTimeline(originalGraph);

  DisjointSet<hipGraphNode_t> disjointSet;
  mergeConcurrentCudaGraphNodes(timeline, disjointSet);

  std::vector<hipGraphNode_t> nodes;
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> edges;
  extractGraphNodesAndEdges(originalGraph, nodes, edges);

  mergeCudaGraphNodesWithSameAnnotation(getRootNode(originalGraph), edges, disjointSet);
  auto optimizationInput = constructOptimizationInput(originalGraph, timeline, disjointSet);

  // Optimize
  auto customGraph = this->optimize<TwoStepOptimizationStrategy>(optimizationInput);
  return customGraph;
}
