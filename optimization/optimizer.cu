#include <type_traits>

#include "optimizer.hpp"
#include "strategies/strategies.hpp"
#include "taskManager.hpp"
#include "../utilities/cudaGraphExecutionTimelineProfiler.hpp"
#include "../utilities/cudaUtilities.hpp"

Optimizer *Optimizer::instance = nullptr;

Optimizer *Optimizer::getInstance() {
  if (instance == nullptr) {
    instance = new Optimizer();
  }
  return instance;
}

typedef std::map<hipGraphNode_t, hipGraphNode_t> CudaGraphNodeDisjointSet;

CudaGraphExecutionTimeline getCudaGraphExecutionTimeline(hipGraph_t graph){
  auto profiler = CudaGraphExecutionTimelineProfiler::getInstance();
  profiler->initialize(graph);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  checkCudaErrors(hipDeviceSynchronize());

  profiler->finalize();

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipStreamDestroy(stream));

  return profiler->getTimeline();
}

void mergeConcurrentCudaGraphNodes(
  hipGraph_t originalGraph,
  const CudaGraphExecutionTimeline &timeline,
  CudaGraphNodeDisjointSet &disjointSet
) {
}

void mergeCudaGraphNodesWithSameAnnotation(hipGraph_t originalGraph, CudaGraphNodeDisjointSet &disjointSet) {
}

OptimizationInput constructOptimizationInput(hipGraph_t originalGraph, const CudaGraphExecutionTimeline &timeline, const CudaGraphNodeDisjointSet &disjointSet) {
}

CustomGraph Optimizer::profileAndOptimize(hipGraph_t originalGraph) {
  // Profile
  auto taskManager = TaskManager::getInstance();
  taskManager->registerDummyKernelHandle(originalGraph);

  auto timeline = getCudaGraphExecutionTimeline(originalGraph);

  CudaGraphNodeDisjointSet disjointSet;
  mergeConcurrentCudaGraphNodes(originalGraph, timeline, disjointSet);
  mergeCudaGraphNodesWithSameAnnotation(originalGraph, disjointSet);

  auto optimizationInput = constructOptimizationInput(originalGraph, timeline, disjointSet);

  // Optimize
  auto customGraph = this->optimize<TwoStepOptimizationStrategy>(optimizationInput);
  return customGraph;
}
