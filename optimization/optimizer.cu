#include "hip/hip_runtime.h"
#include <cassert>
#include <limits>
#include <utility>

#include "../profiling/annotation.hpp"
#include "../utilities/cudaGraphExecutionTimelineProfiler.hpp"
#include "../utilities/cudaGraphUtilities.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/disjointSet.hpp"
#include "optimizer.hpp"
#include "strategies/strategies.hpp"
#include "taskManager.hpp"

Optimizer *Optimizer::instance = nullptr;

Optimizer *Optimizer::getInstance() {
  if (instance == nullptr) {
    instance = new Optimizer();
  }
  return instance;
}

CudaGraphExecutionTimeline getCudaGraphExecutionTimeline(hipGraph_t graph) {
  auto profiler = CudaGraphExecutionTimelineProfiler::getInstance();
  profiler->initialize(graph);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  checkCudaErrors(hipDeviceSynchronize());

  profiler->finalize();

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipStreamDestroy(stream));

  return profiler->getTimeline();
}

void mergeConcurrentCudaGraphNodes(
  CudaGraphExecutionTimeline &timeline,
  DisjointSet<hipGraphNode_t> &disjointSet
) {
  std::map<CudaGraphNodeLifetime, hipGraphNode_t> lifetimeToCudaGraphNodeMap;
  for (auto &[node, lifetime] : timeline) {
    lifetimeToCudaGraphNodeMap[lifetime] = node;
  }

  uint64_t currentWindowEnd = 0;
  hipGraphNode_t currentWindowRepresentativeNode = nullptr;
  for (auto &[lifetime, node] : lifetimeToCudaGraphNodeMap) {
    assert(lifetime.first != 0 && lifetime.second != 0);

    if (currentWindowRepresentativeNode != nullptr && lifetime.first <= currentWindowEnd) {
      disjointSet.unionUnderlyingSets(currentWindowRepresentativeNode, node);
      currentWindowEnd = std::max(currentWindowEnd, lifetime.second);
    } else {
      currentWindowRepresentativeNode = node;
      currentWindowEnd = lifetime.second;
    }
  }
}

void dfs(
  hipGraphNode_t currentNode,
  hipGraphNode_t currentAnnotationNode,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges,
  std::map<hipGraphNode_t, hipGraphNode_t> &nodeToAnnotationMap
) {
  bool isAnnotationNode = false;

  if (!currentAnnotationNode) {
    isAnnotationNode = true;
  } else {
    hipGraphNodeType nodeType;
    checkCudaErrors(hipGraphNodeGetType(currentNode, &nodeType));
    if (nodeType == hipGraphNodeTypeKernel) {
      hipKernelNodeParams nodeParams;
      checkCudaErrors(hipGraphKernelNodeGetParams(currentNode, &nodeParams));
      if (nodeParams.func == TaskManager::getInstance()->getDummyKernelHandle()) {
        isAnnotationNode = true;
      }
    }
  }

  if (isAnnotationNode) {
    currentAnnotationNode = currentNode;
  } else {
    nodeToAnnotationMap[currentNode] = currentAnnotationNode;
  }

  for (auto nextNode : edges[currentNode]) {
    dfs(nextNode, currentAnnotationNode, edges, nodeToAnnotationMap);
  }
}

void mapNodeToAnnotation(
  hipGraph_t originalGraph,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges,
  std::map<hipGraphNode_t, hipGraphNode_t> &nodeToAnnotationMap
) {
  auto rootNode = getRootNode(originalGraph);
  dfs(rootNode, nullptr, edges, nodeToAnnotationMap);
}

OptimizationInput::LogicalNode::DataDependency convertKernelIOToKernelDataDependency(const KernelIO &kernelIO) {
  OptimizationInput::LogicalNode::DataDependency dep;
  for (int i = 0; i < KernelIO::MAX_NUM_PTR; i++) {
    void *ptr = kernelIO.inputs[i];
    if (ptr == nullptr) break;
    dep.inputs.insert(std::make_tuple(ptr, MemoryManager::managedMemoryAddressToSizeMap[ptr]));
  }
  for (int i = 0; i < KernelIO::MAX_NUM_PTR; i++) {
    void *ptr = kernelIO.outputs[i];
    if (ptr == nullptr) break;
    dep.outputs.insert(std::make_tuple(ptr, MemoryManager::managedMemoryAddressToSizeMap[ptr]));
  }
  return dep;
}

void mergeDataDependency(OptimizationInput::LogicalNode &logicalNode, hipGraphNode_t annotationNode) {
  hipGraphNodeType nodeType;
  checkCudaErrors(hipGraphNodeGetType(annotationNode, &nodeType));
  assert(nodeType == hipGraphNodeTypeKernel);
  hipKernelNodeParams nodeParams;
  checkCudaErrors(hipGraphKernelNodeGetParams(annotationNode, &nodeParams));
  assert(nodeParams.func == TaskManager::getInstance()->getDummyKernelHandle());

  auto kernelIOPtr = reinterpret_cast<KernelIO *>(nodeParams.kernelParams[0]);
  auto dataDependencyByAnnotation = convertKernelIOToKernelDataDependency(*kernelIOPtr);

  logicalNode.dataDependency.inputs.insert(dataDependencyByAnnotation.inputs.begin(), dataDependencyByAnnotation.inputs.end());
  logicalNode.dataDependency.outputs.insert(dataDependencyByAnnotation.outputs.begin(), dataDependencyByAnnotation.outputs.end());
}

OptimizationInput constructOptimizationInput(
  hipGraph_t originalGraph,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges,
  CudaGraphExecutionTimeline &timeline,
  DisjointSet<hipGraphNode_t> &disjointSet,
  std::map<hipGraphNode_t, hipGraphNode_t> &nodeToAnnotationMap
) {
  OptimizationInput optimizationInput;

  std::map<hipGraphNode_t, OptimizationInput::NodeId> disjointSetRootToLogicalNodeIndexMap;

  auto getLogicalNodeId = [&](hipGraphNode_t u) {
    auto uRoot = disjointSet.findRoot(u);

    size_t uLogicalNodeId;
    if (disjointSetRootToLogicalNodeIndexMap.find(uRoot) == disjointSetRootToLogicalNodeIndexMap.end()) {
      optimizationInput.nodes.emplace_back();
      uLogicalNodeId = optimizationInput.nodes.size() - 1;
      disjointSetRootToLogicalNodeIndexMap[uRoot] = uLogicalNodeId;
    } else {
      uLogicalNodeId = disjointSetRootToLogicalNodeIndexMap[uRoot];
    }

    optimizationInput.nodes[uLogicalNodeId].nodes.insert(u);

    return uLogicalNodeId;
  };

  // Add nodes and edges, both logical nodes and actual nodes
  for (const auto &[u, destinations] : edges) {
    auto uLogicalNodeId = getLogicalNodeId(u);

    for (auto v : destinations) {
      auto vLogicalNodeId = getLogicalNodeId(v);
      if (uLogicalNodeId == vLogicalNodeId) {
        optimizationInput.nodes[uLogicalNodeId].edges[u].push_back(v);
      } else {
        optimizationInput.edges[uLogicalNodeId].push_back(vLogicalNodeId);
      }
    }
  }

  // Add duration and data dependency
  for (auto &logicalNode : optimizationInput.nodes) {
    uint64_t minStart = std::numeric_limits<uint64_t>::max(), maxEnd = 0;

    for (auto node : logicalNode.nodes) {
      const auto isAnnotationNode = nodeToAnnotationMap.find(node) == nodeToAnnotationMap.end();
      if (isAnnotationNode) continue;

      minStart = std::min(minStart, timeline[node].first);
      maxEnd = std::max(maxEnd, timeline[node].second);

      mergeDataDependency(logicalNode, nodeToAnnotationMap[node]);
    }

    logicalNode.duration = static_cast<float>(maxEnd - minStart) * 1e-9f;
  }

  return optimizationInput;
}

CustomGraph Optimizer::profileAndOptimize(hipGraph_t originalGraph) {
  // Profile
  auto taskManager = TaskManager::getInstance();
  taskManager->registerDummyKernelHandle(originalGraph);

  auto timeline = getCudaGraphExecutionTimeline(originalGraph);

  DisjointSet<hipGraphNode_t> disjointSet;
  mergeConcurrentCudaGraphNodes(timeline, disjointSet);

  std::vector<hipGraphNode_t> nodes;
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> edges;
  extractGraphNodesAndEdges(originalGraph, nodes, edges);

  std::map<hipGraphNode_t, hipGraphNode_t> nodeToAnnotationMap;
  mapNodeToAnnotation(originalGraph, edges, nodeToAnnotationMap);

  auto optimizationInput = constructOptimizationInput(originalGraph, edges, timeline, disjointSet, nodeToAnnotationMap);

  // Optimize
  auto customGraph = this->optimize<TwoStepOptimizationStrategy>(optimizationInput);
  return customGraph;
}
