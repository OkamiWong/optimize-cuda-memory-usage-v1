#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstdlib>
#include <functional>
#include <map>
#include <memory>
#include <queue>
#include <utility>
#include <vector>

#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "taskManager.hpp"

TaskManager *TaskManager::instance = nullptr;

TaskManager *TaskManager::getInstance() {
  if (instance == nullptr) {
    instance = new TaskManager();
  }
  return instance;
}

void extractGraphNodesAndEdges(
  hipGraph_t graph,
  std::vector<hipGraphNode_t> &nodes,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges
) {
  size_t numNodes, numEdges;
  checkCudaErrors(hipGraphGetNodes(graph, nullptr, &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, nullptr, nullptr, &numEdges));
  auto rawNodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
  auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
  checkCudaErrors(hipGraphGetNodes(graph, rawNodes.get(), &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, from.get(), to.get(), &numEdges));

  nodes.clear();
  for (int i = 0; i < numNodes; i++) {
    nodes.push_back(rawNodes[i]);
  }

  edges.clear();
  for (int i = 0; i < numEdges; i++) {
    edges[from[i]].push_back(to[i]);
  }
}

void TaskManager::initializeSequentialExecutionEnvironment() {
  checkCudaErrors(hipStreamCreate(&(this->sequentialStream)));
}

void TaskManager::finalizeSequentialExecutionEnvironment() {
  checkCudaErrors(hipStreamDestroy(this->sequentialStream));
}

void TaskManager::executeNodeSequentially(hipGraphNode_t node) {
  hipGraphNodeType nodeType;
  checkCudaErrors(hipGraphNodeGetType(node, &nodeType));
  if (nodeType == hipGraphNodeTypeKernel) {
    hipKernelNodeParams params;
    checkCudaErrors(hipGraphKernelNodeGetParams(node, &params));

    HIP_LAUNCH_CONFIG config;
    config.gridDimX = params.gridDimX;
    config.gridDimY = params.gridDimY;
    config.gridDimZ = params.gridDimZ;
    config.blockDimX = params.blockDimX;
    config.blockDimY = params.blockDimY;
    config.blockDimZ = params.blockDimZ;
    config.sharedMemBytes = params.sharedMemBytes;
    config.hStream = this->sequentialStream;

    // Currently kernel attributes are ignored
    config.attrs = NULL;
    config.numAttrs = 0;

    // TODO: translate pointers in the parameters of the kernel

    if (params.func != nullptr) {
      checkCudaErrors(hipDrvLaunchKernelEx(
        &config,
        params.func,
        params.kernelParams,
        params.extra
      ));
    } else if (params.kern != nullptr) {
      checkCudaErrors(hipDrvLaunchKernelEx(
        &config,
        reinterpret_cast<hipFunction_t>(params.kern),
        params.kernelParams,
        params.extra
      ));
    } else {
      LOG_TRACE_WITH_INFO("Currently only support params.func != NULL or params.kernel != NULL");
      exit(-1);
    }
  } else if (nodeType == hipGraphNodeTypeMemAlloc) {
    hipMemAllocNodeParams params;
    checkCudaErrors(hipGraphMemAllocNodeGetParams(node, &params));
    void *ptr;
    checkCudaErrors(hipMallocAsync(&ptr, params.bytesize, this->sequentialStream));
    this->actualAddressInSequentialExecution[params.dptr] = ptr;
  } else if (nodeType == hipGraphNodeTypeMemFree) {
    hipDeviceptr_t dptr;
    checkCudaErrors(hipGraphMemFreeNodeGetParams(node, &dptr));
    checkCudaErrors(hipFreeAsync(this->actualAddressInSequentialExecution[dptr], this->sequentialStream));
  } else if (nodeType == hipGraphNodeTypeMemset) {
    hipMemsetParams params;
    checkCudaErrors(hipGraphMemsetNodeGetParams(node, &params));

    uint32_t value = 0;
    if (params.elementSize == 1) {
      uint8_t v = params.value;
      for (int i = 0; i < 4; i++) {
        value += v;
        value <<= 8;
      }
    }
    if (params.elementSize == 2) {
      uint16_t v = params.value;
      for (int i = 0; i < 2; i++) {
        value += v;
        value <<= 16;
      }
    }
    if (params.elementSize == 4) {
      value = params.value;
    }

    checkCudaErrors(hipMemset2DAsync(
      this->actualAddressInSequentialExecution[params.dst],
      params.pitch,
      value,
      params.width,
      params.height,
      this->sequentialStream
    ));
  } else {
    LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
    exit(-1);
  }
}

std::map<GraphNodeId, float> TaskManager::getKernelRunningTimes(hipGraph_t graph) {
  std::vector<hipGraphNode_t> nodes;
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> edges;
  extractGraphNodesAndEdges(graph, nodes, edges);

  std::map<hipGraphNode_t, int> inDegrees;
  for (auto &[u, outEdges] : edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  std::queue<hipGraphNode_t> nodesToExecute;
  for (auto &u : nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
    }
  }

  this->initializeSequentialExecutionEnvironment();

  // Kahn Algorithm
  std::map<GraphNodeId, float> kernelRunningTimes;
  CudaEventClock clock;
  while (!nodesToExecute.empty()) {
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    clock.start(this->sequentialStream);
    this->executeNodeSequentially(u);
    clock.end(this->sequentialStream);
    checkCudaErrors(hipStreamSynchronize(this->sequentialStream));
    kernelRunningTimes[reinterpret_cast<GraphNodeId>(u)] = clock.getTimeInSeconds();

    for (auto &v : edges[u]) {
      inDegrees[v]--;
      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  this->finalizeSequentialExecutionEnvironment();

  return kernelRunningTimes;
}
