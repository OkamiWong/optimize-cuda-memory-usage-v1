#include <hip/hip_runtime.h>

#include <map>
#include <memory>
#include <vector>

#include "../utilities/cudaUtilities.hpp"
#include "taskManager.hpp"

TaskManager *TaskManager::instance = nullptr;

TaskManager *TaskManager::getInstance() {
  if (instance == nullptr) {
    instance = new TaskManager();
  }
  return instance;
}

void extractGraphNodesAndEdges(
  hipGraph_t graph,
  std::vector<hipGraphNode_t> &nodes,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges
) {
  size_t numNodes, numEdges;
  checkCudaErrors(hipGraphGetNodes(graph, nullptr, &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, nullptr, nullptr, &numEdges));
  auto rawNodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
  auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
  checkCudaErrors(hipGraphGetNodes(graph, rawNodes.get(), &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, from.get(), to.get(), &numEdges));

  nodes.clear();
  for (int i = 0; i < numNodes; i++) {
    nodes.push_back(rawNodes[i]);
  }

  edges.clear();
}

std::map<GraphNodeId, float> TaskManager::getKernelRunningTimes(hipGraph_t graph) {
  // Extract nodes and edges

  std::map<GraphNodeId, float> kernelRunningTimes;
  return kernelRunningTimes;
}
