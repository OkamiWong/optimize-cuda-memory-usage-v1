#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstdlib>
#include <functional>
#include <map>
#include <memory>
#include <queue>
#include <utility>
#include <vector>

#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "taskManager.hpp"

TaskManager *TaskManager::instance = nullptr;

TaskManager *TaskManager::getInstance() {
  if (instance == nullptr) {
    instance = new TaskManager();
  }
  return instance;
}

void extractGraphNodesAndEdges(
  hipGraph_t graph,
  std::vector<hipGraphNode_t> &nodes,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges
) {
  size_t numNodes, numEdges;
  checkCudaErrors(hipGraphGetNodes(graph, nullptr, &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, nullptr, nullptr, &numEdges));
  auto rawNodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
  auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
  checkCudaErrors(hipGraphGetNodes(graph, rawNodes.get(), &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, from.get(), to.get(), &numEdges));

  nodes.clear();
  for (int i = 0; i < numNodes; i++) {
    nodes.push_back(rawNodes[i]);
  }

  edges.clear();
  for (int i = 0; i < numEdges; i++) {
    edges[from[i]].push_back(to[i]);
  }
}

void TaskManager::registerDummyKernelHandle(hipGraph_t graph) {
  size_t numRootNodes;
  checkCudaErrors(hipGraphGetRootNodes(graph, NULL, &numRootNodes));
  assert(numRootNodes == 1);

  auto rootNodes = std::make_unique<hipGraphNode_t[]>(numRootNodes);
  checkCudaErrors(hipGraphGetRootNodes(graph, rootNodes.get(), &numRootNodes));

  hipGraphNodeType nodeType;
  checkCudaErrors(hipGraphNodeGetType(rootNodes[0], &nodeType));
  assert(nodeType == hipGraphNodeTypeKernel);

  hipKernelNodeParams rootNodeParams;
  checkCudaErrors(hipGraphKernelNodeGetParams(rootNodes[0], &rootNodeParams));
  this->dummyKernelHandle = rootNodeParams.func;
}

void TaskManager::initializeSequentialExecutionEnvironment() {
  checkCudaErrors(hipStreamCreate(&(this->sequentialStream)));
}

void TaskManager::finalizeSequentialExecutionEnvironment() {
  checkCudaErrors(hipStreamDestroy(this->sequentialStream));
}

bool TaskManager::executeNodeSequentially(hipGraphNode_t node) {
  hipGraphNodeType nodeType;
  checkCudaErrors(hipGraphNodeGetType(node, &nodeType));
  if (nodeType == hipGraphNodeTypeKernel) {
    hipKernelNodeParams params;
    checkCudaErrors(hipGraphKernelNodeGetParams(node, &params));

    if (params.func == this->dummyKernelHandle) {
      return false;
    }

    HIP_LAUNCH_CONFIG config;
    config.gridDimX = params.gridDimX;
    config.gridDimY = params.gridDimY;
    config.gridDimZ = params.gridDimZ;
    config.blockDimX = params.blockDimX;
    config.blockDimY = params.blockDimY;
    config.blockDimZ = params.blockDimZ;
    config.sharedMemBytes = params.sharedMemBytes;
    config.hStream = this->sequentialStream;

    // Currently kernel attributes are ignored
    config.attrs = NULL;
    config.numAttrs = 0;

    if (params.func != nullptr) {
      checkCudaErrors(hipDrvLaunchKernelEx(
        &config,
        params.func,
        params.kernelParams,
        params.extra
      ));
    } else if (params.kern != nullptr) {
      checkCudaErrors(hipDrvLaunchKernelEx(
        &config,
        reinterpret_cast<hipFunction_t>(params.kern),
        params.kernelParams,
        params.extra
      ));
    } else {
      LOG_TRACE_WITH_INFO("Currently only support params.func != NULL or params.kernel != NULL");
      exit(-1);
    }
  } else {
    LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
    exit(-1);
  }

  return true;
}

std::map<hipGraphNode_t, float> TaskManager::getKernelRunningTimes(hipGraph_t graph) {
  std::vector<hipGraphNode_t> nodes;
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> edges;
  extractGraphNodesAndEdges(graph, nodes, edges);

  std::map<hipGraphNode_t, int> inDegrees;
  for (auto &[u, outEdges] : edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  std::queue<hipGraphNode_t> nodesToExecute;
  for (auto &u : nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
    }
  }

  this->initializeSequentialExecutionEnvironment();

  // Kahn Algorithm
  std::map<hipGraphNode_t, float> kernelRunningTimes;
  CudaEventClock clock;
  while (!nodesToExecute.empty()) {
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    clock.start(this->sequentialStream);
    auto isExecuted = this->executeNodeSequentially(u);
    clock.end(this->sequentialStream);
    checkCudaErrors(hipStreamSynchronize(this->sequentialStream));
    if (isExecuted) {
      kernelRunningTimes[u] = clock.getTimeInSeconds();
    }

    for (auto &v : edges[u]) {
      inDegrees[v]--;
      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  this->finalizeSequentialExecutionEnvironment();

  return kernelRunningTimes;
}

void TaskManager::executeOptimizedGraph(const CustomGraph &optimizedGraph) {
}
