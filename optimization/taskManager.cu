#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstdlib>
#include <functional>
#include <map>
#include <memory>
#include <queue>
#include <utility>
#include <vector>

#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "taskManager.hpp"

TaskManager *TaskManager::instance = nullptr;

TaskManager *TaskManager::getInstance() {
  if (instance == nullptr) {
    instance = new TaskManager();
  }
  return instance;
}

void extractGraphNodesAndEdges(
  hipGraph_t graph,
  std::vector<hipGraphNode_t> &nodes,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges
) {
  size_t numNodes, numEdges;
  checkCudaErrors(hipGraphGetNodes(graph, nullptr, &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, nullptr, nullptr, &numEdges));
  auto rawNodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
  auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
  checkCudaErrors(hipGraphGetNodes(graph, rawNodes.get(), &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, from.get(), to.get(), &numEdges));

  nodes.clear();
  for (int i = 0; i < numNodes; i++) {
    nodes.push_back(rawNodes[i]);
  }

  edges.clear();
  for (int i = 0; i < numEdges; i++) {
    edges[from[i]].push_back(to[i]);
  }
}

void executeNode(hipGraphNode_t node) {
  hipGraphNodeType nodeType;
  checkCudaErrors(hipGraphNodeGetType(node, &nodeType));
  if (nodeType == hipGraphNodeTypeKernel) {
    // TODO
  } else if (nodeType == hipGraphNodeTypeMemAlloc) {
    // TODO
  } else if (nodeType == hipGraphNodeTypeMemFree) {
    // TODO
  } else if (nodeType == hipGraphNodeTypeMemset) {
    // TODO
  } else {
    LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
    exit(-1);
  }
}

std::map<GraphNodeId, float> TaskManager::getKernelRunningTimes(hipGraph_t graph) {
  std::vector<hipGraphNode_t> nodes;
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> edges;
  extractGraphNodesAndEdges(graph, nodes, edges);

  std::map<hipGraphNode_t, int> inDegrees;
  for (auto &[u, outEdges] : edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  typedef std::pair<int, hipGraphNode_t> RemainingNode;
  std::priority_queue<RemainingNode, std::vector<RemainingNode>, std::greater<RemainingNode>> remainingNodes;
  for (auto &node : nodes) {
    remainingNodes.push(std::make_pair(inDegrees[node], node));
  }

  // Kahn Algorithm
  std::map<GraphNodeId, float> kernelRunningTimes;
  CudaEventClock clock;

  while (!remainingNodes.empty()) {
    auto [inDegree, u] = remainingNodes.top();
    assert(inDegree == 0);
    remainingNodes.pop();

    clock.start();
    executeNode(u);
    clock.end();
    checkCudaErrors(hipDeviceSynchronize());
    kernelRunningTimes[reinterpret_cast<GraphNodeId>(u)] = clock.getTimeInSeconds();
  }

  return kernelRunningTimes;
}
