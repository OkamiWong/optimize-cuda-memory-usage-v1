#include <hip/hip_runtime.h>

#include <cassert>

#include "../../profiling/annotation.hpp"
#include "../../utilities/cudaUtilities.hpp"
#include "../taskManager.hpp"
#include "strategyUtilities.hpp"

KernelDataDependency convertKernelIOToKernelDataDependency(const KernelIO& kernelIO) {
  KernelDataDependency dep;

  for (int i = 0; i < KernelIO::MAX_NUM_PTR; i++) {
    void* ptr = kernelIO.inputs[i];

    if (ptr == nullptr) break;

    dep.inputs.push_back(std::make_tuple(ptr, MemoryManager::managedMemoryAddressToSizeMap[ptr]));
  }

  for (int i = 0; i < KernelIO::MAX_NUM_PTR; i++) {
    void* ptr = kernelIO.outputs[i];

    if (ptr == nullptr) break;

    dep.outputs.push_back(std::make_tuple(ptr, MemoryManager::managedMemoryAddressToSizeMap[ptr]));
  }

  return dep;
}

std::map<hipGraphNode_t, KernelDataDependency> mapKernelOntoDataDependency(
  std::vector<hipGraphNode_t>& nodes,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>>& edges
) {
  const hipFunction_t dummyKernelHandle = TaskManager::getInstance()->getDummyKernelHandle();

  std::map<hipGraphNode_t, KernelDataDependency> kernelToDataDependencyMap;

  for (auto& node : nodes) {
    hipGraphNodeType nodeType;
    checkCudaErrors(hipGraphNodeGetType(node, &nodeType));
    if (nodeType == hipGraphNodeTypeKernel) {
      hipKernelNodeParams nodeParams;
      checkCudaErrors(hipGraphKernelNodeGetParams(node, &nodeParams));
      if (nodeParams.func == dummyKernelHandle) {
        assert(edges[node].size() == 1);
        auto kernelNode = edges[node][0];
        auto kernelIOPtr = reinterpret_cast<KernelIO*>(nodeParams.kernelParams[0]);
        kernelToDataDependencyMap[kernelNode] = convertKernelIOToKernelDataDependency(*kernelIOPtr);
      }
    }
  }

  return kernelToDataDependencyMap;
}
