#include <hip/hip_runtime.h>

#include "strategyUtilities.hpp"

std::map<hipGraphNode_t, KernelDataDependency> mapKernelOntoDataDependency(
  const std::vector<hipGraphNode_t>& nodes,
  const std::map<hipGraphNode_t, std::vector<hipGraphNode_t>>& edges
) {
  // TODO
}
