#include <cassert>
#include <memory>
#include <queue>

#include "../profiling/memoryManager.hpp"
#include "../profiling/peakMemoryUsageProfiler.hpp"
#include "../utilities/configurationManager.hpp"
#include "../utilities/constants.hpp"
#include "../utilities/cudaGraphUtilities.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "../utilities/utilities.hpp"
#include "executor.hpp"

class OptimizedCudaGraphCreator {
 public:
  OptimizedCudaGraphCreator(hipStream_t stream, hipGraph_t graph) : stream(stream), graph(graph) {}

  void beginCaptureOperation(const std::vector<hipGraphNode_t> &dependencies) {
    checkCudaErrors(hipStreamBeginCaptureToGraph(this->stream, this->graph, dependencies.data(), nullptr, dependencies.size(), hipStreamCaptureModeGlobal));
  }

  std::vector<hipGraphNode_t> endCaptureOperation() {
    checkCudaErrors(hipStreamEndCapture(this->stream, &this->graph));
    return this->getNewLeafNodesAddedByLastCapture();
  };

  hipGraphNode_t addEmptyNode(const std::vector<hipGraphNode_t> &dependencies) {
    hipGraphNode_t newEmptyNode;
    checkCudaErrors(hipGraphAddEmptyNode(&newEmptyNode, this->graph, dependencies.data(), dependencies.size()));
    visited[newEmptyNode] = true;
    return newEmptyNode;
  }

 private:
  hipStream_t stream;
  hipGraph_t graph;
  std::vector<hipGraphNode_t> lastDependencies;
  std::map<hipGraphNode_t, bool> visited;

  std::vector<hipGraphNode_t> getNewLeafNodesAddedByLastCapture() {
    size_t numNodes;
    checkCudaErrors(hipGraphGetNodes(this->graph, nullptr, &numNodes));
    auto nodes = std::make_unique<hipGraphNode_t[]>(numNodes);
    checkCudaErrors(hipGraphGetNodes(this->graph, nodes.get(), &numNodes));

    size_t numEdges;
    checkCudaErrors(hipGraphGetEdges(this->graph, nullptr, nullptr, &numEdges));
    auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
    auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
    checkCudaErrors(hipGraphGetEdges(this->graph, from.get(), to.get(), &numEdges));

    std::map<hipGraphNode_t, bool> hasOutGoingEdge;
    for (int i = 0; i < numEdges; i++) {
      hasOutGoingEdge[from[i]] = true;
    }

    std::vector<hipGraphNode_t> newLeafNodes;
    for (int i = 0; i < numNodes; i++) {
      auto &node = nodes[i];
      if (!visited[node]) {
        visited[node] = true;
        if (!hasOutGoingEdge[node]) {
          newLeafNodes.push_back(node);
        }
      }
    }

    return newLeafNodes;
  }
};

Executor *Executor::instance = nullptr;

Executor *Executor::getInstance() {
  if (instance == nullptr) {
    instance = new Executor();
  }
  return instance;
}

void Executor::executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  float &runningTime,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap
) {
  LOG_TRACE_WITH_INFO("Initialize");

  managedDeviceArrayToHostArrayMap.clear();

  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  std::queue<int> nodesToExecute;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
    }
  }

  hipMemcpyKind prefetchMemcpyKind = ConfigurationManager::getConfig().useNvlink ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
  hipMemcpyKind offloadMemcpyKind = ConfigurationManager::getConfig().useNvlink ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost;

  if (ConfigurationManager::getConfig().useNvlink) {
    enablePeerAccessForNvlink(Constants::DEVICE_ID, Constants::STORAGE_DEVICE_ID);
  }

  LOG_TRACE_WITH_INFO("Initialize managed data distribution");

  for (auto ptr : MemoryManager::managedMemoryAddresses) {
    void *newPtr;
    if (ConfigurationManager::getConfig().useNvlink) {
      checkCudaErrors(hipSetDevice(Constants::STORAGE_DEVICE_ID));
      checkCudaErrors(hipMalloc(&newPtr, MemoryManager::managedMemoryAddressToSizeMap[ptr]));
    } else {
      newPtr = malloc(MemoryManager::managedMemoryAddressToSizeMap[ptr]);
    }

    managedDeviceArrayToHostArrayMap[ptr] = newPtr;
    checkCudaErrors(hipMemcpy(
      newPtr,
      ptr,
      MemoryManager::managedMemoryAddressToSizeMap[ptr],
      hipMemcpyDefault
    ));
    checkCudaErrors(hipFree(ptr));
  }
  checkCudaErrors(hipSetDevice(Constants::DEVICE_ID));
  checkCudaErrors(hipDeviceSynchronize());

  std::map<void *, void *> addressUpdate;

  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  for (auto arrayId : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    auto ptr = MemoryManager::managedMemoryAddresses[arrayId];
    auto size = MemoryManager::managedMemoryAddressToSizeMap[ptr];
    auto newPtr = managedDeviceArrayToHostArrayMap[ptr];

    void *devicePtr;
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(devicePtr, newPtr, size, prefetchMemcpyKind, stream));
    addressUpdate[ptr] = devicePtr;
  }
  hipGraph_t graphForInitialDataDistribution;
  checkCudaErrors(hipStreamEndCapture(stream, &graphForInitialDataDistribution));

  hipGraphExec_t graphExecForInitialDataDistribution;
  checkCudaErrors(hipGraphInstantiate(&graphExecForInitialDataDistribution, graphForInitialDataDistribution, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExecForInitialDataDistribution, stream));
  checkCudaErrors(hipDeviceSynchronize());

  LOG_TRACE_WITH_INFO("Record nodes to a new CUDA Graph");

  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  // Kahn Algorithm
  while (!nodesToExecute.empty()) {
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    std::vector<hipGraphNode_t> newLeafNodes;

    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      auto dataMovementAddress = MemoryManager::managedMemoryAddresses[dataMovement.arrayId];
      auto dataMovementSize = MemoryManager::managedMemoryAddressToSizeMap[dataMovementAddress];
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        void *devicePtr;
        checkCudaErrors(hipMallocAsync(&devicePtr, dataMovementSize, stream));
        checkCudaErrors(hipMemcpyAsync(
          devicePtr,
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          dataMovementSize,
          prefetchMemcpyKind,
          stream
        ));
        addressUpdate[dataMovementAddress] = devicePtr;
      } else {
        void *devicePtr = addressUpdate[dataMovementAddress];
        checkCudaErrors(hipMemcpyAsync(
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          devicePtr,
          dataMovementSize,
          offloadMemcpyKind,
          stream
        ));
        checkCudaErrors(hipFreeAsync(devicePtr, stream));
        addressUpdate.erase(dataMovementAddress);
      }
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      executeRandomTask(
        optimizedGraph.nodeIdToTaskIdMap[u],
        addressUpdate,
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;

      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].end(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  LOG_TRACE_WITH_INFO("Printing the new CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  LOG_TRACE_WITH_INFO("Execute the new CUDA Graph");
  PeakMemoryUsageProfiler peakMemoryUsageProfiler;
  CudaEventClock cudaEventClock;
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  checkCudaErrors(hipGraphUpload(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  if (ConfigurationManager::getConfig().measurePeakMemoryUsage) {
    peakMemoryUsageProfiler.start();
  }

  cudaEventClock.start();
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  if (ConfigurationManager::getConfig().measurePeakMemoryUsage) {
    const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
    LOG_TRACE_WITH_INFO(
      "Peak memory usage (MiB): %.2f",
      static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
    );
  }

  LOG_TRACE_WITH_INFO("Clean up");
  for (auto &[oldAddr, newAddr] : addressUpdate) {
    checkCudaErrors(hipMemcpy(
      managedDeviceArrayToHostArrayMap[oldAddr],
      newAddr,
      MemoryManager::managedMemoryAddressToSizeMap[oldAddr],
      offloadMemcpyKind
    ));
    checkCudaErrors(hipFree(newAddr));
  }
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipGraphExecDestroy(graphExecForInitialDataDistribution));
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graphForInitialDataDistribution));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  if (ConfigurationManager::getConfig().useNvlink) {
    disablePeerAccessForNvlink(Constants::DEVICE_ID, Constants::STORAGE_DEVICE_ID);
  }

  runningTime = cudaEventClock.getTimeInSeconds();
}

void moveAllDataToHost(std::map<void *, void *> &managedDeviceArrayToHostArrayMap) {
  LOG_TRACE();

  for (auto ptr : MemoryManager::managedMemoryAddresses) {
    void *newPtr;
    if (ConfigurationManager::getConfig().useNvlink) {
      checkCudaErrors(hipSetDevice(Constants::STORAGE_DEVICE_ID));
      checkCudaErrors(hipMalloc(&newPtr, MemoryManager::managedMemoryAddressToSizeMap[ptr]));
    } else {
      newPtr = malloc(MemoryManager::managedMemoryAddressToSizeMap[ptr]);
    }

    managedDeviceArrayToHostArrayMap[ptr] = newPtr;
    checkCudaErrors(hipMemcpy(
      newPtr,
      ptr,
      MemoryManager::managedMemoryAddressToSizeMap[ptr],
      hipMemcpyDefault
    ));
    checkCudaErrors(hipFree(ptr));
  }

  checkCudaErrors(hipSetDevice(Constants::DEVICE_ID));
  checkCudaErrors(hipDeviceSynchronize());
}

void moveInitialDataToDevice(
  OptimizationOutput &optimizedGraph,
  const std::map<void *, void *> &managedDeviceArrayToHostArrayMap,
  std::map<void *, void *> &addressUpdate
) {
  LOG_TRACE();

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

  for (auto arrayId : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    auto ptr = MemoryManager::managedMemoryAddresses[arrayId];
    auto size = MemoryManager::managedMemoryAddressToSizeMap[ptr];
    auto newPtr = managedDeviceArrayToHostArrayMap.at(ptr);

    void *devicePtr;
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(devicePtr, newPtr, size, hipMemcpyDefault, stream));
    addressUpdate[ptr] = devicePtr;
  }

  hipGraph_t graphForInitialDataDistribution;
  checkCudaErrors(hipStreamEndCapture(stream, &graphForInitialDataDistribution));

  hipGraphExec_t graphExecForInitialDataDistribution;
  checkCudaErrors(hipGraphInstantiate(&graphExecForInitialDataDistribution, graphForInitialDataDistribution, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExecForInitialDataDistribution, stream));
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipStreamDestroy(stream));
  checkCudaErrors(hipGraphExecDestroy(graphExecForInitialDataDistribution));
  checkCudaErrors(hipGraphDestroy(graphForInitialDataDistribution));
}

hipGraph_t recordOptimizedCudaGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  std::map<int, int> inDegrees,
  std::queue<int> nodesToExecute,
  const std::map<void *, void *> &managedDeviceArrayToHostArrayMap,
  std::map<void *, void *> &addressUpdate
) {
  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  // Kahn Algorithm
  while (!nodesToExecute.empty()) {
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    std::vector<hipGraphNode_t> newLeafNodes;

    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      auto dataMovementAddress = MemoryManager::managedMemoryAddresses[dataMovement.arrayId];
      auto dataMovementSize = MemoryManager::managedMemoryAddressToSizeMap[dataMovementAddress];
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        assert(addressUpdate.count(dataMovementAddress) == 0);
        void *devicePtr;
        checkCudaErrors(hipMallocAsync(&devicePtr, dataMovementSize, stream));
        checkCudaErrors(hipMemcpyAsync(
          devicePtr,
          managedDeviceArrayToHostArrayMap.at(dataMovementAddress),
          dataMovementSize,
          hipMemcpyDefault,
          stream
        ));
        addressUpdate[dataMovementAddress] = devicePtr;
      } else {
        assert(addressUpdate.count(dataMovementAddress) > 0);
        void *devicePtr = addressUpdate[dataMovementAddress];
        checkCudaErrors(hipMemcpyAsync(
          managedDeviceArrayToHostArrayMap.at(dataMovementAddress),
          devicePtr,
          dataMovementSize,
          hipMemcpyDefault,
          stream
        ));
        checkCudaErrors(hipFreeAsync(devicePtr, stream));
        addressUpdate.erase(dataMovementAddress);
      }
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      executeRandomTask(
        optimizedGraph.nodeIdToTaskIdMap[u],
        addressUpdate,
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;

      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].end(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  checkCudaErrors(hipStreamDestroy(stream));

  return graph;
}

void Executor::executeOptimizedGraphRepeatedly(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  ShouldContinue shouldContinue,
  int &numIterations,
  float &runningTime,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap
) {
  LOG_TRACE_WITH_INFO("Initialize");

  managedDeviceArrayToHostArrayMap.clear();

  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  std::queue<int> nodesToExecute;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
    }
  }

  if (ConfigurationManager::getConfig().useNvlink) {
    enablePeerAccessForNvlink(Constants::DEVICE_ID, Constants::STORAGE_DEVICE_ID);
  }

  moveAllDataToHost(managedDeviceArrayToHostArrayMap);

  std::map<void *, void *> addressUpdate;
  moveInitialDataToDevice(
    optimizedGraph,
    managedDeviceArrayToHostArrayMap,
    addressUpdate
  );

  hipGraph_t graph = recordOptimizedCudaGraph(
    optimizedGraph,
    executeRandomTask,
    inDegrees,
    nodesToExecute,
    managedDeviceArrayToHostArrayMap,
    addressUpdate
  );

  LOG_TRACE_WITH_INFO("Printing the first optimized CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  LOG_TRACE_WITH_INFO("Execute optimized CUDA Graphs");
  PeakMemoryUsageProfiler peakMemoryUsageProfiler;
  CudaEventClock cudaEventClock;
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  checkCudaErrors(hipGraphUpload(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  if (ConfigurationManager::getConfig().measurePeakMemoryUsage) {
    peakMemoryUsageProfiler.start();
  }

  std::map<void *, void *> nextAddressUpdate(addressUpdate);

  numIterations = 0;

  cudaEventClock.start();
  while (shouldContinue()) {
    numIterations++;

    checkCudaErrors(hipGraphLaunch(graphExec, stream));

    addressUpdate = nextAddressUpdate;

    hipGraph_t nextGraph = recordOptimizedCudaGraph(
      optimizedGraph,
      executeRandomTask,
      inDegrees,
      nodesToExecute,
      managedDeviceArrayToHostArrayMap,
      nextAddressUpdate
    );
    hipGraphExec_t nextGraphExec;
    checkCudaErrors(hipGraphInstantiate(&nextGraphExec, nextGraph, nullptr, nullptr, 0));
    checkCudaErrors(hipGraphUpload(nextGraphExec, stream));

    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    graph = nextGraph;
    graphExec = nextGraphExec;
  }
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  if (ConfigurationManager::getConfig().measurePeakMemoryUsage) {
    const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
    LOG_TRACE_WITH_INFO(
      "Peak memory usage (MiB): %.2f",
      static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
    );
  }

  LOG_TRACE_WITH_INFO("Clean up");
  for (auto &[oldAddr, newAddr] : addressUpdate) {
    checkCudaErrors(hipMemcpy(
      managedDeviceArrayToHostArrayMap[oldAddr],
      newAddr,
      MemoryManager::managedMemoryAddressToSizeMap[oldAddr],
      hipMemcpyDefault
    ));
    checkCudaErrors(hipFree(newAddr));
  }
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  if (ConfigurationManager::getConfig().useNvlink) {
    disablePeerAccessForNvlink(Constants::DEVICE_ID, Constants::STORAGE_DEVICE_ID);
  }

  runningTime = cudaEventClock.getTimeInSeconds();
}
