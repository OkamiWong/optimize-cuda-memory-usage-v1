#include "hip/hip_runtime.h"
#include <cassert>
#include <memory>
#include <queue>

#include "../profiling/memoryManager.hpp"
#include "../profiling/peakMemoryUsageProfiler.hpp"
#include "../utilities/configurationManager.hpp"
#include "../utilities/constants.hpp"
#include "../utilities/cudaGraphUtilities.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/utilities.hpp"
#include "../utilities/logger.hpp"
#include "executor.hpp"

class OptimizedCudaGraphCreator {
 public:
  OptimizedCudaGraphCreator(hipStream_t stream, hipGraph_t graph) : stream(stream), graph(graph) {}

  void beginCaptureOperation(const std::vector<hipGraphNode_t> &dependencies) {
    checkCudaErrors(hipStreamBeginCaptureToGraph(this->stream, this->graph, dependencies.data(), nullptr, dependencies.size(), hipStreamCaptureModeGlobal));
  }

  std::vector<hipGraphNode_t> endCaptureOperation() {
    checkCudaErrors(hipStreamEndCapture(this->stream, &this->graph));
    return this->getNewLeafNodesAddedByLastCapture();
  };

  hipGraphNode_t addEmptyNode(const std::vector<hipGraphNode_t> &dependencies) {
    hipGraphNode_t newEmptyNode;
    checkCudaErrors(hipGraphAddEmptyNode(&newEmptyNode, this->graph, dependencies.data(), dependencies.size()));
    visited[newEmptyNode] = true;
    return newEmptyNode;
  }

 private:
  hipStream_t stream;
  hipGraph_t graph;
  std::vector<hipGraphNode_t> lastDependencies;
  std::map<hipGraphNode_t, bool> visited;

  std::vector<hipGraphNode_t> getNewLeafNodesAddedByLastCapture() {
    size_t numNodes;
    checkCudaErrors(hipGraphGetNodes(this->graph, nullptr, &numNodes));
    auto nodes = std::make_unique<hipGraphNode_t[]>(numNodes);
    checkCudaErrors(hipGraphGetNodes(this->graph, nodes.get(), &numNodes));

    size_t numEdges;
    checkCudaErrors(hipGraphGetEdges(this->graph, nullptr, nullptr, &numEdges));
    auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
    auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
    checkCudaErrors(hipGraphGetEdges(this->graph, from.get(), to.get(), &numEdges));

    std::map<hipGraphNode_t, bool> hasOutGoingEdge;
    for (int i = 0; i < numEdges; i++) {
      hasOutGoingEdge[from[i]] = true;
    }

    std::vector<hipGraphNode_t> newLeafNodes;
    for (int i = 0; i < numNodes; i++) {
      auto &node = nodes[i];
      if (!visited[node]) {
        visited[node] = true;
        if (!hasOutGoingEdge[node]) {
          newLeafNodes.push_back(node);
        }
      }
    }

    return newLeafNodes;
  }
};

Executor *Executor::instance = nullptr;

Executor *Executor::getInstance() {
  if (instance == nullptr) {
    instance = new Executor();
  }
  return instance;
}

void Executor::executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  float &runningTime,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap
) {
  LOG_TRACE_WITH_INFO("Initialize");

  managedDeviceArrayToHostArrayMap.clear();

  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  std::queue<int> nodesToExecute;
  std::vector<int> rootNodes;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
      rootNodes.push_back(u);
    }
  }

  int storageDeviceId = ConfigurationManager::getConfig().useNvlink ? Constants::STORAGE_DEVICE_ID : hipCpuDeviceId;
  hipMemcpyKind prefetchMemcpyKind = ConfigurationManager::getConfig().useNvlink ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
  hipMemcpyKind offloadMemcpyKind = ConfigurationManager::getConfig().useNvlink ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost;

  if (ConfigurationManager::getConfig().useNvlink) {
    enablePeerAccessForNvlink(Constants::DEVICE_ID, Constants::STORAGE_DEVICE_ID);
  }

  LOG_TRACE_WITH_INFO("Initialize managed data distribution");

  for (auto ptr : MemoryManager::managedMemoryAddresses) {
    void *newPtr;
    if (ConfigurationManager::getConfig().useNvlink) {
      checkCudaErrors(hipSetDevice(Constants::STORAGE_DEVICE_ID));
      checkCudaErrors(hipMalloc(&newPtr, MemoryManager::managedMemoryAddressToSizeMap[ptr]));
    } else {
      newPtr = malloc(MemoryManager::managedMemoryAddressToSizeMap[ptr]);
    }

    managedDeviceArrayToHostArrayMap[ptr] = newPtr;
    checkCudaErrors(hipMemcpy(
      newPtr,
      ptr,
      MemoryManager::managedMemoryAddressToSizeMap[ptr],
      hipMemcpyDefault
    ));
    checkCudaErrors(hipFree(ptr));
  }
  checkCudaErrors(hipSetDevice(Constants::DEVICE_ID));
  checkCudaErrors(hipDeviceSynchronize());

  std::map<void *, void *> addressUpdate;

  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  for (auto arrayId : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    auto ptr = MemoryManager::managedMemoryAddresses[arrayId];
    auto size = MemoryManager::managedMemoryAddressToSizeMap[ptr];
    auto newPtr = managedDeviceArrayToHostArrayMap[ptr];

    void *devicePtr;
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(devicePtr, newPtr, size, prefetchMemcpyKind, stream));
    addressUpdate[ptr] = devicePtr;
  }
  hipGraph_t graphForInitialDataDistribution;
  checkCudaErrors(hipStreamEndCapture(stream, &graphForInitialDataDistribution));

  hipGraphExec_t graphExecForInitialDataDistribution;
  checkCudaErrors(hipGraphInstantiate(&graphExecForInitialDataDistribution, graphForInitialDataDistribution, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExecForInitialDataDistribution, stream));
  checkCudaErrors(hipDeviceSynchronize());

  LOG_TRACE_WITH_INFO("Record nodes to a new CUDA Graph");

  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  // Kahn Algorithm
  while (!nodesToExecute.empty()) {
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    std::vector<hipGraphNode_t> newLeafNodes;

    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      auto dataMovementAddress = MemoryManager::managedMemoryAddresses[dataMovement.arrayId];
      auto dataMovementSize = MemoryManager::managedMemoryAddressToSizeMap[dataMovementAddress];
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        void *devicePtr;
        checkCudaErrors(hipMallocAsync(&devicePtr, dataMovementSize, stream));
        checkCudaErrors(hipMemcpyAsync(
          devicePtr,
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          dataMovementSize,
          prefetchMemcpyKind,
          stream
        ));
        addressUpdate[dataMovementAddress] = devicePtr;
      } else {
        void *devicePtr = addressUpdate[dataMovementAddress];
        checkCudaErrors(hipMemcpyAsync(
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          devicePtr,
          dataMovementSize,
          offloadMemcpyKind,
          stream
        ));
        checkCudaErrors(hipFreeAsync(devicePtr, stream));
        addressUpdate.erase(dataMovementAddress);
      }
      checkCudaErrors(hipPeekAtLastError());
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
      checkCudaErrors(hipPeekAtLastError());
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      executeRandomTask(
        optimizedGraph.nodeIdToTaskIdMap[u],
        addressUpdate,
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;

      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].end(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  LOG_TRACE_WITH_INFO("Printing the new CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  LOG_TRACE_WITH_INFO("Execute the new CUDA Graph");
  PeakMemoryUsageProfiler peakMemoryUsageProfiler;
  CudaEventClock cudaEventClock;
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  checkCudaErrors(hipGraphUpload(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  if (ConfigurationManager::getConfig().measurePeakMemoryUsage) {
    peakMemoryUsageProfiler.start();
  }

  cudaEventClock.start();
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  if (ConfigurationManager::getConfig().measurePeakMemoryUsage) {
    const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
    LOG_TRACE_WITH_INFO(
      "Peak memory usage (MiB): %.2f",
      static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
    );
  }

  LOG_TRACE_WITH_INFO("Clean up");
  for (auto &[oldAddr, newAddr] : addressUpdate) {
    checkCudaErrors(hipMemcpy(
      managedDeviceArrayToHostArrayMap[oldAddr],
      newAddr,
      MemoryManager::managedMemoryAddressToSizeMap[oldAddr],
      offloadMemcpyKind
    ));
    checkCudaErrors(hipFree(newAddr));
  }
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipGraphExecDestroy(graphExecForInitialDataDistribution));
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graphForInitialDataDistribution));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  if (ConfigurationManager::getConfig().useNvlink) {
    disablePeerAccessForNvlink(Constants::DEVICE_ID, Constants::STORAGE_DEVICE_ID);
  }

  runningTime = cudaEventClock.getTimeInSeconds();
}

void Executor::executeOptimizedGraphRepeatedly(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  ShouldContinue shouldContinue,
  int &numIterations,
  float &runningTime,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap
) {
  LOG_TRACE_WITH_INFO("Initialize");

  managedDeviceArrayToHostArrayMap.clear();

  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  std::queue<int> nodesToExecute;
  std::vector<int> rootNodes;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
      rootNodes.push_back(u);
    }
  }

  int storageDeviceId = ConfigurationManager::getConfig().useNvlink ? Constants::STORAGE_DEVICE_ID : hipCpuDeviceId;
  hipMemcpyKind prefetchMemcpyKind = ConfigurationManager::getConfig().useNvlink ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
  hipMemcpyKind offloadMemcpyKind = ConfigurationManager::getConfig().useNvlink ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost;

  if (ConfigurationManager::getConfig().useNvlink) {
    enablePeerAccessForNvlink(Constants::DEVICE_ID, Constants::STORAGE_DEVICE_ID);
  }

  LOG_TRACE_WITH_INFO("Initialize managed data distribution");

  for (auto ptr : MemoryManager::managedMemoryAddresses) {
    void *newPtr;
    if (ConfigurationManager::getConfig().useNvlink) {
      checkCudaErrors(hipSetDevice(Constants::STORAGE_DEVICE_ID));
      checkCudaErrors(hipMalloc(&newPtr, MemoryManager::managedMemoryAddressToSizeMap[ptr]));
    } else {
      newPtr = malloc(MemoryManager::managedMemoryAddressToSizeMap[ptr]);
    }

    managedDeviceArrayToHostArrayMap[ptr] = newPtr;
    checkCudaErrors(hipMemcpy(
      newPtr,
      ptr,
      MemoryManager::managedMemoryAddressToSizeMap[ptr],
      hipMemcpyDefault
    ));
    checkCudaErrors(hipFree(ptr));
  }
  checkCudaErrors(hipSetDevice(Constants::DEVICE_ID));
  checkCudaErrors(hipDeviceSynchronize());

  SystemWallClock clock;
  clock.start();

  LOG_TRACE_WITH_INFO("Record nodes to a new CUDA Graph");

  std::map<void *, void *> addressUpdate;

  std::vector<hipGraphNode_t> newLeafNodes;
  for (auto arrayId : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    auto ptr = MemoryManager::managedMemoryAddresses[arrayId];
    auto size = MemoryManager::managedMemoryAddressToSizeMap[ptr];
    auto newPtr = managedDeviceArrayToHostArrayMap[ptr];

    void *devicePtr;
    optimizedCudaGraphCreator->beginCaptureOperation(newLeafNodes);
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(devicePtr, newPtr, size, prefetchMemcpyKind, stream));
    newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    addressUpdate[ptr] = devicePtr;
  }

  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  for (auto u : rootNodes) {
    nodeToDependentNodesMap[u] = newLeafNodes;
  }

  // Kahn Algorithm
  while (!nodesToExecute.empty()) {
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    newLeafNodes.clear();

    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      auto dataMovementAddress = MemoryManager::managedMemoryAddresses[dataMovement.arrayId];
      auto dataMovementSize = MemoryManager::managedMemoryAddressToSizeMap[dataMovementAddress];
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        void *devicePtr;
        checkCudaErrors(hipMallocAsync(&devicePtr, dataMovementSize, stream));
        checkCudaErrors(hipMemcpyAsync(
          devicePtr,
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          dataMovementSize,
          prefetchMemcpyKind,
          stream
        ));
        addressUpdate[dataMovementAddress] = devicePtr;
      } else {
        void *devicePtr = addressUpdate[dataMovementAddress];
        checkCudaErrors(hipMemcpyAsync(
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          devicePtr,
          dataMovementSize,
          offloadMemcpyKind,
          stream
        ));
        checkCudaErrors(hipFreeAsync(devicePtr, stream));
        addressUpdate.erase(dataMovementAddress);
      }
      checkCudaErrors(hipPeekAtLastError());
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
      checkCudaErrors(hipPeekAtLastError());
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      executeRandomTask(
        optimizedGraph.nodeIdToTaskIdMap[u],
        addressUpdate,
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;

      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].end(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  newLeafNodes = getNodesWithZeroOutDegree(graph);
  for (auto &[oldAddr, newAddr] : addressUpdate) {
    optimizedCudaGraphCreator->beginCaptureOperation(newLeafNodes);
    checkCudaErrors(hipMemcpyAsync(
      managedDeviceArrayToHostArrayMap[oldAddr],
      newAddr,
      MemoryManager::managedMemoryAddressToSizeMap[oldAddr],
      offloadMemcpyKind,
      stream
    ));
    checkCudaErrors(hipFreeAsync(newAddr, stream));
    newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
  }
  checkCudaErrors(hipDeviceSynchronize());

  clock.end();
  LOG_TRACE_WITH_INFO("Time taken for recording graph: %.6f", clock.getTimeInSeconds());

  LOG_TRACE_WITH_INFO("Printing the new CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  LOG_TRACE_WITH_INFO("Execute the new CUDA Graph");
  PeakMemoryUsageProfiler peakMemoryUsageProfiler;
  CudaEventClock cudaEventClock;
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  checkCudaErrors(hipGraphUpload(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  if (ConfigurationManager::getConfig().measurePeakMemoryUsage) {
    peakMemoryUsageProfiler.start();
  }

  numIterations = 0;

  cudaEventClock.start();
  while (shouldContinue()) {
    checkCudaErrors(hipGraphLaunch(graphExec, stream));
    numIterations++;
    checkCudaErrors(hipDeviceSynchronize());
  }
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  if (ConfigurationManager::getConfig().measurePeakMemoryUsage) {
    const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
    LOG_TRACE_WITH_INFO(
      "Peak memory usage (MiB): %.2f",
      static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
    );
  }

  LOG_TRACE_WITH_INFO("Clean up");
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  if (ConfigurationManager::getConfig().useNvlink) {
    disablePeerAccessForNvlink(Constants::DEVICE_ID, Constants::STORAGE_DEVICE_ID);
  }

  runningTime = cudaEventClock.getTimeInSeconds();
}
