#include <cassert>
#include <memory>
#include <queue>

#include "../profiling/memoryManager.hpp"
#include "../profiling/peakMemoryUsageProfiler.hpp"
#include "../utilities/configurationManager.hpp"
#include "../utilities/constants.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "executor.hpp"

class OptimizedCudaGraphCreator {
 public:
  OptimizedCudaGraphCreator(hipStream_t stream, hipGraph_t graph) : stream(stream), graph(graph) {}

  void beginCaptureOperation(const std::vector<hipGraphNode_t> &dependencies) {
    checkCudaErrors(hipStreamBeginCaptureToGraph(this->stream, this->graph, dependencies.data(), nullptr, dependencies.size(), hipStreamCaptureModeGlobal));
  }

  std::vector<hipGraphNode_t> endCaptureOperation() {
    checkCudaErrors(hipStreamEndCapture(this->stream, &this->graph));
    return this->getNewLeafNodesAddedByLastCapture();
  };

  hipGraphNode_t addEmptyNode(const std::vector<hipGraphNode_t> &dependencies) {
    hipGraphNode_t newEmptyNode;
    checkCudaErrors(hipGraphAddEmptyNode(&newEmptyNode, this->graph, dependencies.data(), dependencies.size()));
    return newEmptyNode;
  }

 private:
  hipStream_t stream;
  hipGraph_t graph;
  std::vector<hipGraphNode_t> lastDependencies;
  std::map<hipGraphNode_t, bool> visited;

  std::vector<hipGraphNode_t> getNewLeafNodesAddedByLastCapture() {
    size_t numNodes;
    checkCudaErrors(hipGraphGetNodes(this->graph, nullptr, &numNodes));
    auto nodes = std::make_unique<hipGraphNode_t[]>(numNodes);
    checkCudaErrors(hipGraphGetNodes(this->graph, nodes.get(), &numNodes));

    size_t numEdges;
    checkCudaErrors(hipGraphGetEdges(this->graph, nullptr, nullptr, &numEdges));
    auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
    auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
    checkCudaErrors(hipGraphGetEdges(this->graph, from.get(), to.get(), &numEdges));

    std::map<hipGraphNode_t, bool> hasOutGoingEdge;
    for (int i = 0; i < numEdges; i++) {
      hasOutGoingEdge[from[i]] = true;
    }

    std::vector<hipGraphNode_t> newLeafNodes;
    for (int i = 0; i < numNodes; i++) {
      auto &node = nodes[i];
      if (!visited[node]) {
        visited[node] = true;
        if (!hasOutGoingEdge[node]) {
          newLeafNodes.push_back(node);
        }
      }
    }

    return newLeafNodes;
  }
};

Executor *Executor::instance = nullptr;

Executor *Executor::getInstance() {
  if (instance == nullptr) {
    instance = new Executor();
  }
  return instance;
}

float Executor::executeOptimizedGraph(OptimizationOutput &optimizedGraph, ExecuteRandomTask executeRandomTask) {
  LOG_TRACE_WITH_INFO("Initialize");
  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  std::queue<int> nodesToExecute;
  std::vector<int> rootNodes;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
      rootNodes.push_back(u);
    }
  }

  int storageDeviceId = ConfigurationManager::getConfig().useNvlink ? Constants::STORAGE_DEVICE_ID : hipCpuDeviceId;
  hipMemcpyKind prefetchMemcpyKind = ConfigurationManager::getConfig().useNvlink ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
  hipMemcpyKind offloadMemcpyKind = ConfigurationManager::getConfig().useNvlink ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost;

  if (ConfigurationManager::getConfig().useNvlink) {
    enablePeerAccessForNvlink(Constants::DEVICE_ID, Constants::STORAGE_DEVICE_ID);
  }

  LOG_TRACE_WITH_INFO("Initialize managed data distribution");
  for (auto ptr : MemoryManager::managedMemoryAddresses) {
    checkCudaErrors(hipMemPrefetchAsync(
      ptr,
      MemoryManager::managedMemoryAddressToSizeMap[ptr],
      storageDeviceId
    ));
  }
  checkCudaErrors(hipDeviceSynchronize());

  std::map<void *, void *> addressUpdate;
  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  LOG_TRACE_WITH_INFO("Record nodes to a new CUDA Graph");

  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  for (auto ptr : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    auto size = MemoryManager::managedMemoryAddressToSizeMap[ptr];

    void *devicePtr;
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(devicePtr, ptr, size, prefetchMemcpyKind, stream));
    addressUpdate[ptr] = devicePtr;
  }
  hipGraph_t graphForInitialDataDistribution;
  checkCudaErrors(hipStreamEndCapture(stream, &graphForInitialDataDistribution));

  hipGraphExec_t graphExecForInitialDataDistribution;
  checkCudaErrors(hipGraphInstantiate(&graphExecForInitialDataDistribution, graphForInitialDataDistribution, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExecForInitialDataDistribution, stream));
  checkCudaErrors(hipDeviceSynchronize());

  // Kahn Algorithm
  while (!nodesToExecute.empty()) {
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    std::vector<hipGraphNode_t> newLeafNodes;

    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      auto dataMovementSize = MemoryManager::managedMemoryAddressToSizeMap[dataMovement.address];
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        void *devicePtr;
        checkCudaErrors(hipMallocAsync(&devicePtr, dataMovementSize, stream));
        checkCudaErrors(hipMemcpyAsync(devicePtr, dataMovement.address, dataMovementSize, prefetchMemcpyKind, stream));
        addressUpdate[dataMovement.address] = devicePtr;
      } else {
        void *devicePtr = addressUpdate[dataMovement.address];
        checkCudaErrors(hipMemcpyAsync(dataMovement.address, devicePtr, dataMovementSize, offloadMemcpyKind, stream));
        checkCudaErrors(hipFreeAsync(devicePtr, stream));
        addressUpdate.erase(dataMovement.address);
      }
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      executeRandomTask(
        optimizedGraph.nodeIdToTaskIdMap[u],
        addressUpdate,
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;
      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].begin(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  LOG_TRACE_WITH_INFO("Printing the new CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  LOG_TRACE_WITH_INFO("Execute the new CUDA Graph");
  PeakMemoryUsageProfiler peakMemoryUsageProfiler;
  CudaEventClock cudaEventClock;
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  if (ConfigurationManager::getConfig().measurePeakMemoryUsage) {
    peakMemoryUsageProfiler.start();
  }

  cudaEventClock.start();
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  if (ConfigurationManager::getConfig().measurePeakMemoryUsage) {
    const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
    LOG_TRACE_WITH_INFO(
      "Peak memory usage (MiB): %.2f\n",
      static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
    );
  }

  LOG_TRACE_WITH_INFO("Clean up");
  for (auto &[oldAddr, newAddr] : addressUpdate) {
    checkCudaErrors(hipMemcpy(oldAddr, newAddr, MemoryManager::managedMemoryAddressToSizeMap[oldAddr], offloadMemcpyKind));
    checkCudaErrors(hipFree(newAddr));
  }
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipGraphExecDestroy(graphExecForInitialDataDistribution));
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graphForInitialDataDistribution));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  if (ConfigurationManager::getConfig().useNvlink) {
    disablePeerAccessForNvlink(Constants::DEVICE_ID, Constants::STORAGE_DEVICE_ID);
  }

  return cudaEventClock.getTimeInSeconds();
}
