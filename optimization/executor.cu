#include <cassert>
#include <memory>
#include <queue>

#include "../profiling/memoryManager.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "executor.hpp"

class OptimizedCudaGraphCreator {
 public:
  OptimizedCudaGraphCreator(hipStream_t stream, hipGraph_t graph) : stream(stream), graph(graph) {}

  void beginCaptureOperation(const std::vector<hipGraphNode_t> &dependencies) {
    checkCudaErrors(hipStreamBeginCaptureToGraph(this->stream, this->graph, dependencies.data(), nullptr, dependencies.size(), hipStreamCaptureModeGlobal));
  }

  std::vector<hipGraphNode_t> endCaptureOperation() {
    checkCudaErrors(hipStreamEndCapture(this->stream, &this->graph));
    return this->getNewLeafNodesAddedByLastCapture();
  };

  hipGraphNode_t addEmptyNode(const std::vector<hipGraphNode_t> &dependencies) {
    hipGraphNode_t newEmptyNode;
    checkCudaErrors(hipGraphAddEmptyNode(&newEmptyNode, this->graph, dependencies.data(), dependencies.size()));
    return newEmptyNode;
  }

 private:
  hipStream_t stream;
  hipGraph_t graph;
  std::vector<hipGraphNode_t> lastDependencies;
  std::map<hipGraphNode_t, bool> visited;

  std::vector<hipGraphNode_t> getNewLeafNodesAddedByLastCapture() {
    size_t numNodes;
    checkCudaErrors(hipGraphGetNodes(this->graph, nullptr, &numNodes));
    auto nodes = std::make_unique<hipGraphNode_t[]>(numNodes);
    checkCudaErrors(hipGraphGetNodes(this->graph, nodes.get(), &numNodes));

    size_t numEdges;
    checkCudaErrors(hipGraphGetEdges(this->graph, nullptr, nullptr, &numEdges));
    auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
    auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
    checkCudaErrors(hipGraphGetEdges(this->graph, from.get(), to.get(), &numEdges));

    std::map<hipGraphNode_t, bool> hasOutGoingEdge;
    for (int i = 0; i < numEdges; i++) {
      hasOutGoingEdge[from[i]] = true;
    }

    std::vector<hipGraphNode_t> newLeafNodes;
    for (int i = 0; i < numNodes; i++) {
      auto &node = nodes[i];
      if (!visited[node]) {
        visited[node] = true;
        if (!hasOutGoingEdge[node]) {
          newLeafNodes.push_back(node);
        }
      }
    }

    return newLeafNodes;
  }
};

Executor *Executor::instance = nullptr;

Executor *Executor::getInstance() {
  if (instance == nullptr) {
    instance = new Executor();
  }
  return instance;
}

float Executor::executeOptimizedGraph(OptimizationOutput &optimizedGraph, ExecuteRandomTask executeRandomTask) {
  LOG_TRACE_WITH_INFO("Initialize");
  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  std::queue<int> nodesToExecute;
  std::vector<int> rootNodes;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
      rootNodes.push_back(u);
    }
  }

  LOG_TRACE_WITH_INFO("Initialize managed data distribution");
  for (auto ptr : MemoryManager::managedMemoryAddresses) {
    checkCudaErrors(hipMemPrefetchAsync(ptr, MemoryManager::managedMemoryAddressToSizeMap[ptr], hipCpuDeviceId));
  }
  checkCudaErrors(hipDeviceSynchronize());

  std::map<void *, void *> addressUpdate;
  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  LOG_TRACE_WITH_INFO("Record nodes to a new CUDA Graph");

  optimizedCudaGraphCreator->beginCaptureOperation(std::vector<hipGraphNode_t>());
  for (const auto &[ptr, size] : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    void *devicePtr;
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(devicePtr, ptr, size, hipMemcpyHostToDevice, stream));
    addressUpdate[ptr] = devicePtr;
  }
  auto initialMemOperationLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();

  for (auto u : rootNodes) {
    nodeToDependentNodesMap[u].insert(
      nodeToDependentNodesMap[u].begin(),
      initialMemOperationLeafNodes.begin(),
      initialMemOperationLeafNodes.end()
    );
  }

  // Kahn Algorithm
  while (!nodesToExecute.empty()) {
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    std::vector<hipGraphNode_t> newLeafNodes;

    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        void *devicePtr;
        checkCudaErrors(hipMallocAsync(&devicePtr, dataMovement.size, stream));
        checkCudaErrors(hipMemcpyAsync(devicePtr, dataMovement.address, dataMovement.size, hipMemcpyHostToDevice, stream));
        addressUpdate[dataMovement.address] = devicePtr;
      } else {
        void *devicePtr = addressUpdate[dataMovement.address];
        checkCudaErrors(hipMemcpyAsync(dataMovement.address, devicePtr, dataMovement.size, hipMemcpyDeviceToHost, stream));
        checkCudaErrors(hipFreeAsync(devicePtr, stream));
        addressUpdate.erase(dataMovement.address);
      }
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      executeRandomTask(
        optimizedGraph.nodeIdToTaskIdMap[u],
        addressUpdate,
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;
      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].begin(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  LOG_TRACE_WITH_INFO("Printing the new CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  LOG_TRACE_WITH_INFO("Execute the new CUDA Graph");
  CudaEventClock cudaEventClock;
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  cudaEventClock.start();
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  LOG_TRACE_WITH_INFO("Clean up");
  for (auto &[oldAddr, newAddr] : addressUpdate) {
    checkCudaErrors(hipMemcpy(oldAddr, newAddr, MemoryManager::managedMemoryAddressToSizeMap[oldAddr], hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(newAddr));
  }
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  return cudaEventClock.getTimeInSeconds();
}
