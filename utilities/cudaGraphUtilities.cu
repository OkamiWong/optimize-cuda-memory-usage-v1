#include <hip/hip_runtime.h>

#include <cassert>
#include <map>
#include <memory>
#include <vector>

#include "cudaGraphUtilities.hpp"
#include "cudaUtilities.hpp"

namespace memopt {

void extractGraphNodesAndEdges(
  hipGraph_t graph,
  std::vector<hipGraphNode_t> &nodes,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges
) {
  size_t numNodes, numEdges;
  checkCudaErrors(hipGraphGetNodes(graph, nullptr, &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, nullptr, nullptr, &numEdges));
  auto rawNodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
  auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
  checkCudaErrors(hipGraphGetNodes(graph, rawNodes.get(), &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, from.get(), to.get(), &numEdges));

  nodes.clear();
  for (int i = 0; i < numNodes; i++) {
    nodes.push_back(rawNodes[i]);
  }

  edges.clear();
  for (int i = 0; i < numEdges; i++) {
    edges[from[i]].push_back(to[i]);
  }
}

hipGraphNode_t getRootNode(hipGraph_t graph) {
  size_t numRootNodes;
  checkCudaErrors(hipGraphGetRootNodes(graph, nullptr, &numRootNodes));
  assert(numRootNodes == 1);

  auto rootNodes = std::make_unique<hipGraphNode_t[]>(numRootNodes);
  checkCudaErrors(hipGraphGetRootNodes(graph, rootNodes.get(), &numRootNodes));
  return rootNodes[0];
}

std::vector<hipGraphNode_t> getNodesWithZeroOutDegree(hipGraph_t graph) {
  std::vector<hipGraphNode_t> nodesWithZeroOutDegree;

  std::vector<hipGraphNode_t> nodes;
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> edges;
  extractGraphNodesAndEdges(graph, nodes, edges);

  for (auto u : nodes) {
    if (edges[u].size() == 0) {
      nodesWithZeroOutDegree.push_back(u);
    }
  }

  return nodesWithZeroOutDegree;
}

void getKernelNodeParams(hipGraphNode_t kernelNode, hipKernelNodeParams &nodeParams) {
  hipGraphNodeType nodeType;
  checkCudaErrors(hipGraphNodeGetType(kernelNode, &nodeType));
  assert(nodeType == hipGraphNodeTypeKernel);

  // Why switch to driver API:
  // https://forums.developer.nvidia.com/t/cuda-runtime-api-error-for-cuda-graph-and-opencv/215408/13
  checkCudaErrors(hipGraphKernelNodeGetParams(kernelNode, &nodeParams));
}

bool compareKernelNodeFunctionHandle(hipGraphNode_t kernelNode, hipFunction_t functionHandle) {
  hipGraphNodeType nodeType;
  checkCudaErrors(hipGraphNodeGetType(kernelNode, &nodeType));
  if (nodeType == hipGraphNodeTypeKernel) {
    hipKernelNodeParams nodeParams;
    getKernelNodeParams(kernelNode, nodeParams);

    if (nodeParams.func == functionHandle) {
      return true;
    }
  }
  return false;
}

}  // namespace memopt
