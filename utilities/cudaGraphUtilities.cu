#include <cassert>
#include <map>
#include <memory>
#include <vector>

#include "cudaGraphUtilities.hpp"
#include "cudaUtilities.hpp"

void extractGraphNodesAndEdges(
  hipGraph_t graph,
  std::vector<hipGraphNode_t> &nodes,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges
) {
  size_t numNodes, numEdges;
  checkCudaErrors(hipGraphGetNodes(graph, nullptr, &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, nullptr, nullptr, &numEdges));
  auto rawNodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
  auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
  checkCudaErrors(hipGraphGetNodes(graph, rawNodes.get(), &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, from.get(), to.get(), &numEdges));

  nodes.clear();
  for (int i = 0; i < numNodes; i++) {
    nodes.push_back(rawNodes[i]);
  }

  edges.clear();
  for (int i = 0; i < numEdges; i++) {
    edges[from[i]].push_back(to[i]);
  }
}

hipGraphNode_t getRootNode(hipGraph_t graph) {
  size_t numRootNodes;
  checkCudaErrors(hipGraphGetRootNodes(graph, NULL, &numRootNodes));
  assert(numRootNodes == 1);

  auto rootNodes = std::make_unique<hipGraphNode_t[]>(numRootNodes);
  checkCudaErrors(hipGraphGetRootNodes(graph, rootNodes.get(), &numRootNodes));
  return rootNodes[0];
}
