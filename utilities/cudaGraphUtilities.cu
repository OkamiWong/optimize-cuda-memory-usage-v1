#include <cassert>
#include <map>
#include <memory>
#include <vector>

#include "cudaGraphUtilities.hpp"
#include "cudaUtilities.hpp"

void extractGraphNodesAndEdges(
  hipGraph_t graph,
  std::vector<hipGraphNode_t> &nodes,
  std::map<hipGraphNode_t, std::vector<hipGraphNode_t>> &edges
) {
  size_t numNodes, numEdges;
  checkCudaErrors(hipGraphGetNodes(graph, nullptr, &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, nullptr, nullptr, &numEdges));
  auto rawNodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
  auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
  checkCudaErrors(hipGraphGetNodes(graph, rawNodes.get(), &numNodes));
  checkCudaErrors(hipGraphGetEdges(graph, from.get(), to.get(), &numEdges));

  nodes.clear();
  for (int i = 0; i < numNodes; i++) {
    nodes.push_back(rawNodes[i]);
  }

  edges.clear();
  for (int i = 0; i < numEdges; i++) {
    edges[from[i]].push_back(to[i]);
  }
}

hipGraphNode_t getRootNode(hipGraph_t graph) {
  size_t numRootNodes;
  checkCudaErrors(hipGraphGetRootNodes(graph, nullptr, &numRootNodes));
  assert(numRootNodes == 1);

  auto rootNodes = std::make_unique<hipGraphNode_t[]>(numRootNodes);
  checkCudaErrors(hipGraphGetRootNodes(graph, rootNodes.get(), &numRootNodes));
  return rootNodes[0];
}
