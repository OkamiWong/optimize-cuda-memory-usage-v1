#include <cassert>
#include <cstdio>
#include <memory>

#include "cudaGraphExecutionTimelineProfiler.hpp"
#include "cudaUtilities.hpp"
#include "cuptiUtilities.hpp"

CudaGraphExecutionTimelineProfiler *CudaGraphExecutionTimelineProfiler::instance = nullptr;

CudaGraphExecutionTimelineProfiler *CudaGraphExecutionTimelineProfiler::getInstance() {
  if (instance == nullptr) {
    instance = new CudaGraphExecutionTimelineProfiler();
  }
  return instance;
}

void CudaGraphExecutionTimelineProfiler::consumeActivityRecord(CUpti_Activity *record) {
  switch (record->kind) {
    case CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL: {
      auto kernelActivityRecord = reinterpret_cast<CUpti_ActivityKernel9 *>(record);
      this->graphNodeIdToLifetimeMap[kernelActivityRecord->graphNodeId] =
        std::make_pair(
          static_cast<uint64_t>(kernelActivityRecord->start),
          static_cast<uint64_t>(kernelActivityRecord->end)
        );
      break;
    }
    case CUPTI_ACTIVITY_KIND_MEMSET: {
      auto memsetActivityRecord = reinterpret_cast<CUpti_ActivityMemset4 *>(record);
      this->graphNodeIdToLifetimeMap[memsetActivityRecord->graphNodeId] =
        std::make_pair(
          static_cast<uint64_t>(memsetActivityRecord->start),
          static_cast<uint64_t>(memsetActivityRecord->end)
        );
      break;
    }
    default: {
      printf("Warning: Unknown CUPTI activity (%d)\n", record->kind);
      break;
    }
  }
}

void CUPTIAPI bufferRequested(uint8_t **buffer, size_t *size, size_t *maxNumRecords) {
  auto rawBuffer = (uint8_t *)malloc(BUF_SIZE + ALIGN_SIZE);
  if (rawBuffer == nullptr) {
    printf("Error: Out of memory\n");
    exit(-1);
  }

  *size = BUF_SIZE;
  *buffer = ALIGN_BUFFER(rawBuffer, ALIGN_SIZE);
  *maxNumRecords = 0;
}

void CUPTIAPI bufferCompleted(hipCtx_t ctx, uint32_t streamId, uint8_t *buffer, size_t size, size_t validSize) {
  CUptiResult status;
  CUpti_Activity *record = nullptr;

  do {
    status = cuptiActivityGetNextRecord(buffer, validSize, &record);
    if (status == CUPTI_SUCCESS) {
      CudaGraphExecutionTimelineProfiler::getInstance()->consumeActivityRecord(record);
    } else if (status == CUPTI_ERROR_MAX_LIMIT_REACHED) {
      break;
    } else {
      CUPTI_CALL(status);
    }
  } while (1);

  // Report any records dropped from the queue
  size_t dropped;
  CUPTI_CALL(cuptiActivityGetNumDroppedRecords(ctx, streamId, &dropped));
  if (dropped != 0) {
    printf("Dropped %u activity records\n", (unsigned int)dropped);
  }

  free(buffer);
}

void CudaGraphExecutionTimelineProfiler::initialize(hipGraph_t graph) {
  CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferRequested, bufferCompleted));

  CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL));
  CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_MEMSET));

  this->finalized = false;
  this->graph = graph;
  this->graphNodeIdToLifetimeMap.clear();
}

void CudaGraphExecutionTimelineProfiler::finalize() {
  CUPTI_CALL(cuptiGetLastError());

  CUPTI_CALL(cuptiActivityFlushAll(1));

  CUPTI_CALL(cuptiActivityDisable(CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL));
  CUPTI_CALL(cuptiActivityDisable(CUPTI_ACTIVITY_KIND_MEMSET));

  this->finalized = true;
}

CudaGraphExecutionTimeline CudaGraphExecutionTimelineProfiler::getTimeline() {
  assert(this->finalized);

  size_t numNodes;
  checkCudaErrors(hipGraphGetNodes(this->graph, nullptr, &numNodes));
  auto nodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  checkCudaErrors(hipGraphGetNodes(this->graph, nodes.get(), &numNodes));

  CudaGraphExecutionTimeline timeline;

  uint64_t tempNodeId;
  for (int i = 0; i < numNodes; i++) {
    CUPTI_CALL(cuptiGetGraphNodeId(nodes[i], &tempNodeId));
    timeline[nodes[i]] = this->graphNodeIdToLifetimeMap[tempNodeId];
  }

  return timeline;
}
