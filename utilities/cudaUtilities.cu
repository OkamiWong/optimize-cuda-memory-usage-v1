#include "hip/hip_runtime.h"
#include "cudaUtilities.hpp"

__global__ void warmUp() {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + static_cast<float>(tid);
}

void warmUpCudaDevice() {
  warmUp<<<32, 32>>>();
  hipDeviceSynchronize();
}

void initializeCudaDevice(bool displayDeviceInfo) {
  checkCudaErrors(hipSetDevice(CudaConstants::DEVICE_ID));

  if (displayDeviceInfo) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, CudaConstants::DEVICE_ID));
    printf("GPU Device %d: %s\n", CudaConstants::DEVICE_ID, deviceProp.name);
    printf("Compute Capability: %d.%d\n\n", deviceProp.major, deviceProp.minor);
  }

  warmUpCudaDevice();
}

CudaEventClock::CudaEventClock() {
  checkCudaErrors(hipEventCreate(&this->startEvent));
  checkCudaErrors(hipEventCreate(&this->endEvent));
}

CudaEventClock::~CudaEventClock() {
  checkCudaErrors(hipEventDestroy(this->startEvent));
  checkCudaErrors(hipEventDestroy(this->endEvent));
}

void CudaEventClock::start(hipStream_t stream) {
  checkCudaErrors(hipEventRecord(this->startEvent, stream));
}

void CudaEventClock::end(hipStream_t stream) {
  checkCudaErrors(hipEventRecord(this->endEvent, stream));
}

float CudaEventClock::getTimeInSeconds() {
  float time;
  checkCudaErrors(hipEventElapsedTime(&time, this->startEvent, this->endEvent));
  return time * 1e-3f;
}
