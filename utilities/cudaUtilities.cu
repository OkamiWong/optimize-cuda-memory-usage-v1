#include "hip/hip_runtime.h"
#include <cassert>

#include "cudaUtilities.hpp"

namespace memopt {

__global__ void warmUp() {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + static_cast<float>(tid);
}

void warmUpCudaDevice() {
  warmUp<<<32, 32>>>();
  hipDeviceSynchronize();
}

void initializeCudaDevice(bool displayDeviceInfo) {
  checkCudaErrors(hipSetDevice(Constants::DEVICE_ID));

  if (displayDeviceInfo) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, Constants::DEVICE_ID));
    printf("GPU Device %d: %s\n", Constants::DEVICE_ID, deviceProp.name);
    printf("Compute Capability: %d.%d\n\n", deviceProp.major, deviceProp.minor);
  }

  warmUpCudaDevice();
}

void enablePeerAccessForNvlink(int deviceA, int deviceB) {
  int currentDevice;
  checkCudaErrors(hipGetDevice(&currentDevice));

  int canAccessPeerAToB, canAccessPeerBToA;
  checkCudaErrors(hipDeviceCanAccessPeer(&canAccessPeerAToB, deviceA, deviceB));
  checkCudaErrors(hipDeviceCanAccessPeer(&canAccessPeerBToA, deviceB, deviceA));

  assert(canAccessPeerAToB);
  assert(canAccessPeerBToA);

  checkCudaErrors(hipSetDevice(deviceA));
  checkCudaErrors(hipDeviceEnablePeerAccess(deviceB, 0));
  checkCudaErrors(hipSetDevice(deviceB));
  checkCudaErrors(hipDeviceEnablePeerAccess(deviceA, 0));

  checkCudaErrors(hipSetDevice(currentDevice));
}

void disablePeerAccessForNvlink(int deviceA, int deviceB) {
  int currentDevice;
  checkCudaErrors(hipGetDevice(&currentDevice));

  checkCudaErrors(hipSetDevice(deviceA));
  checkCudaErrors(hipDeviceDisablePeerAccess(deviceB));
  checkCudaErrors(hipSetDevice(deviceB));
  checkCudaErrors(hipDeviceDisablePeerAccess(deviceA));

  checkCudaErrors(hipSetDevice(currentDevice));
}

CudaEventClock::CudaEventClock() {
  checkCudaErrors(hipEventCreate(&this->startEvent));
  checkCudaErrors(hipEventCreate(&this->endEvent));
}

CudaEventClock::~CudaEventClock() {
  checkCudaErrors(hipEventDestroy(this->startEvent));
  checkCudaErrors(hipEventDestroy(this->endEvent));
}

void CudaEventClock::start(hipStream_t stream) {
  checkCudaErrors(hipEventRecord(this->startEvent, stream));
}

void CudaEventClock::end(hipStream_t stream) {
  checkCudaErrors(hipEventRecord(this->endEvent, stream));
}

float CudaEventClock::getTimeInSeconds() {
  float time;
  checkCudaErrors(hipEventElapsedTime(&time, this->startEvent, this->endEvent));
  return time * 1e-3f;
}

}  // namespace memopt
