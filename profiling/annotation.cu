#include "hip/hip_runtime.h"
#include <cstring>

#include "annotation.hpp"

__global__ void dummyKernelForAnnotation(KernelIO io) {
  return;
}

__host__ void annotateNextKernel(
  std::initializer_list<void *> inputs,
  std::initializer_list<void *> outputs,
  hipStream_t stream
) {
  KernelIO io;
  memset(io.inputs, 0, KernelIO::MAX_NUM_PTR * sizeof(void *));
  memset(io.outputs, 0, KernelIO::MAX_NUM_PTR * sizeof(void *));
  memcpy(io.inputs, std::data(inputs), inputs.size() * sizeof(void *));
  memcpy(io.outputs, std::data(outputs), outputs.size() * sizeof(void *));
  dummyKernelForAnnotation<<<1, 1, 0, stream>>>(io);
}
