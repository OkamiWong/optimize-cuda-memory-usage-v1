#include "hip/hip_runtime.h"
#include <chrono>

#include "../utilities/configurationManager.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "peakMemoryUsageProfiler.hpp"

namespace memopt {

PeakMemoryUsageProfiler::PeakMemoryUsageProfiler(int sampleIntervalMilliseconds)
    : sampleIntervalMilliseconds(sampleIntervalMilliseconds) {}

void PeakMemoryUsageProfiler::periodicallyCheckMemoryUsage() {
  checkCudaErrors(hipSetDevice(ConfigurationManager::getConfig().mainDeviceId));

  size_t peakMemoryUsage = 0;

  size_t free, total;
  while (!this->stopFlag) {
    checkCudaErrors(hipMemGetInfo(&free, &total));
    peakMemoryUsage = std::max(peakMemoryUsage, total - free);
    std::this_thread::sleep_for(std::chrono::milliseconds(this->sampleIntervalMilliseconds));
  }

  this->peakMemoryUsagePromise.set_value(peakMemoryUsage);
}

void PeakMemoryUsageProfiler::start() {
  this->stopFlag = false;
  this->peakMemoryUsagePromise = std::promise<size_t>();
  this->monitorThread = std::thread(&PeakMemoryUsageProfiler::periodicallyCheckMemoryUsage, this);
}

size_t PeakMemoryUsageProfiler::end() {
  this->stopFlag = true;
  this->monitorThread.join();
  return this->peakMemoryUsagePromise.get_future().get();
}

}  // namespace memopt
