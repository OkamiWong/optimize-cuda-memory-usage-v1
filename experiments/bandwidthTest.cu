#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <limits>
#include <memory>
#include <sstream>
#include <string>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "../include/csv.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/utilities.hpp"

constexpr int REPETITION = 100;
constexpr int PCIE_DEVICE_ID = 0;
constexpr int NVLINK_DEVICE_ID_A = 1;
constexpr int NVLINK_DEVICE_ID_B = 2;

void printHeader() {
  std::stringstream ss;
  auto csvWriter = csv::make_csv_writer(ss);
  csvWriter << std::make_tuple("kind", "size(Byte)", "time(s)", "speed(GB/s)");
  fputs(ss.str().c_str(), stdout);
}

void printDataOfTheSameKind(const std::string &kind, const std::vector<size_t> &sizes, const std::vector<float> &times) {
  std::stringstream ss;
  auto csvWriter = csv::make_csv_writer(ss);
  for (int i = 0; i < sizes.size(); i++) {
    csvWriter << std::make_tuple(
      kind,
      sizes[i],
      toStringWithPrecision(times[i], 6),
      toStringWithPrecision(static_cast<float>(sizes[i]) / times[i] / 1e9, 3)
    );
  }
  fputs(ss.str().c_str(), stdout);
}

void enablePeerAccessForNvlink() {
  int canAccessPeerAToB, canAccessPeerBToA;
  checkCudaErrors(hipDeviceCanAccessPeer(&canAccessPeerAToB, NVLINK_DEVICE_ID_A, NVLINK_DEVICE_ID_B));
  checkCudaErrors(hipDeviceCanAccessPeer(&canAccessPeerBToA, NVLINK_DEVICE_ID_B, NVLINK_DEVICE_ID_A));

  assert(canAccessPeerAToB);
  assert(canAccessPeerBToA);

  checkCudaErrors(hipSetDevice(NVLINK_DEVICE_ID_A));
  checkCudaErrors(hipDeviceEnablePeerAccess(NVLINK_DEVICE_ID_B, 0));
  checkCudaErrors(hipSetDevice(NVLINK_DEVICE_ID_B));
  checkCudaErrors(hipDeviceEnablePeerAccess(NVLINK_DEVICE_ID_A, 0));
}

void disablePeerAccessForNvlink() {
  checkCudaErrors(hipSetDevice(NVLINK_DEVICE_ID_A));
  checkCudaErrors(hipDeviceDisablePeerAccess(NVLINK_DEVICE_ID_B));
  checkCudaErrors(hipSetDevice(NVLINK_DEVICE_ID_B));
  checkCudaErrors(hipDeviceDisablePeerAccess(NVLINK_DEVICE_ID_A));
}

template <typename T>
__global__ void initializeArrayKernel(T *array, T initialValue, int count) {
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < count) {
    array[i] = initialValue;
  }
}

void testNvlinkBandwidth(const std::vector<size_t> &sizes, bool useUnifiedMemory, bool noHeader) {
  enablePeerAccessForNvlink();

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  std::vector<float> deviceToDeviceTimes;

  for (auto size : sizes) {
    float minDeviceToDeviceTime = std::numeric_limits<float>::max();
    if (useUnifiedMemory) {
      // TODO
    } else {
      int *arrayOnDeviceA;
      checkCudaErrors(hipSetDevice(NVLINK_DEVICE_ID_A));
      hipMalloc(&arrayOnDeviceA, size);
      initializeArrayKernel<<<(size / 512) + 1, 512>>>(arrayOnDeviceA, 0, size / sizeof(int));
      checkCudaErrors(hipDeviceSynchronize());

      int *arrayOnDeviceB;
      checkCudaErrors(hipSetDevice(NVLINK_DEVICE_ID_B));
      hipMalloc(&arrayOnDeviceB, size);
      initializeArrayKernel<<<(size / 512) + 1, 512>>>(arrayOnDeviceB, 0, size / sizeof(int));
      checkCudaErrors(hipDeviceSynchronize());

      CudaEventClock clock;
      for (int i = 0; i < REPETITION; i++) {
        clock.start(stream);
        checkCudaErrors(hipMemcpyAsync(arrayOnDeviceA, arrayOnDeviceB, size, hipMemcpyDeviceToDevice, stream));
        clock.end(stream);
        checkCudaErrors(hipStreamSynchronize(stream));
        minDeviceToDeviceTime = std::min(minDeviceToDeviceTime, clock.getTimeInSeconds());
      }

      checkCudaErrors(hipFree(arrayOnDeviceA));
      checkCudaErrors(hipFree(arrayOnDeviceB));

      deviceToDeviceTimes.push_back(minDeviceToDeviceTime);
    }
  }

  checkCudaErrors(hipStreamDestroy(stream));

  disablePeerAccessForNvlink();

  if (!noHeader) {
    printHeader();
  }

  if (useUnifiedMemory) {
    printDataOfTheSameKind("NVLink-DeviceToDevice-UnifiedMemory", sizes, deviceToDeviceTimes);
  } else {
    printDataOfTheSameKind("NVLink-DeviceToDevice", sizes, deviceToDeviceTimes);
  }
}

void testPcieBandwidth(const std::vector<size_t> &sizes, bool useUnifiedMemory, bool noHeader) {
  std::vector<float> hostToDeviceTimes, deviceToHostTimes;

  warmUpCudaDevice();

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  for (auto size : sizes) {
    float minHostToDeviceTime = std::numeric_limits<float>::max();
    float minDeviceToHostTime = std::numeric_limits<float>::max();

    if (useUnifiedMemory) {
      int *array;
      checkCudaErrors(hipMallocManaged(&array, size));
      memset(array, 0, size);

      CudaEventClock clock;

      for (int i = 0; i < REPETITION; i++) {
        clock.start(stream);
        checkCudaErrors(hipMemPrefetchAsync(array, size, PCIE_DEVICE_ID, stream));
        clock.end(stream);
        checkCudaErrors(hipStreamSynchronize(stream));
        minHostToDeviceTime = std::min(minHostToDeviceTime, clock.getTimeInSeconds());

        clock.start(stream);
        checkCudaErrors(hipMemPrefetchAsync(array, size, hipCpuDeviceId));
        clock.end(stream);
        checkCudaErrors(hipStreamSynchronize(stream));
        minDeviceToHostTime = std::min(minDeviceToHostTime, clock.getTimeInSeconds());
      }

      checkCudaErrors(hipFree(array));

      hostToDeviceTimes.push_back(minHostToDeviceTime);
      deviceToHostTimes.push_back(minDeviceToHostTime);
    } else {
      int *hostArray;
      checkCudaErrors(hipHostMalloc(&hostArray, size));
      memset(hostArray, 0, size);

      int *deviceArray;
      checkCudaErrors(hipMalloc(&deviceArray, size));

      CudaEventClock clock;
      for (int i = 0; i < REPETITION; i++) {
        clock.start(stream);
        checkCudaErrors(hipMemcpyAsync(deviceArray, hostArray, size, hipMemcpyHostToDevice, stream));
        clock.end(stream);
        checkCudaErrors(hipStreamSynchronize(stream));
        minHostToDeviceTime = std::min(minHostToDeviceTime, clock.getTimeInSeconds());

        clock.start(stream);
        checkCudaErrors(hipMemcpyAsync(hostArray, deviceArray, size, hipMemcpyDeviceToHost, stream));
        clock.end(stream);
        checkCudaErrors(hipStreamSynchronize(stream));
        minDeviceToHostTime = std::min(minDeviceToHostTime, clock.getTimeInSeconds());
      }

      checkCudaErrors(hipHostFree(hostArray));
      checkCudaErrors(hipFree(deviceArray));

      hostToDeviceTimes.push_back(minHostToDeviceTime);
      deviceToHostTimes.push_back(minDeviceToHostTime);
    }
  }

  checkCudaErrors(hipStreamDestroy(stream));

  if (!noHeader) {
    printHeader();
  }

  if (useUnifiedMemory) {
    printDataOfTheSameKind("PCIe-HostToDevice-UnifiedMemory", sizes, hostToDeviceTimes);
    printDataOfTheSameKind("PCIe-DeviceToHost-UnifiedMemory", sizes, deviceToHostTimes);
  } else {
    printDataOfTheSameKind("PCIe-HostToDevice", sizes, hostToDeviceTimes);
    printDataOfTheSameKind("PCIe-DeviceToHost", sizes, deviceToHostTimes);
  }
}

int main(int argc, char **argv) {
  auto cmdl = argh::parser(argc, argv);

  bool useNvlink = cmdl["use-nvlink"];
  bool useUnifiedMemory = cmdl["use-unified-memory"];
  bool useLogarithmicScale = cmdl["use-log-scale"];
  bool noHeader = cmdl["no-header"];

  size_t startSize, endSize, stepSize;
  cmdl("start-size", 10'000'000ull) >> startSize;  // 10 MB
  cmdl("end-size", 100'000'000ull) >> endSize;     // 100 MB
  cmdl("step-size", 10'000'000ull) >> stepSize;    // 10 MB

  std::vector<size_t> sizes;
  size_t s = startSize;
  while (s <= endSize) {
    sizes.push_back(s);
    if (useLogarithmicScale) {
      s *= stepSize;
    } else {
      s += stepSize;
    }
  }

  if (useNvlink) {
    testNvlinkBandwidth(sizes, useUnifiedMemory, noHeader);
  } else {
    testPcieBandwidth(sizes, useUnifiedMemory, noHeader);
  }

  return 0;
}