#include "hip/hip_runtime.h"
#include <algorithm>
#include <limits>
#include <memory>
#include <sstream>
#include <string>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "../include/csv.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/utilities.hpp"

constexpr int REPETITION = 100;
constexpr int DEVICE_0_ID = 0;
constexpr int DEVICE_1_ID = 1;

void printHeader() {
  std::stringstream ss;
  auto csvWriter = csv::make_csv_writer(ss);
  csvWriter << std::make_tuple("kind", "size(Byte)", "time(s)", "speed(GB/s)");
  fputs(ss.str().c_str(), stdout);
}

void printDataOfTheSameKind(const std::string &kind, const std::vector<size_t> &sizes, const std::vector<float> &times) {
  std::stringstream ss;
  auto csvWriter = csv::make_csv_writer(ss);
  for (int i = 0; i < sizes.size(); i++) {
    csvWriter << std::make_tuple(
      kind,
      sizes[i],
      toStringWithPrecision(times[i], 6),
      toStringWithPrecision(static_cast<float>(sizes[i]) / times[i] / 1e9, 3)
    );
  }
  fputs(ss.str().c_str(), stdout);
}

void testNvlinkBandwidth(const std::vector<size_t> &sizes) {
  // TODO
}

void testPcieBandwidth(const std::vector<size_t> &sizes, bool useUnifiedMemory, bool noHeader) {
  std::vector<float> hostToDeviceTimes, deviceToHostTimes;

  warmUpCudaDevice();

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  for (auto size : sizes) {
    float minHostToDeviceTime = std::numeric_limits<float>::max();
    float minDeviceToHostTime = std::numeric_limits<float>::max();

    if (useUnifiedMemory) {
      int *array;
      checkCudaErrors(hipMallocManaged(&array, size));
      memset(array, 0, size);

      CudaEventClock clock;

      for (int i = 0; i < REPETITION; i++) {
        clock.start(stream);
        checkCudaErrors(hipMemPrefetchAsync(array, size, DEVICE_0_ID, stream));
        clock.end(stream);
        checkCudaErrors(hipStreamSynchronize(stream));
        minHostToDeviceTime = std::min(minHostToDeviceTime, clock.getTimeInSeconds());

        clock.start(stream);
        checkCudaErrors(hipMemPrefetchAsync(array, size, hipCpuDeviceId));
        clock.end(stream);
        checkCudaErrors(hipStreamSynchronize(stream));
        minDeviceToHostTime = std::min(minDeviceToHostTime, clock.getTimeInSeconds());
      }

      checkCudaErrors(hipFree(array));

      hostToDeviceTimes.push_back(minHostToDeviceTime);
      deviceToHostTimes.push_back(minDeviceToHostTime);
    } else {
      int *hostArray;
      checkCudaErrors(hipHostMalloc(&hostArray, size));
      memset(hostArray, 0, size);

      int *deviceArray;
      checkCudaErrors(hipMalloc(&deviceArray, size));

      CudaEventClock clock;
      for (int i = 0; i < REPETITION; i++) {
        clock.start(stream);
        checkCudaErrors(hipMemcpyAsync(deviceArray, hostArray, size, hipMemcpyHostToDevice, stream));
        clock.end(stream);
        checkCudaErrors(hipStreamSynchronize(stream));
        minHostToDeviceTime = std::min(minHostToDeviceTime, clock.getTimeInSeconds());

        clock.start(stream);
        checkCudaErrors(hipMemcpyAsync(hostArray, deviceArray, size, hipMemcpyDeviceToHost, stream));
        clock.end(stream);
        checkCudaErrors(hipStreamSynchronize(stream));
        minDeviceToHostTime = std::min(minDeviceToHostTime, clock.getTimeInSeconds());
      }

      checkCudaErrors(hipHostFree(hostArray));
      checkCudaErrors(hipFree(deviceArray));

      hostToDeviceTimes.push_back(minHostToDeviceTime);
      deviceToHostTimes.push_back(minDeviceToHostTime);
    }
  }

  checkCudaErrors(hipStreamDestroy(stream));

  if (!noHeader) {
    printHeader();
  }
  printDataOfTheSameKind("PCIe-HostToDevice", sizes, hostToDeviceTimes);
  printDataOfTheSameKind("PCIe-DeviceToHost", sizes, deviceToHostTimes);
}

int main(int argc, char **argv) {
  auto cmdl = argh::parser(argc, argv);

  bool useNvlink = cmdl["use-nvlink"];
  bool useUnifiedMemory = cmdl["use-unified-memory"];
  bool useLogarithmicScale = cmdl["use-log-scale"];
  bool noHeader = cmdl["no-header"];

  size_t startSize, endSize, stepSize;
  cmdl("start-size", 100'000'000ull) >> startSize;  // 100 MB
  cmdl("end-size", 1'000'000'000ull) >> endSize;     // 1 GB
  cmdl("step-size", 100'000'000ull) >> stepSize;    // 100 MB

  std::vector<size_t> sizes;
  size_t s = startSize;
  while (s <= endSize) {
    sizes.push_back(s);
    if (useLogarithmicScale) {
      s *= stepSize;
    } else {
      s += stepSize;
    }
  }

  if (useNvlink) {
    testNvlinkBandwidth(sizes);
  } else {
    testPcieBandwidth(sizes, useUnifiedMemory, noHeader);
  }

  return 0;
}