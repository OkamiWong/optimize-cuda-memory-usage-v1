#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <limits>
#include <memory>
#include <sstream>
#include <string>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "../include/csv.hpp"
#include "memopt.hpp"

using namespace memopt;

constexpr int COMPUTE_DEVICE_ID = 0;
constexpr int STORAGE_DEVICE_ID = hipCpuDeviceId;

template <typename T>
__global__ void initializeArrayKernel(T *array, T initialValue, size_t count) {
  const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < count) {
    array[i] = initialValue;
  }
}

#define TBSIZE 1024
#define DOT_NUM_BLOCKS 256

template <class T>
__global__ void dot_kernel(const T *a, const T *b, T *sum, int array_size) {
  __shared__ T tb_sum[TBSIZE];

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  const size_t local_i = threadIdx.x;

  tb_sum[local_i] = 0.0;
  for (; i < array_size; i += blockDim.x * gridDim.x)
    tb_sum[local_i] += a[i] * b[i];

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
    __syncthreads();
    if (local_i < offset) {
      tb_sum[local_i] += tb_sum[local_i + offset];
    }
  }

  if (local_i == 0)
    sum[blockIdx.x] = tb_sum[local_i];
}

void warmUpDataMovement(int deviceA, int deviceB) {
  const size_t ARRAY_SIZE = 1ull << 30;
  const size_t ARRAY_LENGTH = ARRAY_SIZE / sizeof(int);

  int *arrayOnA;
  if (deviceA == hipCpuDeviceId) {
    checkCudaErrors(hipHostMalloc(&arrayOnA, ARRAY_SIZE));
    memset(arrayOnA, 0, ARRAY_SIZE);
  } else {
    checkCudaErrors(hipSetDevice(deviceA));
    checkCudaErrors(hipMalloc(&arrayOnA, ARRAY_SIZE));
    initializeArrayKernel<<<ARRAY_LENGTH / 1024, 1024>>>(arrayOnA, 0, ARRAY_LENGTH);
  }

  int *arrayOnB;
  if (deviceB == hipCpuDeviceId) {
    checkCudaErrors(hipHostMalloc(&arrayOnB, ARRAY_SIZE));
  } else {
    checkCudaErrors(hipSetDevice(deviceB));
    checkCudaErrors(hipMalloc(&arrayOnB, ARRAY_SIZE));
    initializeArrayKernel<<<ARRAY_LENGTH / 1024, 1024>>>(arrayOnB, 0, ARRAY_LENGTH);
  }

  checkCudaErrors(hipMemcpy(arrayOnA, arrayOnB, ARRAY_SIZE, hipMemcpyDefault));
  checkCudaErrors(hipDeviceSynchronize());

  if (deviceA == hipCpuDeviceId) {
    hipHostFree(arrayOnA);
  } else {
    checkCudaErrors(hipFree(arrayOnA));
  }

  if (deviceB == hipCpuDeviceId) {
    hipHostFree(arrayOnB);
  } else {
    checkCudaErrors(hipFree(arrayOnB));
  }
}

void runOptimizedStream(size_t arraySize, int numberOfKernels, int prefetchCycleLength) {
  const size_t arrayLength = arraySize / sizeof(float);
  constexpr size_t BLOCK_SIZE = 1024;
  const size_t GRID_SIZE = arrayLength / BLOCK_SIZE;

  assert(arrayLength % BLOCK_SIZE == 0ull);

  warmUpDataMovement(STORAGE_DEVICE_ID, COMPUTE_DEVICE_ID);
  warmUpDataMovement(COMPUTE_DEVICE_ID, STORAGE_DEVICE_ID);

  checkCudaErrors(hipSetDevice(COMPUTE_DEVICE_ID));

  // Initialize data
  auto aOnComputeDevice = std::make_unique<float *[]>(numberOfKernels);
  auto bOnComputeDevice = std::make_unique<float *[]>(numberOfKernels);
  auto cOnComputeDevice = std::make_unique<float *[]>(numberOfKernels);

  auto aOnStorageDevice = std::make_unique<float *[]>(numberOfKernels);
  auto bOnStorageDevice = std::make_unique<float *[]>(numberOfKernels);

  checkCudaErrors(hipSetDevice(COMPUTE_DEVICE_ID));
  hipStream_t computeStream, dataMovementStream;
  checkCudaErrors(hipStreamCreate(&computeStream));
  checkCudaErrors(hipStreamCreate(&dataMovementStream));

  for (int i = 0; i < numberOfKernels; i++) {
    if (i != 1 && i % prefetchCycleLength == 1) {
      LOG_TRACE_WITH_INFO("Kernel %d is prefetched", i);

      checkCudaErrors(hipHostMalloc(&aOnStorageDevice[i], arraySize));
      checkCudaErrors(hipHostMalloc(&bOnStorageDevice[i], arraySize));
      memset(aOnStorageDevice[i], 0, arraySize);
      memset(bOnStorageDevice[i], 0, arraySize);
    } else {
      checkCudaErrors(hipMallocAsync(&aOnComputeDevice[i], arraySize, dataMovementStream));
      checkCudaErrors(hipMallocAsync(&bOnComputeDevice[i], arraySize, dataMovementStream));
      initializeArrayKernel<<<GRID_SIZE, BLOCK_SIZE, 0, dataMovementStream>>>(aOnComputeDevice[i], (float)0, arrayLength);
      initializeArrayKernel<<<GRID_SIZE, BLOCK_SIZE, 0, dataMovementStream>>>(bOnComputeDevice[i], (float)0, arrayLength);
      checkCudaErrors(hipDeviceSynchronize());
    }

    checkCudaErrors(hipMalloc(&cOnComputeDevice[i], DOT_NUM_BLOCKS * sizeof(float)));
    checkCudaErrors(hipMemset(cOnComputeDevice[i], 0, DOT_NUM_BLOCKS * sizeof(float)));
  }

  // Compute
  auto prefetchEvents = std::make_unique<hipEvent_t[]>(numberOfKernels / prefetchCycleLength);
  for (int i = 0; i < numberOfKernels / prefetchCycleLength; i++) {
    checkCudaErrors(hipEventCreate(&prefetchEvents[i]));
  }

  hipEvent_t endOfKernelEvent;
  checkCudaErrors(hipEventCreate(&endOfKernelEvent));

  CudaEventClock clock;
  clock.start(computeStream);

  for (int i = 0; i < numberOfKernels; i++) {
    if (i % prefetchCycleLength == 1) {
      if (i + prefetchCycleLength < numberOfKernels) {
        checkCudaErrors(hipEventRecord(endOfKernelEvent, computeStream));
        checkCudaErrors(hipStreamWaitEvent(dataMovementStream, endOfKernelEvent));

        checkCudaErrors(hipMallocAsync(&aOnComputeDevice[i + prefetchCycleLength], arraySize, dataMovementStream));
        checkCudaErrors(hipMallocAsync(&bOnComputeDevice[i + prefetchCycleLength], arraySize, dataMovementStream));
        checkCudaErrors(hipMemcpyAsync(aOnComputeDevice[i + prefetchCycleLength], aOnStorageDevice[i + prefetchCycleLength], arraySize, hipMemcpyDefault, dataMovementStream));
        checkCudaErrors(hipMemcpyAsync(bOnComputeDevice[i + prefetchCycleLength], bOnStorageDevice[i + prefetchCycleLength], arraySize, hipMemcpyDefault, dataMovementStream));
        checkCudaErrors(hipEventRecord(prefetchEvents[(i - 1) / prefetchCycleLength], dataMovementStream));
      }
      if (i != 1) {
        checkCudaErrors(hipStreamWaitEvent(computeStream, prefetchEvents[(i - 1) / prefetchCycleLength - 1]));
      }
    }

    dot_kernel<<<DOT_NUM_BLOCKS, TBSIZE, 0, computeStream>>>(aOnComputeDevice[i], bOnComputeDevice[i], cOnComputeDevice[i], arraySize / sizeof(float));
    checkCudaErrors(hipFreeAsync(aOnComputeDevice[i], computeStream));
    checkCudaErrors(hipFreeAsync(bOnComputeDevice[i], computeStream));
  }

  clock.end(computeStream);
  checkCudaErrors(hipStreamSynchronize(computeStream));

  const float runningTime = clock.getTimeInSeconds();
  const float bandwidth = static_cast<float>(arraySize) * 2.0 * numberOfKernels / 1e9 / runningTime;
  LOG_TRACE_WITH_INFO("Total running time (s): %.6f", runningTime);
  LOG_TRACE_WITH_INFO("Bandwidth (GB/s): %.2f", bandwidth);

  for (int i = 0; i < numberOfKernels / prefetchCycleLength; i++) {
    checkCudaErrors(hipEventDestroy(prefetchEvents[i]));
  }

  checkCudaErrors(hipEventDestroy(endOfKernelEvent));

  checkCudaErrors(hipStreamDestroy(computeStream));
  checkCudaErrors(hipStreamDestroy(dataMovementStream));
}

int main(int argc, char **argv) {
  auto cmdl = argh::parser(argc, argv);

  size_t arraySize;
  cmdl("array-size", 1'073'741'824ull) >> arraySize;  // 1GiB by default

  int numberOfKernels;
  cmdl("number-of-kernels", 22) >> numberOfKernels;  // 22 kernels in total by default: 1th kernel, 2nd kernel, ..., 22th kernel.

  int prefetchCycleLength;
  cmdl("prefetch-cycle-length", 4) >> prefetchCycleLength;  // Prefetch the 6th, 10th,..., 22nd kernels by default

  runOptimizedStream(arraySize, numberOfKernels, prefetchCycleLength);

  return 0;
}
