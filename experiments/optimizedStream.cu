#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <limits>
#include <memory>
#include <sstream>
#include <string>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "../include/csv.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/logger.hpp"
#include "../utilities/utilities.hpp"

constexpr int NVLINK_DEVICE_ID_A = 1;
constexpr int NVLINK_DEVICE_ID_B = 2;

void enablePeerAccessForNvlink() {
  int canAccessPeerAToB, canAccessPeerBToA;
  checkCudaErrors(hipDeviceCanAccessPeer(&canAccessPeerAToB, NVLINK_DEVICE_ID_A, NVLINK_DEVICE_ID_B));
  checkCudaErrors(hipDeviceCanAccessPeer(&canAccessPeerBToA, NVLINK_DEVICE_ID_B, NVLINK_DEVICE_ID_A));

  assert(canAccessPeerAToB);
  assert(canAccessPeerBToA);

  checkCudaErrors(hipSetDevice(NVLINK_DEVICE_ID_A));
  checkCudaErrors(hipDeviceEnablePeerAccess(NVLINK_DEVICE_ID_B, 0));
  checkCudaErrors(hipSetDevice(NVLINK_DEVICE_ID_B));
  checkCudaErrors(hipDeviceEnablePeerAccess(NVLINK_DEVICE_ID_A, 0));
}

void disablePeerAccessForNvlink() {
  checkCudaErrors(hipSetDevice(NVLINK_DEVICE_ID_A));
  checkCudaErrors(hipDeviceDisablePeerAccess(NVLINK_DEVICE_ID_B));
  checkCudaErrors(hipSetDevice(NVLINK_DEVICE_ID_B));
  checkCudaErrors(hipDeviceDisablePeerAccess(NVLINK_DEVICE_ID_A));
}

template <typename T>
__global__ void initializeArrayKernel(T *array, T initialValue, size_t count) {
  const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < count) {
    array[i] = initialValue;
  }
}

template <typename T>
__global__ void addKernel(const T *a, const T *b, T *c) {
  const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i] + b[i];
}

template <typename T>
__global__ void checkResultKernel(const T *c, const T expectedValue) {
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (c[i] != expectedValue) {
    printf("[checkResultKernel] found c[%d] = %f, while expectedValue = %f\n", i, c[i], expectedValue);
  }
}

void warmUpDevice(const int deviceId) {
  checkCudaErrors(hipSetDevice(deviceId));
  constexpr size_t WARMUP_ARRAY_SIZE = 1024ull * 1024 * 1024;  // 1GiB
  constexpr size_t WARMUP_ARRAY_LENGTH = WARMUP_ARRAY_SIZE / sizeof(float);
  constexpr size_t BLOCK_SIZE = 1024;
  constexpr size_t GRID_SIZE = WARMUP_ARRAY_LENGTH / BLOCK_SIZE;

  constexpr float initA = 1;
  constexpr float initB = 2;
  constexpr float expectedC = initA + initB;

  float *a, *b, *c;
  checkCudaErrors(hipMalloc(&a, WARMUP_ARRAY_SIZE));
  checkCudaErrors(hipMalloc(&b, WARMUP_ARRAY_SIZE));
  checkCudaErrors(hipMalloc(&c, WARMUP_ARRAY_SIZE));

  initializeArrayKernel<<<GRID_SIZE, BLOCK_SIZE>>>(a, initA, WARMUP_ARRAY_SIZE);
  initializeArrayKernel<<<GRID_SIZE, BLOCK_SIZE>>>(b, initB, WARMUP_ARRAY_SIZE);

  addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c);

  checkResultKernel<<<GRID_SIZE, BLOCK_SIZE>>>(c, expectedC);

  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipFree(a));
  checkCudaErrors(hipFree(b));
  checkCudaErrors(hipFree(c));

  checkCudaErrors(hipDeviceSynchronize());
}

void runOptimizedStreamWithNvlink(size_t arraySize, int numberOfKernels, int prefetchCycleLength) {
  const size_t arrayLength = arraySize / sizeof(float);
  constexpr size_t BLOCK_SIZE = 1024;
  const size_t GRID_SIZE = arrayLength / BLOCK_SIZE;

  assert(arrayLength % BLOCK_SIZE == 0ull);

  constexpr int COMPUTE_DEVICE_ID = NVLINK_DEVICE_ID_A;
  constexpr int STORAGE_DEVICE_ID = NVLINK_DEVICE_ID_B;

  constexpr float initA = 1;
  constexpr float initB = 2;
  constexpr float expectedC = initA + initB;

  enablePeerAccessForNvlink();

  warmUpDevice(COMPUTE_DEVICE_ID);
  warmUpDevice(STORAGE_DEVICE_ID);

  // Initialize data
  auto aOnComputeDevice = std::make_unique<float *[]>(numberOfKernels);
  auto bOnComputeDevice = std::make_unique<float *[]>(numberOfKernels);
  auto cOnComputeDevice = std::make_unique<float *[]>(numberOfKernels);

  auto aOnStorageDevice = std::make_unique<float *[]>(numberOfKernels);
  auto bOnStorageDevice = std::make_unique<float *[]>(numberOfKernels);

  checkCudaErrors(hipSetDevice(COMPUTE_DEVICE_ID));
  hipStream_t computeStream, dataMovementStream;
  checkCudaErrors(hipStreamCreate(&computeStream));
  checkCudaErrors(hipStreamCreate(&dataMovementStream));

  for (int i = 0; i < numberOfKernels; i++) {
    if (i != 1 && i % prefetchCycleLength == 1) {
      checkCudaErrors(hipSetDevice(STORAGE_DEVICE_ID));
      checkCudaErrors(hipMalloc(&aOnStorageDevice[i], arraySize));
      checkCudaErrors(hipMalloc(&bOnStorageDevice[i], arraySize));
      initializeArrayKernel<<<GRID_SIZE, BLOCK_SIZE>>>(aOnStorageDevice[i], initA, arrayLength);
      initializeArrayKernel<<<GRID_SIZE, BLOCK_SIZE>>>(bOnStorageDevice[i], initB, arrayLength);
      checkCudaErrors(hipDeviceSynchronize());
    } else {
      checkCudaErrors(hipSetDevice(COMPUTE_DEVICE_ID));
      checkCudaErrors(hipMallocAsync(&aOnComputeDevice[i], arraySize, dataMovementStream));
      checkCudaErrors(hipMallocAsync(&bOnComputeDevice[i], arraySize, dataMovementStream));
      initializeArrayKernel<<<GRID_SIZE, BLOCK_SIZE, 0, dataMovementStream>>>(aOnComputeDevice[i], initA, arrayLength);
      initializeArrayKernel<<<GRID_SIZE, BLOCK_SIZE, 0, dataMovementStream>>>(bOnComputeDevice[i], initB, arrayLength);
      checkCudaErrors(hipDeviceSynchronize());
    }
  }

  // Compute
  checkCudaErrors(hipSetDevice(COMPUTE_DEVICE_ID));

  auto prefetchEvents = std::make_unique<hipEvent_t[]>(numberOfKernels / prefetchCycleLength);
  for (int i = 0; i < numberOfKernels / prefetchCycleLength; i++) {
    checkCudaErrors(hipEventCreate(&prefetchEvents[i]));
  }

  CudaEventClock clock;
  clock.start(computeStream);

  for (int i = 0; i < numberOfKernels; i++) {
    if (i % prefetchCycleLength == 1) {
      if (i + prefetchCycleLength < numberOfKernels) {
        checkCudaErrors(hipMallocAsync(&aOnComputeDevice[i + prefetchCycleLength], arraySize, dataMovementStream));
        checkCudaErrors(hipMallocAsync(&bOnComputeDevice[i + prefetchCycleLength], arraySize, dataMovementStream));
        checkCudaErrors(hipMemcpyAsync(aOnComputeDevice[i + prefetchCycleLength], aOnStorageDevice[i + prefetchCycleLength], arraySize, hipMemcpyDeviceToDevice, dataMovementStream));
        checkCudaErrors(hipMemcpyAsync(bOnComputeDevice[i + prefetchCycleLength], bOnStorageDevice[i + prefetchCycleLength], arraySize, hipMemcpyDeviceToDevice, dataMovementStream));
        checkCudaErrors(hipEventRecord(prefetchEvents[(i - 1) / prefetchCycleLength], dataMovementStream));
      }
      if (i != 1) {
        checkCudaErrors(hipStreamWaitEvent(computeStream, prefetchEvents[(i - 1) / prefetchCycleLength - 1]));
      }
    }

    checkCudaErrors(hipMallocAsync(&cOnComputeDevice[i], arraySize, computeStream));
    addKernel<<<GRID_SIZE, BLOCK_SIZE, 0, computeStream>>>(aOnComputeDevice[i], bOnComputeDevice[i], cOnComputeDevice[i]);
    checkResultKernel<<<1, 1, 0, computeStream>>>(cOnComputeDevice[i], expectedC);
    checkCudaErrors(hipFreeAsync(aOnComputeDevice[i], computeStream));
    checkCudaErrors(hipFreeAsync(bOnComputeDevice[i], computeStream));
    checkCudaErrors(hipFreeAsync(cOnComputeDevice[i], computeStream));
  }

  clock.end(computeStream);
  checkCudaErrors(hipStreamSynchronize(computeStream));

  const float runningTime = clock.getTimeInSeconds();
  const float bandwidth = static_cast<float>(arraySize) * 3.0 * numberOfKernels / 1e9 / runningTime;
  LOG_TRACE_WITH_INFO("Total running time (s): %.6f", runningTime);
  LOG_TRACE_WITH_INFO("Bandwidth (GB/s): %.2f", bandwidth);

  for (int i = 0; i < numberOfKernels / prefetchCycleLength; i++) {
    checkCudaErrors(hipEventDestroy(prefetchEvents[i]));
  }

  checkCudaErrors(hipStreamDestroy(computeStream));
  checkCudaErrors(hipStreamDestroy(dataMovementStream));
  disablePeerAccessForNvlink();
}

int main(int argc, char **argv) {
  auto cmdl = argh::parser(argc, argv);

  size_t arraySize;
  cmdl("array-size", 1'073'741'824ull) >> arraySize;  // 1GiB by default

  int numberOfKernels;
  cmdl("number-of-kernels", 21) >> numberOfKernels;  // 21 kernels in total by default: 0th kernel, 1st kernel, ..., 20th kernel.

  int prefetchCycleLength;
  cmdl("prefetch-cycle-length", 4) >> prefetchCycleLength;  // Prefetch the 5th, 9th, 13th, ... kernels by default

  runOptimizedStreamWithNvlink(arraySize, numberOfKernels, prefetchCycleLength);

  return 0;
}
